#include <fixedBodyData.h>
#include <sphere.h>
#include <plane.h>
#include <utilities.cuh>

FixedBodyData::FixedBodyData() {}

FixedBodyData::FixedBodyData(int _threadsPerBlock, const std::vector<FixedBody*>& fixedBodies) : threadsPerBlock(_threadsPerBlock) {
    numSpheres = 0;
    numPlanes = 0;
    for (auto fixedBody : fixedBodies) {
        switch (fixedBody->getType())
        {
        case BodyType::Sphere:
            numSpheres++;
            break;
        case BodyType::Plane:
            numPlanes++;
            break;
        default:
            break;
        }
    }
    if (numSpheres > 0) {
        hipMalloc(&dev_spheres, numSpheres * sizeof(Sphere));
    }
    if (numSpheres > 0) {
        hipMalloc(&dev_planes, numPlanes * sizeof(Plane));
    }
    int sphereIdx = 0;
    int floorIdx = 0;
    for (auto fixedBody : fixedBodies) {
        switch (fixedBody->getType())
        {
        case BodyType::Sphere:
            hipMemcpy(dev_spheres + sphereIdx, (Sphere*)fixedBody, sizeof(Sphere), hipMemcpyHostToDevice);
            sphereIdx++;
            break;
        case BodyType::Plane:
            hipMemcpy(dev_planes + floorIdx, (Plane*)fixedBody, sizeof(Plane), hipMemcpyHostToDevice);
            floorIdx++;
            break;
        default:
            break;
        }
    }
}

FixedBodyData::~FixedBodyData() {
    hipFree(dev_spheres);
}

void FixedBodyData::HandleCollisions(glm::vec3* X, glm::vec3* V, int numVerts, float muT, float muN) {
    int numBlocks = (numVerts + threadsPerBlock - 1) / threadsPerBlock;
    if (numSpheres > 0) {
        handleSphereCollision << <numBlocks, threadsPerBlock >> > (X, V, numVerts, dev_spheres, numSpheres, muT, muN);
    }
    if (numPlanes > 0) {
        handleFloorCollision << <numBlocks, threadsPerBlock >> > (X, V, numVerts, dev_planes, numPlanes, muT, muN);
    }
}
