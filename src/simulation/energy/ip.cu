#include <energy/ip.h>
#include <energy/corotated.h>
#include <hip/hip_runtime.h>

IPEnergy::IPEnergy(const SolverData<double>& solverData) : inertia(solverData, nnz, solverData.numVerts, solverData.mass),
elastic(new CorotatedEnergy<double>(solverData, nnz))
{
    hipMalloc(&gradient, sizeof(double) * solverData.numVerts * 3);
    hipMalloc(&hessianVal, sizeof(double) * nnz);
    hipMalloc(&hessianRowIdx, sizeof(int) * nnz);
    hipMalloc(&hessianColIdx, sizeof(int) * nnz);
    inertia.SetHessianPtr(hessianVal, hessianRowIdx, hessianColIdx);
    elastic->SetHessianPtr(hessianVal, hessianRowIdx, hessianColIdx);
}

IPEnergy::~IPEnergy()
{
    hipFree(gradient);
    hipFree(hessianVal);
    hipFree(hessianRowIdx);
    hipFree(hessianColIdx);
}

double IPEnergy::Val(const glm::dvec3* Xs, const SolverData<double>& solverData, double h2) const
{
    return inertia.Val(Xs, solverData) + h2 * (gravity.Val(Xs, solverData) + elastic->Val(Xs, solverData));
}

void IPEnergy::Gradient(const SolverData<double>& solverData, double h2) const
{
    hipMemset(gradient, 0, sizeof(double) * solverData.numVerts * 3);
    inertia.Gradient(gradient, solverData, 1);
    gravity.Gradient(gradient, solverData, h2);
    elastic->Gradient(gradient, solverData, h2);
}

void IPEnergy::Hessian(const SolverData<double>& solverData, double h2) const
{
    inertia.Hessian(solverData, 1);
    gravity.Hessian(solverData, h2);
    elastic->Hessian(solverData, h2);
}