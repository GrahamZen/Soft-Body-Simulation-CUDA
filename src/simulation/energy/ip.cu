#include <energy/ip.h>
#include <energy/corotated.h>
#include <hip/hip_runtime.h>

IPEnergy::IPEnergy(const SolverData<double>& solverData, double dHat) : inertia(solverData, nnz, solverData.numVerts, solverData.mass),
elastic(new CorotatedEnergy<double>(solverData, nnz)), barrier(solverData, nnz, dHat)
{
    hipMalloc(&gradient, sizeof(double) * solverData.numVerts * 3);
    hipMalloc(&hessianVal, sizeof(double) * nnz);
    hipMalloc(&hessianRowIdx, sizeof(int) * nnz);
    hipMalloc(&hessianColIdx, sizeof(int) * nnz);
    inertia.SetHessianPtr(hessianVal, hessianRowIdx, hessianColIdx);
    barrier.SetHessianPtr(hessianVal, hessianRowIdx, hessianColIdx);
    elastic->SetHessianPtr(hessianVal, hessianRowIdx, hessianColIdx);
}

IPEnergy::~IPEnergy()
{
    hipFree(gradient);
    hipFree(hessianVal);
    hipFree(hessianRowIdx);
    hipFree(hessianColIdx);
}

double IPEnergy::Val(const glm::dvec3* Xs, const SolverData<double>& solverData, double h2) const
{
    return inertia.Val(Xs, solverData) + h2 * (gravity.Val(Xs, solverData) + elastic->Val(Xs, solverData) + barrier.Val(Xs, solverData));
}

void IPEnergy::Gradient(const SolverData<double>& solverData, double h2) const
{
    hipMemset(gradient, 0, sizeof(double) * solverData.numVerts * 3);
    inertia.Gradient(gradient, solverData, 1);
    gravity.Gradient(gradient, solverData, h2);
    elastic->Gradient(gradient, solverData, h2);
    barrier.Gradient(gradient, solverData, h2);
}

void IPEnergy::Hessian(const SolverData<double>& solverData, double h2) const
{
    hipMemset(hessianVal, 0, sizeof(double) * nnz);
    inertia.Hessian(solverData, 1);
    gravity.Hessian(solverData, h2);
    elastic->Hessian(solverData, h2);
    barrier.Hessian(solverData, h2);
}

double IPEnergy::InitStepSize(const SolverData<double>& solverData, double* p) const
{
    return barrier.InitStepSize(solverData, p);
}
