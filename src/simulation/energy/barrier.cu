#include "hip/hip_runtime.h"
#include <energy/barrier.h>
#include <solver/solverUtil.cuh>
#include <fixedBodyData.h>
#include <plane.h>
#include <cylinder.h>
#include <sphere.h>
#include <glm/glm.hpp>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>

namespace Barrier {
    template <typename HighP>
    __global__ void hessianKern(HighP* hessianVal, int* hessianRowIdx, int* hessianColIdx, glm::tvec3<HighP>* X, int numVerts,
        Plane* planes, int numPlanes, Cylinder* cylinders, int numCylinders, Sphere* spheres, int numSpheres,
        HighP dhat, HighP* contact_area, HighP coef) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numVerts)
            return;
        glm::tvec3<HighP> x = X[idx];
        for (int j = 0; j < numPlanes; j++) {
            const Plane& plane = planes[j];
            glm::tvec3<HighP> floorPos = glm::tvec3<HighP>(plane.m_model[3]);
            glm::tvec3<HighP> floorUp = plane.m_floorUp;
            HighP d = glm::dot(x - floorPos, floorUp);
            if (d < dhat)
            {
                glm::tmat3x3<HighP> hess = coef * contact_area[idx] * dhat * plane.kappa / (2 * d * d * dhat) * (d + dhat) * glm::outerProduct(floorUp, floorUp);
                for (int k = 0; k < 3; k++)
                {
                    for (int l = 0; l < 3; l++)
                    {
                        int rowIdx = idx * 3 + k;
                        int colIdx = idx * 3 + l;
                        int index = idx * 9 + k * 3 + l;
                        hessianVal[index] = hess[k][l];
                        hessianRowIdx[index] = rowIdx;
                        hessianColIdx[index] = colIdx;
                    }
                }
            }
        }
        for (int j = 0; j < numCylinders; j++) {
            const Cylinder cy = cylinders[j];
            glm::tvec3<HighP> axis = glm::tvec3<HighP>(glm::normalize(cy.m_model * glm::vec4(0.f, 1.f, 0.f, 0.f)));
            glm::tmat3x3<HighP> nnT = glm::tmat3x3<HighP>(1.f) - glm::outerProduct(axis, axis);
            glm::tvec3<HighP> cylinderCenter = glm::tvec3<HighP>(cy.m_model[3]);
            HighP cylinderRadius = cy.m_radius;
            glm::tvec3<HighP> n = nnT * (x - cylinderCenter);
            HighP d = glm::length(n) - cylinderRadius;
            glm::tvec3<HighP> normal = glm::normalize(n);
            if (d < dhat)
            {
                glm::tmat3x3<HighP> hess = coef * contact_area[idx] * dhat * cy.kappa / (2 * d * d * dhat) * (d + dhat) * glm::outerProduct(normal, normal);
                for (int k = 0; k < 3; k++)
                {
                    for (int l = 0; l < 3; l++)
                    {
                        int rowIdx = idx * 3 + k;
                        int colIdx = idx * 3 + l;
                        int index = idx * 9 + k * 3 + l;
                        hessianVal[index] += hess[k][l];
                        hessianRowIdx[index] = rowIdx;
                        hessianColIdx[index] = colIdx;
                    }
                }
            }
        }
        for (int j = 0; j < numSpheres; j++) {
            const Sphere& sphere = spheres[j];
            glm::tvec3<HighP> sphereCenter = glm::tvec3<HighP>(sphere.m_model[3]);
            HighP sphereRadius = sphere.m_radius;
            glm::tvec3<HighP> n = x - sphereCenter;
            HighP d = glm::length(n) - sphereRadius;
            glm::tvec3<HighP> normal = glm::normalize(n);
            if (d < dhat)
            {
                glm::tmat3x3<HighP> hess = coef * contact_area[idx] * dhat * sphere.kappa / (2 * d * d * dhat) * (d + dhat) * glm::outerProduct(normal, normal);
                for (int k = 0; k < 3; k++)
                {
                    for (int l = 0; l < 3; l++)
                    {
                        int rowIdx = idx * 3 + k;
                        int colIdx = idx * 3 + l;
                        int index = idx * 9 + k * 3 + l;
                        hessianVal[index] += hess[k][l];
                        hessianRowIdx[index] = rowIdx;
                        hessianColIdx[index] = colIdx;
                    }
                }
            }
        }
    }

    template <typename HighP>
    __global__ void gradientKern(HighP* grad, glm::tvec3<HighP>* X, int numVerts, Plane* planes, int numPlanes, Cylinder* cylinders, int numCylinders, Sphere* spheres, int numSpheres,
        HighP dhat, HighP* contact_area, HighP coef) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numVerts)
            return;
        const glm::tvec3<HighP> x = X[idx];
        for (int j = 0; j < numPlanes; j++) {
            const Plane& plane = planes[j];
            glm::tvec3<HighP> floorPos = glm::tvec3<HighP>(plane.m_model[3]);
            glm::tvec3<HighP> floorUp = plane.m_floorUp;
            HighP d = glm::dot(x - floorPos, floorUp);
            if (d < dhat)
            {
                HighP s = d / dhat;
                glm::tvec3<HighP> gradient = coef * contact_area[idx] * dhat * (plane.kappa / 2 * (log(s) / dhat + (s - 1) / d)) * floorUp;
                grad[idx * 3] += gradient.x;
                grad[idx * 3 + 1] += gradient.y;
                grad[idx * 3 + 2] += gradient.z;
            }
        }
        for (int j = 0; j < numCylinders; j++) {
            const Cylinder cy = cylinders[j];
            glm::tvec3<HighP> axis = glm::tvec3<HighP>(glm::normalize(cy.m_model * glm::vec4(0.f, 1.f, 0.f, 0.f)));
            glm::tmat3x3<HighP> nnT = glm::tmat3x3<HighP>(1.f) - glm::outerProduct(axis, axis);
            glm::tvec3<HighP> cylinderCenter = glm::tvec3<HighP>(cy.m_model[3]);
            HighP cylinderRadius = cy.m_radius;
            glm::tvec3<HighP> n = nnT * (x - cylinderCenter);
            HighP d = glm::length(n) - cylinderRadius;
            glm::tvec3<HighP> normal = glm::normalize(n);
            if (d < dhat)
            {
                HighP s = d / dhat;
                glm::tvec3<HighP> gradient = coef * contact_area[idx] * dhat * (cy.kappa / 2 * (log(s) / dhat + (s - 1) / d)) * normal;
                grad[idx * 3] += gradient.x;
                grad[idx * 3 + 1] += gradient.y;
                grad[idx * 3 + 2] += gradient.z;
            }
        }
        for (int j = 0; j < numSpheres; j++) {
            const Sphere& sphere = spheres[j];
            glm::tvec3<HighP> sphereCenter = glm::tvec3<HighP>(sphere.m_model[3]);
            HighP sphereRadius = sphere.m_radius;
            glm::tvec3<HighP> n = x - sphereCenter;
            HighP d = glm::length(n) - sphereRadius;
            glm::tvec3<HighP> normal = glm::normalize(n);
            if (d < dhat)
            {
                HighP s = d / dhat;
                glm::tvec3<HighP> gradient = coef * contact_area[idx] * dhat * (sphere.kappa / 2 * (log(s) / dhat + (s - 1) / d)) * normal;
                grad[idx * 3] += gradient.x;
                grad[idx * 3 + 1] += gradient.y;
                grad[idx * 3 + 2] += gradient.z;
            }
        }
    }
}

template <typename HighP>
int BarrierEnergy<HighP>::NNZ(const SolverData<HighP>& solverData) const { return solverData.numVerts * 9; }

template <typename HighP>
BarrierEnergy<HighP>::BarrierEnergy(const SolverData<HighP>& solverData, int& hessianIdxOffset, HighP dhat) :dhat(dhat), Energy<HighP>(hessianIdxOffset)
{
    hessianIdxOffset += NNZ(solverData);
}

template <typename HighP>
HighP BarrierEnergy<HighP>::Val(const glm::tvec3<HighP>* Xs, const SolverData<HighP>& solverData) const {
    const Plane* planes = solverData.pFixedBodies->dev_planes;
    const Cylinder* cylinders = solverData.pFixedBodies->dev_cylinders;
    const Sphere* spheres = solverData.pFixedBodies->dev_spheres;
    int numSpheres = solverData.pFixedBodies->numSpheres;
    int numCylinders = solverData.pFixedBodies->numCylinders;
    int numPlanes = solverData.pFixedBodies->numPlanes;
    HighP dhat = this->dhat;
    HighP sum = thrust::transform_reduce(
        thrust::counting_iterator<indexType>(0),
        thrust::counting_iterator<indexType>(solverData.numVerts),
        [=]__host__ __device__(indexType vertIdx) {
        const glm::tvec3<HighP> x = Xs[vertIdx];
        HighP sum = 0.0;
        for (int j = 0; j < numPlanes; j++)
        {
            const Plane& plane = planes[j];
            glm::tvec3<HighP> floorPos = glm::tvec3<HighP>(plane.m_model[3]);
            glm::tvec3<HighP> floorUp = plane.m_floorUp;
            HighP d = glm::dot(x - floorPos, floorUp);
            if (d < dhat)
            {
                HighP s = d / dhat;
                sum += solverData.contact_area[vertIdx] * dhat * plane.kappa * 0.5 * (s - 1) * log(s);
            }
        }
        for (int j = 0; j < numCylinders; j++) {
            const Cylinder cy = cylinders[j];
            glm::tvec3<HighP> axis = glm::tvec3<HighP>(glm::normalize(cy.m_model * glm::vec4(0.f, 1.f, 0.f, 0.f)));
            glm::tmat3x3<HighP> nnT = glm::tmat3x3<HighP>(1.f) - glm::outerProduct(axis, axis);
            glm::tvec3<HighP> cylinderCenter = glm::tvec3<HighP>(cy.m_model[3]);
            HighP cylinderRadius = cy.m_radius;
            glm::tvec3<HighP> n = nnT * (x - cylinderCenter);
            HighP d = glm::length(n) - cylinderRadius;
            if (d < dhat)
            {
                HighP s = d / dhat;
                sum += solverData.contact_area[vertIdx] * dhat * cy.kappa * 0.5 * (s - 1) * log(s);
            }
        }
        for (int j = 0; j < numSpheres; j++) {
            const Sphere& sphere = spheres[j];
            glm::tvec3<HighP> sphereCenter = glm::tvec3<HighP>(sphere.m_model[3]);
            HighP sphereRadius = sphere.m_radius;
            glm::tvec3<HighP> n = x - sphereCenter;
            HighP d = glm::length(n) - sphereRadius;
            glm::tvec3<HighP> normal = glm::normalize(n);
            if (d < dhat)
            {
                HighP s = d / dhat;
                sum += solverData.contact_area[vertIdx] * dhat * sphere.kappa * 0.5 * (s - 1) * log(s);
            }
        }
        return sum;
    },
        0.0,
        thrust::plus<HighP>());
    return sum;
}

template<typename HighP>
void BarrierEnergy<HighP>::Gradient(HighP* grad, const SolverData<HighP>& solverData, HighP coef) const
{
    int threadsPerBlock = 256;
    int numBlocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    Barrier::gradientKern << <numBlocks, threadsPerBlock >> > (grad, solverData.X, solverData.numVerts, solverData.pFixedBodies->dev_planes, solverData.pFixedBodies->numPlanes,
        solverData.pFixedBodies->dev_cylinders, solverData.pFixedBodies->numCylinders, solverData.pFixedBodies->dev_spheres, solverData.pFixedBodies->numSpheres, dhat, solverData.contact_area, coef);
}

template <typename HighP>
void BarrierEnergy<HighP>::Hessian(const SolverData<HighP>& solverData, HighP coef) const
{
    int threadsPerBlock = 256;
    int numBlocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    Barrier::hessianKern << <numBlocks, threadsPerBlock >> > (hessianVal, hessianRowIdx, hessianColIdx, solverData.X, solverData.numVerts,
        solverData.pFixedBodies->dev_planes, solverData.pFixedBodies->numPlanes, solverData.pFixedBodies->dev_cylinders, solverData.pFixedBodies->numCylinders, solverData.pFixedBodies->dev_spheres, solverData.pFixedBodies->numSpheres, dhat, solverData.contact_area, coef);
}

template<typename HighP>
HighP BarrierEnergy<HighP>::InitStepSize(const SolverData<HighP>& solverData, HighP* p) const
{
    const Plane* planes = solverData.pFixedBodies->dev_planes;
    const Cylinder* cylinders = solverData.pFixedBodies->dev_cylinders;
    const Sphere* spheres = solverData.pFixedBodies->dev_spheres;
    int numSpheres = solverData.pFixedBodies->numSpheres;
    int numCylinders = solverData.pFixedBodies->numCylinders;
    int numPlanes = solverData.pFixedBodies->numPlanes;
    return thrust::transform_reduce(
        thrust::counting_iterator<indexType>(0),
        thrust::counting_iterator<indexType>(solverData.numVerts),
        [=]__host__ __device__(indexType vertIdx) {
        HighP alpha = 1.0;
        glm::tvec3<HighP> localP{ -p[vertIdx * 3], -p[vertIdx * 3 + 1], -p[vertIdx * 3 + 2] };
        const glm::tvec3<HighP> x = solverData.X[vertIdx];
        for (int j = 0; j < numPlanes; j++)
        {
            glm::tvec3<HighP> floorUp = planes[j].m_floorUp;
            glm::tvec3<HighP> floorPos = glm::tvec3<HighP>(planes[j].m_model[3]);
            HighP p_n = glm::dot(localP, floorUp);
            if (p_n < 0)
            {
                alpha = min(alpha, 0.9 * glm::dot(floorUp, x - floorPos) / -p_n);
            }
        }
        for (int j = 0; j < numCylinders; j++)
        {
            const Cylinder cy = cylinders[j];
            glm::tvec3<HighP> axis = glm::tvec3<HighP>(glm::normalize(cy.m_model * glm::vec4(0.f, 1.f, 0.f, 0.f)));
            glm::tmat3x3<HighP> nnT = glm::tmat3x3<HighP>(1.f) - glm::outerProduct(axis, axis);
            glm::tvec3<HighP> cylinderCenter = glm::tvec3<HighP>(cy.m_model[3]);
            HighP cylinderRadius = cy.m_radius;
            glm::tvec3<HighP> n = nnT * (x - cylinderCenter);
            HighP p_n = glm::dot(localP, n);
            if (p_n < 0)
            {
                glm::tvec3<HighP> pp = nnT * localP;
                HighP pp2 = glm::dot(pp, pp);
                HighP n2 = glm::dot(n, n);
                HighP ndotpp = glm::dot(n, pp);
                alpha = min(alpha, 0.9 * (-ndotpp - sqrt(ndotpp * ndotpp - pp2 * (n2 - cylinderRadius * cylinderRadius))) / pp2);
            }
        }
        for (int j = 0; j < numSpheres; j++) {
            const Sphere& sphere = spheres[j];
            glm::tvec3<HighP> sphereCenter = glm::tvec3<HighP>(sphere.m_model[3]);
            HighP sphereRadius = sphere.m_radius;
            glm::tvec3<HighP> n = x - sphereCenter;
            HighP p_n = glm::dot(localP, n);
            if (p_n < 0)
            {
                HighP ndotp = glm::dot(n, localP);
                HighP pp = glm::dot(localP, localP);
                HighP nn = glm::dot(n, n);
                alpha = min(alpha, 0.9 * (-ndotp - sqrt(ndotp * ndotp - pp * (nn - sphereRadius * sphereRadius))) / pp);
            }
        }
        return alpha;
    },
        1.0,
        thrust::minimum<HighP>());
}

template class BarrierEnergy<float>;
template class BarrierEnergy<double>;