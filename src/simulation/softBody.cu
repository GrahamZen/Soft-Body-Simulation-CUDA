#include <simulation/softBody.h>

SoftBody::~SoftBody()
{
    if (solverData.inv_Dm)
        hipFree(solverData.inv_Dm);
    if (solverData.dev_ExtForce)
        hipFree(solverData.dev_ExtForce);
    if (solverData.V0)
        hipFree(solverData.V0);
}

void SoftBody::Reset()
{
    hipMemcpy(solverData.X, solverData.X0, sizeof(glm::vec3) * solverData.numVerts, hipMemcpyDeviceToDevice);
    hipMemcpy(solverData.XTilt, solverData.X0, sizeof(glm::vec3) * solverData.numVerts, hipMemcpyDeviceToDevice);
    hipMemset(solverData.V, 0, sizeof(glm::vec3) * solverData.numVerts);
}
