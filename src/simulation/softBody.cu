#include <simulation/softBody.h>

SoftBody::~SoftBody()
{
    hipFree(solverData.Tet);
    hipFree(solverData.Force);
    hipFree(solverData.V);
    hipFree(solverData.inv_Dm);

    delete solver;
}

void SoftBody::Reset()
{
    hipMemcpy(solverData.X, solverData.X0, sizeof(glm::vec3) * solverData.numVerts, hipMemcpyDeviceToDevice);
    hipMemcpy(solverData.XTilt, solverData.X0, sizeof(glm::vec3) * solverData.numVerts, hipMemcpyDeviceToDevice);
    hipMemset(solverData.V, 0, sizeof(glm::vec3) * solverData.numVerts);
}
