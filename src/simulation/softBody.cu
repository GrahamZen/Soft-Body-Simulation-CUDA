#include "hip/hip_runtime.h"
#include <simulationContext.h>
#include <glm/glm.hpp>
#include <glm/gtx/norm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <utilities.cuh>

SoftBody::SoftBody(SimulationCUDAContext* context, SoftBodyAttribute& _attrib, SoftBodyData* dataPtr)
    : mpSimContext(context), attrib(_attrib), Tet(dataPtr->Tets), X(dataPtr->dev_X), numTets(dataPtr->numTets), numVerts(dataPtr->numVerts)
{
    vertices.resize(numVerts);
    hipMemcpy(vertices.data(), X, sizeof(glm::vec3) * numVerts, hipMemcpyDeviceToHost);
    idx.resize(numTets * 4);
    hipMemcpy(idx.data(), Tet, sizeof(int) * numTets * 4, hipMemcpyDeviceToHost);

    hipMalloc((void**)&X0, sizeof(glm::vec3) * numVerts);
    hipMemcpy(X0, X, sizeof(glm::vec3) * numVerts, hipMemcpyDeviceToDevice);
    Mesh::numTets = numTets;

    InitModel();

    hipMalloc((void**)&Force, sizeof(glm::vec3) * numVerts);
    hipMemset(Force, 0, sizeof(glm::vec3) * numVerts);
    hipMalloc((void**)&V, sizeof(glm::vec3) * numVerts);
    hipMemset(V, 0, sizeof(glm::vec3) * numVerts);
    hipMalloc((void**)&inv_Dm, sizeof(glm::mat4) * numTets);
    hipMalloc((void**)&V_sum, sizeof(glm::vec3) * numVerts);
    hipMemset(V_sum, 0, sizeof(glm::vec3) * numVerts);
    createTetrahedron();
    hipMalloc((void**)&V_num, sizeof(int) * numVerts);
    hipMemset(V_num, 0, sizeof(int) * numVerts);
    hipMalloc((void**)&V0, sizeof(float) * numTets);
    hipMemset(V0, 0, sizeof(float) * numTets);
    int threadsPerBlock = 64;
    int blocks = (numTets + threadsPerBlock - 1) / threadsPerBlock;
    computeInvDmV0 << < blocks, threadsPerBlock >> > (V0, inv_Dm, numTets, X, Tet);
}

SoftBody::~SoftBody()
{
    hipFree(Tet);
    hipFree(Force);
    hipFree(V);
    hipFree(inv_Dm);
    hipFree(V_sum);

    hipFree(sn);
    hipFree(b);
    hipFree(masses);

    if (useEigen)
    {
        free(bHost);
    }
    else
    {
        hipFree(ARow);
        hipFree(ACol);
        hipFree(AVal);
    }
}


void SoftBody::PDSolver()
{
    if (!solverReady)
    {
        solverPrepare();
        solverReady = true;
    }
    PDSolverStep();
}


void SoftBody::Laplacian_Smoothing(float blendAlpha)
{
    hipMemset(V_sum, 0, sizeof(glm::vec3) * numVerts);
    hipMemset(V_num, 0, sizeof(int) * numVerts);
    int threadsPerBlock = 64;
    int blocks = (numTets + threadsPerBlock - 1) / threadsPerBlock;
    LaplacianGatherKern << < blocks, threadsPerBlock >> > (V, V_sum, V_num, numTets, Tet);
    LaplacianKern << < (numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (V, V_sum, V_num, numVerts, Tet, blendAlpha);
}

void SoftBody::Update()
{
    _Update();
}

void SoftBody::Reset()
{
    hipMemset(Force, 0, sizeof(glm::vec3) * numVerts);
    hipMemset(V, 0, sizeof(glm::vec3) * numVerts);
    hipMemcpy(X, X0, sizeof(glm::vec3) * numVerts, hipMemcpyDeviceToDevice);
    InitModel();
}

void SoftBody::_Update()
{
    int threadsPerBlock = 64;
    AddGravity << <(numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (Force, V, attrib.mass, numVerts, jump);
    // Laplacian_Smoothing();
    glm::vec3 floorPos = glm::vec3(0.0f, -4.0f, 0.0f);
    glm::vec3 floorUp = glm::vec3(0.0f, 1.0f, 0.0f);
    //ComputeForces << <(numTets + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (Force, X, Tet, numTets, inv_Dm, stiffness_0, stiffness_1);
    if (useGPUSolver)
    {
        PDSolver();
    }
    else
    {
        Eigen::MatrixXf positionsFloat;
        using RowMajorMatrixX3f = Eigen::Matrix<float, Eigen::Dynamic, 3, Eigen::RowMajor>;
        RowMajorMatrixX3f velocitiesFloat(numVerts, 3);
        positionsFloat.resizeLike(model.positions().transpose());
        hipMemcpy(positionsFloat.data(), X, numVerts * sizeof(glm::vec3), hipMemcpyDeviceToHost);
        hipMemcpy(velocitiesFloat.data(), V, numVerts * sizeof(glm::vec3), hipMemcpyDeviceToHost);
        model.positions() = positionsFloat.transpose().cast<double>();
        model.velocity() = velocitiesFloat.cast<double>();
        PdSolver();
        positionsFloat = model.positions().cast<float>().transpose();
        hipMemcpy(X, positionsFloat.data(), numVerts * sizeof(glm::vec3), hipMemcpyHostToDevice);
        velocitiesFloat = model.velocity().cast<float>();
        hipMemcpy(V, velocitiesFloat.data(), numVerts * sizeof(glm::vec3), hipMemcpyHostToDevice);
    }
    HandleFloorCollision << <(numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (X, V, numVerts, floorPos, floorUp, attrib.muT, attrib.muN);
}