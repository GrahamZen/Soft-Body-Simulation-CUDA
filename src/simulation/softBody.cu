#include "hip/hip_runtime.h"
#include <softBody.h>
#include <simulation/simulationContext.h>
#include <glm/glm.hpp>
#include <glm/gtx/norm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <utilities.cuh>
#include <pdSolver.h>

SoftBody::SoftBody(SimulationCUDAContext* context, SolverAttribute& _attrib, SolverData* dataPtr)
    :solverData(*dataPtr), solver(new PdSolver{ context, _attrib, solverData }), threadsPerBlock(context->GetThreadsPerBlock())
{
    Mesh::numTets = solverData.numTets;
    Mesh::numTris = solverData.numTris;
    if (numTris == 0)
        createTetrahedron();
    else
        createMesh();
}

SoftBody::~SoftBody()
{
    hipFree(solverData.Tet);
    hipFree(solverData.Force);
    hipFree(solverData.V);
    hipFree(solverData.inv_Dm);

    delete solver;
}

void SoftBody::Update()
{
    solver->Update(solverData);
}

void SoftBody::Reset()
{
    hipMemcpy(solverData.X, solverData.X0, sizeof(glm::vec3) * solverData.numVerts, hipMemcpyDeviceToDevice);
    hipMemcpy(solverData.XTilt, solverData.X0, sizeof(glm::vec3) * solverData.numVerts, hipMemcpyDeviceToDevice);
    hipMemset(solverData.V, 0, sizeof(glm::vec3) * solverData.numVerts);
}
