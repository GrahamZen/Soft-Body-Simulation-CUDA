#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <sceneStructs.h>
#include <glm/glm.hpp>
#include <glm/gtx/norm.hpp>
#include <glm/gtc/matrix_transform.hpp> 
#include <simulationContext.h>
#include <utilities.h>
#include <utilities.cuh>
#include <iostream>
#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)

static GuiDataContainer* guiData = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
SimulationCUDAContext::SimulationCUDAContext()
{
}

SimulationCUDAContext::~SimulationCUDAContext()
{
    for (auto softbody : softBodies) {
        delete softbody;
    }
}

void SimulationCUDAContext::Update()
{
    for (auto softbody : softBodies) {
        softbody->Update();
        glm::vec3* pos;
        glm::vec4* nor;
        softbody->mapDevicePtr(&pos, &nor);
        dim3 numThreadsPerBlock(softbody->getTetNumber() / 32 + 1);
        PopulatePos << <numThreadsPerBlock, 32 >> > (pos, softbody->getX(), softbody->getTet(), softbody->getTetNumber());
        RecalculateNormals << <softbody->getTetNumber() * 4 / 32 + 1, 32 >> > (nor, pos, 4 * softbody->getTetNumber());
        softbody->unMapDevicePtr();
    }
}

void SimulationCUDAContext::Reset()
{
    for (auto softbody : softBodies) {
        softbody->Reset();
    }
}

void SimulationCUDAContext::addSoftBody(SoftBody* softbody)
{
    softBodies.push_back(softbody);
}

void SimulationCUDAContext::draw(ShaderProgram* shaderProgram)
{
    for (auto softBody : softBodies)
        shaderProgram->draw(*softBody, 0);
}

SoftBody::SoftBody(const char* nodeFileName, const char* eleFileName, SimulationCUDAContext* context, const glm::vec3& pos, const glm::vec3& scale,
    const glm::vec3& rot, float mass, float stiffness_0, float stiffness_1, float damp, float muN, float muT, bool centralize, int startIndex)
    : simContext(context), mass(mass), stiffness_0(stiffness_0), stiffness_1(stiffness_1), damp(damp), muN(muN), muT(muT)
{
    std::vector<glm::vec3> vertices = loadNodeFile(nodeFileName, centralize);
    number = vertices.size();
    hipMalloc((void**)&X, sizeof(glm::vec3) * number);
    hipMemcpy(X, vertices.data(), sizeof(glm::vec3) * number, hipMemcpyHostToDevice);

    // transform
    glm::mat4 model = glm::mat4(1.0f);
    model = glm::translate(model, pos);
    model = glm::scale(model, scale);
    model = glm::rotate(model, glm::radians(rot.x), glm::vec3(1.0f, 0.0f, 0.0f));
    model = glm::rotate(model, glm::radians(rot.y), glm::vec3(0.0f, 1.0f, 0.0f));
    model = glm::rotate(model, glm::radians(rot.z), glm::vec3(0.0f, 0.0f, 1.0f));
    int threadsPerBlock = 64;
    int blocks = (number + threadsPerBlock - 1) / threadsPerBlock;
    TransformVertices << < blocks, threadsPerBlock >> > (X, model, number);

    hipMalloc((void**)&X0, sizeof(glm::vec3) * number);
    hipMemcpy(X0, X, sizeof(glm::vec3) * number, hipMemcpyDeviceToDevice);

    std::vector<GLuint> idx = loadEleFile(eleFileName, startIndex);
    tet_number = idx.size() / 4;
    hipMalloc((void**)&Tet, sizeof(GLuint) * idx.size());
    hipMemcpy(Tet, idx.data(), sizeof(GLuint) * idx.size(), hipMemcpyHostToDevice);

    Mesh::tet_number = tet_number;

    hipMalloc((void**)&Force, sizeof(glm::vec3) * number);
    hipMemset(Force, 0, sizeof(glm::vec3) * number);
    hipMalloc((void**)&V, sizeof(glm::vec3) * number);
    hipMemset(V, 0, sizeof(glm::vec3) * number);
    hipMalloc((void**)&inv_Dm, sizeof(glm::mat4) * tet_number);
    hipMalloc((void**)&V_sum, sizeof(glm::vec3) * number);
    hipMemset(V_sum, 0, sizeof(glm::vec3) * number);
    createTetrahedron();
    hipMalloc((void**)&V_num, sizeof(int) * number);
    hipMemset(V_num, 0, sizeof(int) * number);
    blocks = (tet_number + threadsPerBlock - 1) / threadsPerBlock;
    computeInvDm << < blocks, threadsPerBlock >> > (inv_Dm, tet_number, X, Tet);
}

SoftBody::~SoftBody()
{
    hipFree(X);
    hipFree(Tet);
    hipFree(Force);
    hipFree(V);
    hipFree(inv_Dm);
    hipFree(V_sum);
}

void SoftBody::mapDevicePtr(glm::vec3** bufPosDevPtr, glm::vec4** bufNorDevPtr)
{
    size_t size;
    hipGraphicsMapResources(1, &cuda_bufPos_resource, 0);
    hipGraphicsResourceGetMappedPointer((void**)bufPosDevPtr, &size, cuda_bufPos_resource);

    hipGraphicsMapResources(1, &cuda_bufNor_resource, 0);
    hipGraphicsResourceGetMappedPointer((void**)bufNorDevPtr, &size, cuda_bufNor_resource);
}

void SoftBody::unMapDevicePtr()
{
    hipGraphicsUnmapResources(1, &cuda_bufPos_resource, 0);
    hipGraphicsUnmapResources(1, &cuda_bufNor_resource, 0);
}

void SoftBody::Laplacian_Smoothing(float blendAlpha)
{
    hipMemset(V_sum, 0, sizeof(glm::vec3) * number);
    hipMemset(V_num, 0, sizeof(int) * number);
    int threadsPerBlock = 64;
    int blocks = (tet_number + threadsPerBlock - 1) / threadsPerBlock;
    LaplacianGatherKern << < blocks, threadsPerBlock >> > (V, V_sum, V_num, tet_number, Tet);
    LaplacianKern << < (number + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (V, V_sum, V_num, number, Tet, blendAlpha);
}

void SoftBody::Update()
{
    for (int l = 0; l < 10; l++)
        _Update();
}

void SoftBody::Reset()
{
    hipMemset(Force, 0, sizeof(glm::vec3) * number);
    hipMemset(V, 0, sizeof(glm::vec3) * number);
    hipMemcpy(X, X0, sizeof(glm::vec3) * number, hipMemcpyDeviceToDevice);
}

void SoftBody::_Update()
{
    int threadsPerBlock = 64;
    AddGravity << <(number + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (Force, V, mass, number, jump);
    Laplacian_Smoothing();
    glm::vec3 floorPos = glm::vec3(0.0f, -4.0f, 0.0f);
    glm::vec3 floorUp = glm::vec3(0.0f, 1.0f, 0.0f);
    ComputeForces << <(tet_number + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (Force, X, Tet, tet_number, inv_Dm, stiffness_0, stiffness_1);
    UpdateParticles << <(number + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (X, V, Force, number, mass, simContext->getDt(), damp, floorPos, floorUp, muT, muN);
}
