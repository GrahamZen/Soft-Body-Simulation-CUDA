#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <sceneStructs.h>
#include <simulationContext.h>
#include <utilities.cuh>
#include <iostream>
#include <glm/gtc/matrix_transform.hpp>
#include <bvh.cuh>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)

// TODO: static variables for device memory, any extra info you need, etc
// ...

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */

SimulationCUDAContext::~SimulationCUDAContext()
{
    for (auto name : namesSoftBodies) {
        delete[]name;
    }
    hipFree(dev_Xs);
    hipFree(dev_Tets);
    hipFree(dev_Vs);
    hipFree(dev_Fs);
    hipFree(dev_X0s);
    hipFree(dev_XTilts);
    for (auto softbody : softBodies) {
        delete softbody;
    }
}

AABB SimulationCUDAContext::GetAABB() const
{
    thrust::device_ptr<glm::vec3> dev_ptr(dev_Xs);
    thrust::device_ptr<glm::vec3> dev_ptrTilts(dev_XTilts);
    return computeBoundingBox(dev_ptr, dev_ptr + numVerts).expand(computeBoundingBox(dev_ptrTilts, dev_ptrTilts + numVerts));
}

int SimulationCUDAContext::GetVertCnt() const {
    return numVerts;
}

int SimulationCUDAContext::GetTetCnt() const {
    return numTets;
}

void DataLoader::CollectData(const char* nodeFileName, const char* eleFileName, const char* faceFileName, const glm::vec3& pos, const glm::vec3& scale, const glm::vec3& rot,
    bool centralize, int startIndex, SoftBody::SoftBodyAttribute attrib)
{
    SoftBodyData softBodyData;
    auto vertices = loadNodeFile(nodeFileName, centralize, softBodyData.numVerts);
    hipMalloc((void**)&softBodyData.dev_X, sizeof(glm::vec3) * softBodyData.numVerts);
    hipMemcpy(softBodyData.dev_X, vertices.data(), sizeof(glm::vec3) * softBodyData.numVerts, hipMemcpyHostToDevice);

    // transform
    glm::mat4 model = glm::mat4(1.0f);
    model = glm::translate(model, pos);
    model = glm::scale(model, scale);
    model = glm::rotate(model, glm::radians(rot.x), glm::vec3(1.0f, 0.0f, 0.0f));
    model = glm::rotate(model, glm::radians(rot.y), glm::vec3(0.0f, 1.0f, 0.0f));
    model = glm::rotate(model, glm::radians(rot.z), glm::vec3(0.0f, 0.0f, 1.0f));
    int blocks = (softBodyData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    TransformVertices << < blocks, threadsPerBlock >> > (softBodyData.dev_X, model, softBodyData.numVerts);

    auto tetIdx = loadEleFile(eleFileName, startIndex, softBodyData.numTets);
    hipMalloc((void**)&softBodyData.Tets, sizeof(GLuint) * tetIdx.size());
    hipMemcpy(softBodyData.Tets, tetIdx.data(), sizeof(GLuint) * tetIdx.size(), hipMemcpyHostToDevice);
    auto triIdx = loadFaceFile(faceFileName, startIndex, softBodyData.numTris);
    if (!triIdx.empty()) {
        hipMalloc((void**)&softBodyData.Tris, sizeof(GLuint) * triIdx.size());
        hipMemcpy(softBodyData.Tris, triIdx.data(), sizeof(GLuint) * triIdx.size(), hipMemcpyHostToDevice);
    }
    else {
        softBodyData.Tris = nullptr;
        softBodyData.numTris = 0;
    }
    CollectEdges(triIdx);
    totalNumVerts += softBodyData.numVerts;
    totalNumTets += softBodyData.numTets;

    m_softBodyData.push_back({ softBodyData, attrib });
}

void DataLoader::AllocData(std::vector<int>& startIndices, glm::vec3*& gX, glm::vec3*& gX0, glm::vec3*& gXTilt,
    glm::vec3*& gV, glm::vec3*& gF, GLuint*& gEdges, GLuint*& gTet, int& numVerts, int& numTets)
{
    numVerts = totalNumVerts;
    numTets = totalNumTets;
    hipMalloc((void**)&gX, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gX0, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gXTilt, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gV, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gF, sizeof(glm::vec3) * totalNumVerts);
    hipMemset(gV, 0, sizeof(glm::vec3) * totalNumVerts);
    hipMemset(gF, 0, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gEdges, sizeof(GLuint) * totalNumEdges * 2);
    hipMalloc((void**)&gTet, sizeof(GLuint) * totalNumTets * 4);
    int vertOffset = 0, tetOffset = 0, edgeOffset = 0;
    thrust::device_ptr<GLuint> dev_gTetPtr(gTet);
    thrust::device_ptr<GLuint> dev_gEdgesPtr(gEdges);
    for (int i = 0; i < m_softBodyData.size(); i++)
    {
        auto& softBodyData = m_softBodyData[i];
        startIndices.push_back(vertOffset);
        auto& data = softBodyData.first;
        hipMemcpy(gX + vertOffset, data.dev_X, sizeof(glm::vec3) * data.numVerts, hipMemcpyDeviceToDevice);
        thrust::transform(data.Tets, data.Tets + data.numTets * 4, dev_gTetPtr + tetOffset, [vertOffset] __device__(GLuint x) {
            return x + vertOffset;
        });
        hipMemcpy(gEdges + edgeOffset, m_edges[i].data(), sizeof(GLuint) * m_edges[i].size(), hipMemcpyHostToDevice);
        thrust::transform(dev_gEdgesPtr + edgeOffset, dev_gEdgesPtr + edgeOffset + m_edges[i].size(), dev_gEdgesPtr + edgeOffset,
            [vertOffset] __device__(GLuint x) {
            return x + vertOffset;
        });
        hipFree(data.dev_X);
        data.dev_X = gX + vertOffset;
        data.dev_X0 = gX0 + vertOffset;
        data.dev_XTilt = gXTilt + vertOffset;
        data.dev_V = gV + vertOffset;
        data.dev_F = gF + vertOffset;
        vertOffset += data.numVerts;
        tetOffset += data.numTets * 4;
        edgeOffset += m_edges[i].size();
    }
    hipMemcpy(gX0, gX, sizeof(glm::vec3) * totalNumVerts, hipMemcpyDeviceToDevice);
    hipMemcpy(gXTilt, gX, sizeof(glm::vec3) * totalNumVerts, hipMemcpyDeviceToDevice);
}

void SimulationCUDAContext::CCD()
{
    float* tIs = m_bvh.DetectCollisionCandidates(dev_Tets, dev_Xs, dev_XTilts);
    int blocks = (numVerts + threadsPerBlock - 1) / threadsPerBlock;
    //CCDKernel << <blocks, threadsPerBlock >> > (dev_Xs, dev_XTilts, tIs, numVerts);
}

void SimulationCUDAContext::Update()
{
    for (auto softbody : softBodies) {
        softbody->Update();
    }
    if (context->guiData->handleCollision)
        m_bvh.BuildBVHTree(GetAABB(), numTets, dev_Xs, dev_XTilts, dev_Tets);
    HandleFloorCollision << <(numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (dev_XTilts, dev_Vs, numVerts, glm::vec3(0.f, floorY, 0.f), floorUp, muT, muN);
    //if(context->guiData->handleCollision)
    //    CCD();
    hipMemcpy(dev_Xs, dev_XTilts, sizeof(glm::vec3) * numVerts, hipMemcpyDeviceToDevice);
    if (context->guiData->ObjectVis) {
        PrepareRenderData();
    }
    if (context->guiData->handleCollision && context->guiData->BVHVis)
        m_bvh.PrepareRenderData();
}

void SimulationCUDAContext::PrepareRenderData() {
    for (auto softbody : softBodies) {
        glm::vec3* pos;
        glm::vec4* nor;
        softbody->Mesh::mapDevicePtr(&pos, &nor);
        if (softbody->getTriNumber() == 0) {
            dim3 numThreadsPerBlock(softbody->getTetNumber() / threadsPerBlock + 1);
            PopulatePos << <numThreadsPerBlock, threadsPerBlock >> > (pos, softbody->getX(), softbody->getTet(), softbody->getTetNumber());
            RecalculateNormals << <softbody->getTetNumber() * 4 / threadsPerBlock + 1, threadsPerBlock >> > (nor, pos, 4 * softbody->getTetNumber());
            softbody->Mesh::unMapDevicePtr();
        }
        else {
            dim3 numThreadsPerBlock(softbody->getTriNumber() / threadsPerBlock + 1);
            PopulateTriPos << <numThreadsPerBlock, threadsPerBlock >> > (pos, softbody->getX(), softbody->getTri(), softbody->getTriNumber());
            RecalculateNormals << <softbody->getTriNumber() / threadsPerBlock + 1, threadsPerBlock >> > (nor, pos, softbody->getTriNumber());
            softbody->Mesh::unMapDevicePtr();
        }
    }
}


