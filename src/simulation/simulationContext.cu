#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <sceneStructs.h>
#include <simulationContext.h>
#include <utilities.cuh>
#include <iostream>


#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)

// TODO: static variables for device memory, any extra info you need, etc
// ...

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */

void SimulationCUDAContext::Update()
{
    //m_bvh.BuildBVHTree(0, GetAABB(), GetTetCnt(), softBodies);
    for (auto softbody : softBodies) {
        softbody->Update();
        glm::vec3* pos;
        glm::vec4* nor;
        softbody->mapDevicePtr(&pos, &nor);
        dim3 numThreadsPerBlock(softbody->getTetNumber() / 32 + 1);

        PopulatePos << <numThreadsPerBlock, 32 >> > (pos, softbody->getX(), softbody->getTet(), softbody->getTetNumber());
        RecalculateNormals << <softbody->getTetNumber() * 4 / 32 + 1, 32 >> > (nor, pos, 4 * softbody->getTetNumber());
        softbody->unMapDevicePtr();
    }
}

void SoftBody::mapDevicePtr(glm::vec3** bufPosDevPtr, glm::vec4** bufNorDevPtr)
{
    size_t size;
    hipGraphicsMapResources(1, &cuda_bufPos_resource, 0);
    hipGraphicsResourceGetMappedPointer((void**)bufPosDevPtr, &size, cuda_bufPos_resource);

    hipGraphicsMapResources(1, &cuda_bufNor_resource, 0);
    hipGraphicsResourceGetMappedPointer((void**)bufNorDevPtr, &size, cuda_bufNor_resource);
}

void SoftBody::unMapDevicePtr()
{
    hipGraphicsUnmapResources(1, &cuda_bufPos_resource, 0);
    hipGraphicsUnmapResources(1, &cuda_bufNor_resource, 0);
}