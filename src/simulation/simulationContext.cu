#include "hip/hip_runtime.h"
#include <utilities.cuh>
#include <simulation/simulationContext.h>
#include <simulation/softBody.h>
#include <glm/gtc/matrix_transform.hpp>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <simulation/MshLoader.h>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)

// TODO: static variables for device memory, any extra info you need, etc
// ...

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */

__global__ void CCDKernel(glm::vec3* X, glm::vec3* XTilt, glm::vec3* V, dataType* tI, glm::vec3* normal, float muT, float muN, int numVerts);

SimulationCUDAContext::~SimulationCUDAContext()
{
    for (auto name : namesSoftBodies) {
        delete[]name;
    }
    hipFree(mSolverData.X);
    hipFree(mSolverData.Tet);
    hipFree(mSolverData.V);
    hipFree(mSolverData.Force);
    hipFree(mSolverData.X0);
    hipFree(mSolverData.XTilt);
    for (auto softbody : softBodies) {
        delete softbody;
    }
    hipFree(dev_Normals);
    delete mSolver;
}

int SimulationCUDAContext::GetVertCnt() const {
    return mSolverData.numVerts;
}

int SimulationCUDAContext::GetNumQueries() const {
    return mCollisionDetection.GetNumQueries();
}

int SimulationCUDAContext::GetTetCnt() const {
    return mSolverData.numTets;
}

void DataLoader::CollectData(const char* nodeFileName, const char* eleFileName, const char* faceFileName, const glm::vec3& pos, const glm::vec3& scale, const glm::vec3& rot,
    bool centralize, int startIndex, SolverAttribute attrib)
{
    SolverData softBodyData;
    auto vertices = loadNodeFile(nodeFileName, centralize, softBodyData.numVerts);
    hipMalloc((void**)&softBodyData.X, sizeof(glm::vec3) * softBodyData.numVerts);
    hipMemcpy(softBodyData.X, vertices.data(), sizeof(glm::vec3) * softBodyData.numVerts, hipMemcpyHostToDevice);

    // transform
    glm::mat4 model = glm::mat4(1.0f);
    model = glm::translate(model, pos);
    model = glm::scale(model, scale);
    model = glm::rotate(model, glm::radians(rot.x), glm::vec3(1.0f, 0.0f, 0.0f));
    model = glm::rotate(model, glm::radians(rot.y), glm::vec3(0.0f, 1.0f, 0.0f));
    model = glm::rotate(model, glm::radians(rot.z), glm::vec3(0.0f, 0.0f, 1.0f));
    int blocks = (softBodyData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    TransformVertices << < blocks, threadsPerBlock >> > (softBodyData.X, model, softBodyData.numVerts);

    auto tetIdx = loadEleFile(eleFileName, startIndex, softBodyData.numTets);
    hipMalloc((void**)&softBodyData.Tet, sizeof(indexType) * tetIdx.size());
    hipMemcpy(softBodyData.Tet, tetIdx.data(), sizeof(indexType) * tetIdx.size(), hipMemcpyHostToDevice);
    auto triIdx = loadFaceFile(faceFileName, startIndex, softBodyData.numTris);
    if (!triIdx.empty()) {
        hipMalloc((void**)&softBodyData.Tri, sizeof(indexType) * triIdx.size());
        hipMemcpy(softBodyData.Tri, triIdx.data(), sizeof(indexType) * triIdx.size(), hipMemcpyHostToDevice);
    }
    else {
        softBodyData.Tri = nullptr;
        softBodyData.numTris = 0;
    }
    CollectEdges(triIdx);
    totalNumVerts += softBodyData.numVerts;
    totalNumTets += softBodyData.numTets;

    m_softBodyData.push_back({ softBodyData, attrib });
}

void DataLoader::CollectData(const char* mshFileName, const glm::vec3& pos, const glm::vec3& scale, const glm::vec3& rot,
    bool centralize, int startIndex, SolverAttribute attrib)
{
    SolverData softBodyData;
    igl::MshLoader _loader(mshFileName);
    auto nodes = _loader.get_nodes();
    std::vector<float> vertices(nodes.size());
    softBodyData.numVerts = nodes.size() / 3;
    std::transform(nodes.begin(), nodes.end(), vertices.begin(), [](igl::MshLoader::Float f) {
        return static_cast<float>(f);
        });
    hipMalloc((void**)&softBodyData.X, sizeof(glm::vec3) * softBodyData.numVerts);
    hipMemcpy(softBodyData.X, vertices.data(), sizeof(glm::vec3) * softBodyData.numVerts, hipMemcpyHostToDevice);

    // transform
    glm::mat4 model = glm::mat4(1.0f);
    model = glm::translate(model, pos);
    model = glm::scale(model, scale);
    model = glm::rotate(model, glm::radians(rot.x), glm::vec3(1.0f, 0.0f, 0.0f));
    model = glm::rotate(model, glm::radians(rot.y), glm::vec3(0.0f, 1.0f, 0.0f));
    model = glm::rotate(model, glm::radians(rot.z), glm::vec3(0.0f, 0.0f, 1.0f));
    int blocks = (softBodyData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    TransformVertices << < blocks, threadsPerBlock >> > (softBodyData.X, model, softBodyData.numVerts);

    auto elements = _loader.get_elements();
    std::vector<indexType> tetIdx(elements.size());
    std::transform(elements.begin(), elements.end(), tetIdx.begin(), [](int i) {
        return static_cast<indexType>(i);
        });
    softBodyData.numTets = tetIdx.size() / 4;
    hipMalloc((void**)&softBodyData.Tet, sizeof(indexType) * tetIdx.size());
    hipMemcpy(softBodyData.Tet, tetIdx.data(), sizeof(indexType) * tetIdx.size(), hipMemcpyHostToDevice);
    std::vector<indexType> triIdx;
    if (!triIdx.empty()) {
        hipMalloc((void**)&softBodyData.Tri, sizeof(indexType) * triIdx.size());
        hipMemcpy(softBodyData.Tri, triIdx.data(), sizeof(indexType) * triIdx.size(), hipMemcpyHostToDevice);
    }
    else {
        softBodyData.Tri = nullptr;
        softBodyData.numTris = 0;
    }
    CollectEdges(triIdx);
    totalNumVerts += softBodyData.numVerts;
    totalNumTets += softBodyData.numTets;

    m_softBodyData.push_back({ softBodyData, attrib });
}

void DataLoader::AllocData(std::vector<int>& startIndices, glm::vec3*& gX, glm::vec3*& gX0, glm::vec3*& gXTilt,
    glm::vec3*& gV, glm::vec3*& gF, indexType*& gEdges, indexType*& gTet, indexType*& gTetFather, int& numVerts, int& numTets)
{
    numVerts = totalNumVerts;
    numTets = totalNumTets;
    hipMalloc((void**)&gX, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gX0, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gXTilt, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gV, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gF, sizeof(glm::vec3) * totalNumVerts);
    hipMemset(gV, 0, sizeof(glm::vec3) * totalNumVerts);
    hipMemset(gF, 0, sizeof(glm::vec3) * totalNumVerts);
    hipMalloc((void**)&gEdges, sizeof(indexType) * totalNumEdges * 2);
    hipMalloc((void**)&gTet, sizeof(indexType) * totalNumTets * 4);
    hipMalloc((void**)&gTetFather, sizeof(indexType) * totalNumTets);
    int vertOffset = 0, tetOffset = 0, edgeOffset = 0;
    thrust::device_ptr<indexType> dev_gTetPtr(gTet);
    thrust::device_ptr<indexType> dev_gEdgesPtr(gEdges);
    thrust::device_ptr<indexType> dev_gTetFatherPtr(gTetFather);
    for (int i = 0; i < m_softBodyData.size(); i++)
    {
        auto& softBodyData = m_softBodyData[i];
        startIndices.push_back(vertOffset);
        auto& data = softBodyData.first;
        hipMemcpy(gX + vertOffset, data.X, sizeof(glm::vec3) * data.numVerts, hipMemcpyDeviceToDevice);
        thrust::transform(data.Tet, data.Tet + data.numTets * 4, dev_gTetPtr + tetOffset, [vertOffset] __device__(indexType x) {
            return x + vertOffset;
        });
        thrust::fill(dev_gTetFatherPtr + tetOffset / 4, dev_gTetFatherPtr + tetOffset / 4 + data.numTets, i);
        hipMemcpy(gEdges + edgeOffset, m_edges[i].data(), sizeof(indexType) * m_edges[i].size(), hipMemcpyHostToDevice);
        thrust::transform(dev_gEdgesPtr + edgeOffset, dev_gEdgesPtr + edgeOffset + m_edges[i].size(), dev_gEdgesPtr + edgeOffset,
            [vertOffset] __device__(indexType x) {
            return x + vertOffset;
        });
        hipFree(data.X);
        data.X = gX + vertOffset;
        data.X0 = gX0 + vertOffset;
        data.XTilt = gXTilt + vertOffset;
        data.V = gV + vertOffset;
        data.Force = gF + vertOffset;
        vertOffset += data.numVerts;
        tetOffset += data.numTets * 4;
        edgeOffset += m_edges[i].size();
    }
    hipMemcpy(gX0, gX, sizeof(glm::vec3) * totalNumVerts, hipMemcpyDeviceToDevice);
    hipMemcpy(gXTilt, gX, sizeof(glm::vec3) * totalNumVerts, hipMemcpyDeviceToDevice);
}

void SimulationCUDAContext::CCD()
{
    mCollisionDetection.DetectCollision(dev_tIs, dev_Normals);
    int blocks = (mSolverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    CCDKernel << <blocks, threadsPerBlock >> > (mSolverData.X, mSolverData.XTilt, mSolverData.V, dev_tIs, dev_Normals, mSolverParams.muT, mSolverParams.muN, mSolverData.numVerts);
}

void SimulationCUDAContext::PrepareRenderData() {
    for (auto softbody : softBodies) {
        glm::vec3* pos;
        glm::vec4* nor;
        softbody->Mesh::MapDevicePtr(&pos, &nor);
        if (softbody->GetNumTris() == 0) {
            dim3 numThreadsPerBlock(softbody->GetNumTets() / threadsPerBlock + 1);
            PopulatePos << <numThreadsPerBlock, threadsPerBlock >> > (pos, softbody->GetSolverData().X, softbody->GetSolverData().Tet, softbody->GetNumTets());
            RecalculateNormals << <softbody->GetNumTets() * 4 / threadsPerBlock + 1, threadsPerBlock >> > (nor, pos, 4 * softbody->GetNumTets());
            softbody->Mesh::UnMapDevicePtr();
        }
        else {
            dim3 numThreadsPerBlock(softbody->GetNumTris() / threadsPerBlock + 1);
            PopulateTriPos << <numThreadsPerBlock, threadsPerBlock >> > (pos, softbody->GetSolverData().X, softbody->GetSolverData().Tri, softbody->GetNumTris());
            RecalculateNormals << <softbody->GetNumTris() / threadsPerBlock + 1, threadsPerBlock >> > (nor, pos, softbody->GetNumTris());
            softbody->Mesh::UnMapDevicePtr();
        }
    }
}