#include <IPC/ipc.h>
#include <energy/corotated.h>
#include <hip/hip_runtime.h>

IPCSolver::IPCSolver(int threadsPerBlock, const SolverData<double>& solverData)
    :FEMSolver(threadsPerBlock)
{
    hipMalloc((void**)&solverData.V0, sizeof(float) * solverData.numTets);
    hipMemset(solverData.V0, 0, sizeof(float) * solverData.numTets);
    hipMalloc((void**)&solverData.DmInv, sizeof(glm::mat4) * solverData.numTets); 
    int blocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    computeInvDmV0 << < blocks, threadsPerBlock >> > (solverData.V0, solverData.DmInv, solverData.numTets, solverData.X, solverData.Tet);
 
    energies.push_back(new InertiaEnergy<double>(solverData, nnz, solverData.numVerts, solverData.mass));
    energies.push_back(new GravityEnergy<double>);
    energies.push_back(new CorotatedEnergy<double>(solverData, nnz));
    hipMalloc(&gradient, sizeof(double) * solverData.numVerts * 3);
    hipMalloc(&hessianVal, sizeof(double) * nnz);
    hipMalloc(&hessianRowIdx, sizeof(int) * nnz);
    hipMalloc(&hessianColIdx, sizeof(int) * nnz);
    for (int i = 0; i < energies.size(); i++)
    {
        energies[i]->SetHessianPtr(hessianVal, hessianRowIdx, hessianColIdx);
    }
}

IPCSolver::~IPCSolver()
{
    hipFree(gradient);
    hipFree(hessianVal);
    hipFree(hessianRowIdx);
    hipFree(hessianColIdx);
}

void IPCSolver::Update(SolverData<double>& solverData, SolverParams& solverParams)
{
    SolverStep(solverData, solverParams);
}

void IPCSolver::SolverPrepare(SolverData<double>& solverData, SolverParams& solverParams)
{
}

void IPCSolver::SolverStep(SolverData<double>& solverData, SolverParams& solverParams)
{
    for (int i = 0; i < energies.size(); i++)
    {
        energies[i]->Gradient(gradient, solverData);
        energies[i]->Hessian(solverData);
    }
}
