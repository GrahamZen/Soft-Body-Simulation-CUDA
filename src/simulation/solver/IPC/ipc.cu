#include <IPC/ipc.h>
#include <hip/hip_runtime.h>

IPCSolver::IPCSolver(int threadsPerBlock, const SolverData<double>& solverData)
    :FEMSolver(threadsPerBlock), numVerts(solverData.numVerts), inertia(numVerts, solverData.mass)
{
    hipMalloc(&gradient, sizeof(double) * numVerts * 3);
}

IPCSolver::~IPCSolver()
{
    hipFree(gradient);
}

void IPCSolver::Update(SolverData<double>& solverData, SolverParams& solverParams)
{
}

void IPCSolver::SolverPrepare(SolverData<double>& solverData, SolverParams& solverParams)
{
}

void IPCSolver::SolverStep(SolverData<double>& solverData, SolverParams& solverParams)
{
}
