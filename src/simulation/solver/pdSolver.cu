#include <pdSolver.h>
#include <utilities.cuh>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <simulationContext.h>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)

void PdSolver::SolverPrepare(SolverData& solverData, SolverAttribute& attrib)
{
    int vertBlocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    float dt = mcrpSimContext->GetDt();
    float const m_1_dt2 = attrib.mass / (dt * dt);
    int len = solverData.numVerts * 3 + 48 * solverData.numTets;
    int ASize = 3 * solverData.numVerts;

    hipMalloc((void**)&sn, sizeof(float) * ASize);
    hipMalloc((void**)&b, sizeof(float) * ASize);
    hipMalloc((void**)&masses, sizeof(float) * ASize);

    int* AIdx;
    hipMalloc((void**)&AIdx, sizeof(int) * len);
    hipMemset(AIdx, 0, sizeof(int) * len);

    float* tmpVal;
    hipMalloc((void**)&tmpVal, sizeof(int) * len);
    hipMemset(tmpVal, 0, sizeof(int) * len);

    computeSiTSi << < tetBlocks, threadsPerBlock >> > (AIdx, tmpVal, V0, solverData.inv_Dm, solverData.Tet, attrib.stiffness_0, solverData.numTets, solverData.numVerts);
    setMDt_2 << < vertBlocks, threadsPerBlock >> > (AIdx, tmpVal, 48 * solverData.numTets, m_1_dt2, solverData.numVerts);

    bHost = (float*)malloc(sizeof(float) * ASize);

    int* AIdxHost = (int*)malloc(sizeof(int) * len);
    float* tmpValHost = (float*)malloc(sizeof(float) * len);

    hipMemcpy(AIdxHost, AIdx, sizeof(int) * len, hipMemcpyDeviceToHost);
    hipMemcpy(tmpValHost, tmpVal, sizeof(float) * len, hipMemcpyDeviceToHost);

    std::vector<Eigen::Triplet<float>> A_triplets;

    for (auto i = 0; i < len; ++i)
    {
        A_triplets.push_back({ AIdxHost[i] / ASize, AIdxHost[i] % ASize, tmpValHost[i] });
    }
    Eigen::SparseMatrix<float> A(ASize, ASize);

    A.setFromTriplets(A_triplets.begin(), A_triplets.end());
    cholesky_decomposition_.compute(A);

    free(AIdxHost);
    free(tmpValHost);

    int* newIdx;
    float* newVal;

    hipMalloc((void**)&newIdx, sizeof(int) * len);
    hipMalloc((void**)&newVal, sizeof(float) * len);

    thrust::sort_by_key(thrust::device, AIdx, AIdx + len, tmpVal);


    thrust::pair<int*, float*> newEnd = thrust::reduce_by_key(thrust::device, AIdx, AIdx + len, tmpVal, newIdx, newVal);

    int* ARow;
    int* ACol;
    float* AVal;

    nnzNumber = newEnd.first - newIdx;
    std::cout << nnzNumber << std::endl;

    hipMalloc((void**)&ARow, sizeof(int) * nnzNumber);
    hipMemset(ARow, 0, sizeof(int) * nnzNumber);

    hipMalloc((void**)&ACol, sizeof(int) * nnzNumber);
    hipMemset(ACol, 0, sizeof(int) * nnzNumber);

    hipMalloc((void**)&AVal, sizeof(float) * nnzNumber);
    hipMemcpy(AVal, newVal, sizeof(float) * nnzNumber, hipMemcpyDeviceToDevice);

    int* ARowTmp;
    hipMalloc((void**)&ARowTmp, sizeof(int) * nnzNumber);
    hipMemset(ARowTmp, 0, sizeof(int) * nnzNumber);

    int blocks = (nnzNumber + threadsPerBlock - 1) / threadsPerBlock;

    initAMatrix << < blocks, threadsPerBlock >> > (newIdx, ARowTmp, ACol, ASize, nnzNumber);

    // transform ARow into csr format
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseXcoo2csr(handle, ARowTmp, nnzNumber, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t descrA;
    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    size_t cholSize = 0;
    size_t internalSize = 0;
    cusolverSpCreateCsrcholInfo(&d_info);
    cusolverSpXcsrcholAnalysis(cusolverHandle, ASize, nnzNumber, descrA, ARow, ACol, d_info);
    cusolverSpScsrcholBufferInfo(cusolverHandle, ASize, nnzNumber, descrA, AVal, ARow, ACol, d_info, &internalSize, &cholSize);
    hipMalloc(&buffer_gpu, sizeof(char) * cholSize);
    cusolverSpScsrcholFactor(cusolverHandle, ASize, nnzNumber, descrA, AVal, ARow, ACol, d_info, buffer_gpu);

    hipFree(newIdx);
    hipFree(newVal);
    hipFree(ARowTmp);
    hipFree(ARow);
    hipFree(ACol);
    hipFree(AVal);

    hipFree(AIdx);
    hipFree(tmpVal);
}


void PdSolver::SolverStep(SolverData& solverData, SolverAttribute& attrib)
{

    float dt = mcrpSimContext->GetDt();
    float const dtInv = 1.0f / dt;
    float const dt2 = dt * dt;
    float const dt2_m_1 = dt2 / attrib.mass;
    float const m_1_dt2 = 1.f / dt2_m_1;


    int vertBlocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;

    glm::vec3 gravity{ 0.0f, -mcrpSimContext->GetGravity() * attrib.mass, 0.0f };
    thrust::device_ptr<glm::vec3> dev_ptr(solverData.dev_ExtForce);
    thrust::fill(thrust::device, dev_ptr, dev_ptr + solverData.numVerts, gravity);
    //computeSn << < vertBlocks, threadsPerBlock >> > (sn, dt, dt2_m_1, solverData.X, solverData.V, thrust::raw_pointer_cast(solverData.dev_ExtForce.data()), masses, m_1_dt2, solverData.numVerts);
    computeSn << < vertBlocks, threadsPerBlock >> > (sn, dt, dt2_m_1, solverData.X, solverData.V, solverData.dev_ExtForce, masses, m_1_dt2, solverData.numVerts);
    checkCUDAError("computeSn");
    for (int i = 0; i < mcrpSimContext->GetNumIterations(); i++)
    {
        hipMemset(b, 0, sizeof(float) * solverData.numVerts * 3);
        computeLocal << < tetBlocks, threadsPerBlock >> > (V0, attrib.stiffness_0, b, solverData.inv_Dm, sn, solverData.Tet, solverData.numTets);
        addM_h2Sn << < vertBlocks, threadsPerBlock >> > (b, masses, solverData.numVerts);

        if (mcrpSimContext->IsEigenGlobalSolver())
        {
            hipMemcpy(bHost, b, sizeof(float) * (solverData.numVerts * 3), hipMemcpyDeviceToHost);
            Eigen::VectorXf bh = Eigen::Map<Eigen::VectorXf, Eigen::Unaligned>(bHost, solverData.numVerts * 3);
            Eigen::VectorXf res = cholesky_decomposition_.solve(bh);
            hipMemcpy(sn, res.data(), sizeof(float) * (solverData.numVerts * 3), hipMemcpyHostToDevice);
        }
        else
        {
            cusolverSpScsrcholSolve(cusolverHandle, solverData.numVerts * 3, b, sn, d_info, buffer_gpu);
        }
    }

    updateVelPos << < vertBlocks, threadsPerBlock >> > (sn, dtInv, solverData.XTilt, solverData.V, solverData.numVerts);
}


void PdSolver::Update(SolverData& solverData, SolverAttribute& attrib)
{
    AddExternal << <(solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (solverData.V, solverData.numVerts, attrib.jump, attrib.mass, mcrpSimContext->GetExtForce().jump);
    if (!solverReady)
    {
        SolverPrepare(solverData, attrib);
        solverReady = true;
    }
    SolverStep(solverData, attrib);
}


void PdSolver::Laplacian_Smoothing(float blendAlpha)
{
    //hipMemset(V_sum, 0, sizeof(glm::vec3) * solverData.numVerts);
    //hipMemset(V_num, 0, sizeof(int) * solverData.numVerts);
    //int blocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    //LaplacianGatherKern << < blocks, threadsPerBlock >> > (V, V_sum, V_num, solverData.numTets, solverData.Tet);
    //LaplacianKern << < (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (V, V_sum, V_num, solverData.numVerts, solverData.Tet, blendAlpha);
}