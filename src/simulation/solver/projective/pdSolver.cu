#include <simulation/solver/linear/cholesky.h>
#include <simulation/solver/projective/pdSolver.h>
#include <simulation/solver/projective/pdUtil.cuh>
#include <fixedBodyData.h>
#include <collision/bvh.h>

#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

PdSolver::PdSolver(int threadsPerBlock, const SolverData& solverData) : FEMSolver(threadsPerBlock)
{
    hipMalloc((void**)&solverData.dev_ExtForce, sizeof(glm::vec3) * solverData.numVerts);
    hipMemset(solverData.dev_ExtForce, 0, sizeof(glm::vec3) * solverData.numVerts);
    hipMalloc((void**)&solverData.V0, sizeof(float) * solverData.numTets);
    hipMemset(solverData.V0, 0, sizeof(float) * solverData.numTets);
    hipMalloc((void**)&solverData.inv_Dm, sizeof(glm::mat4) * solverData.numTets);

    int blocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    PdUtil::computeInvDmV0 << < blocks, threadsPerBlock >> > (solverData.V0, solverData.inv_Dm, solverData.numTets, solverData.X, solverData.Tet);
}

PdSolver::~PdSolver() {
    if (ls) {
        free(ls);
    }
    hipFree(sn);
    hipFree(b);
    hipFree(masses);
    free(bHost);
}

void PdSolver::SolverPrepare(SolverData& solverData, SolverParams& solverParams)
{
    int vertBlocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    float dt = solverParams.dt;
    float const m_1_dt2 = solverParams.solverAttr.mass / (dt * dt);
    int len = solverData.numVerts * 3 + 48 * solverData.numTets;
    int ASize = 3 * solverData.numVerts;

    hipMalloc((void**)&sn, sizeof(float) * ASize);
    hipMalloc((void**)&b, sizeof(float) * ASize);
    hipMalloc((void**)&masses, sizeof(float) * ASize);

    int* AIdx;
    hipMalloc((void**)&AIdx, sizeof(int) * len);
    hipMemset(AIdx, 0, sizeof(int) * len);

    float* tmpVal;
    hipMalloc((void**)&tmpVal, sizeof(int) * len);
    hipMemset(tmpVal, 0, sizeof(int) * len);

    PdUtil::computeSiTSi << < tetBlocks, threadsPerBlock >> > (AIdx, tmpVal, solverData.V0, solverData.inv_Dm, solverData.Tet, solverParams.solverAttr.stiffness_0, solverData.numTets, solverData.numVerts);
    PdUtil::setMDt_2 << < vertBlocks, threadsPerBlock >> > (AIdx, tmpVal, 48 * solverData.numTets, m_1_dt2, solverData.numVerts);

    bHost = (float*)malloc(sizeof(float) * ASize);
    int* AIdxHost = (int*)malloc(sizeof(int) * len);
    float* tmpValHost = (float*)malloc(sizeof(float) * len);

    hipMemcpy(AIdxHost, AIdx, sizeof(int) * len, hipMemcpyDeviceToHost);
    hipMemcpy(tmpValHost, tmpVal, sizeof(float) * len, hipMemcpyDeviceToHost);

    std::vector<Eigen::Triplet<float>> A_triplets;

    for (auto i = 0; i < len; ++i)
    {
        A_triplets.push_back({ AIdxHost[i] / ASize, AIdxHost[i] % ASize, tmpValHost[i] });
    }
    Eigen::SparseMatrix<float> A(ASize, ASize);

    A.setFromTriplets(A_triplets.begin(), A_triplets.end());
    cholesky_decomposition_.compute(A);

    ls = new CholeskySpLinearSolver(threadsPerBlock, AIdx, tmpVal, ASize, len);

    hipFree(AIdx);
    hipFree(tmpVal);
}


void PdSolver::SolverStep(SolverData& solverData, SolverParams& solverParams)
{
    float dt = solverParams.dt;
    float const dtInv = 1.0f / dt;
    float const dt2 = dt * dt;
    float const dt2_m_1 = dt2 / solverParams.solverAttr.mass;
    float const m_1_dt2 = 1.f / dt2_m_1;

    int vertBlocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;

    glm::vec3 gravity{ 0.0f, -solverParams.gravity * solverParams.solverAttr.mass, 0.0f };
    thrust::device_ptr<glm::vec3> dev_ptr(solverData.dev_ExtForce);
    thrust::fill(thrust::device, dev_ptr, dev_ptr + solverData.numVerts, gravity);
    PdUtil::computeSn << < vertBlocks, threadsPerBlock >> > (sn, dt, dt2_m_1, solverData.X, solverData.V, solverData.dev_ExtForce, masses, m_1_dt2, solverData.numVerts);
    for (int i = 0; i < solverParams.numIterations; i++)
    {
        hipMemset(b, 0, sizeof(float) * solverData.numVerts * 3);
        PdUtil::computeLocal << < tetBlocks, threadsPerBlock >> > (solverData.V0, solverParams.solverAttr.stiffness_0, b, solverData.inv_Dm, sn, solverData.Tet, solverData.numTets);
        PdUtil::addM_h2Sn << < vertBlocks, threadsPerBlock >> > (b, masses, solverData.numVerts);

        if (useEigen)
        {
            hipMemcpy(bHost, b, sizeof(float) * (solverData.numVerts * 3), hipMemcpyDeviceToHost);
            Eigen::VectorXf bh = Eigen::Map<Eigen::VectorXf, Eigen::Unaligned>(bHost, solverData.numVerts * 3);
            Eigen::VectorXf res = cholesky_decomposition_.solve(bh);
            hipMemcpy(sn, res.data(), sizeof(float) * (solverData.numVerts * 3), hipMemcpyHostToDevice);
        }
        else
        {
            ls->Solve(solverData.numVerts * 3, b, sn);
        }
    }

    PdUtil::updateVelPos << < vertBlocks, threadsPerBlock >> > (sn, dtInv, solverData.XTilde, solverData.V, solverData.numVerts);
}

void PdSolver::Update(SolverData& solverData, SolverParams& solverParams)
{
    AddExternal << <(solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (solverData.V, solverData.numVerts, solverParams.solverAttr.jump, solverParams.solverAttr.mass, solverParams.extForce.jump);
    if (!solverReady)
    {
        SolverPrepare(solverData, solverParams);
        solverReady = true;
    }
    SolverStep(solverData, solverParams);
    if (solverParams.handleCollision) {
        solverParams.pCollisionDetection->DetectCollision(solverData.dev_tIs, solverData.dev_Normals);
        int blocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
        PdUtil::CCDKernel << <blocks, threadsPerBlock >> > (solverData.X, solverData.XTilde, solverData.V, solverData.dev_tIs, solverData.dev_Normals, solverParams.muT, solverParams.muN, solverData.numVerts);
    }else
        hipMemcpy(solverData.X, solverData.XTilde, sizeof(glm::vec3) * solverData.numVerts, hipMemcpyDeviceToDevice);
    solverData.pFixedBodies->HandleCollisions(solverData.XTilde, solverData.V, solverData.numVerts, solverParams.muT, solverParams.muN);
}