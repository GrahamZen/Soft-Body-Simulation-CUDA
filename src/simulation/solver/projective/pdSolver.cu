#include "hip/hip_runtime.h"
#include <simulation/solver/linear/cholesky.h>
#include <simulation/solver/linear/jacobi.h>
#include <simulation/solver/projective/pdSolver.h>
#include <simulation/solver/solverUtil.cuh>
#include <simulation/solver/projective/pdUtil.cuh>
#include <fixedBodyData.h>
#include <collision/bvh.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>

struct gravity_force {
    const float g;
    gravity_force(float _g) : g(_g) {}
    __device__ glm::vec3 operator()(float mass) const { return  glm::vec3{ 0.0f, -g * mass, 0.0f }; }
};

float computeError(thrust::device_ptr<float> sn, thrust::device_ptr<float> sn_old, int size);

PdSolver::PdSolver(int threadsPerBlock, const SolverData<float>& solverData) : FEMSolver(threadsPerBlock, solverData), solverType(SolverType::CuSolverCholesky)
{
    hipMalloc((void**)&solverData.ExtForce, sizeof(glm::vec3) * solverData.numVerts);
    hipMemset(solverData.ExtForce, 0, sizeof(glm::vec3) * solverData.numVerts);
    performanceData = { {"local step", 0.0f}, {"global step", 0.0f}, {"collision handling(fixed)", 0.0f}, {"collision handling(mesh)", 0.0f} };
}

PdSolver::~PdSolver() {
    if (ls) {
        free(ls);
    }
    hipFree(sn);
    hipFree(sn_old);
    hipFree(b);
    hipFree(massDt_2s);
    free(bHost);
}

void PdSolver::SolverPrepare(SolverData<float>& solverData, const SolverParams<float>& solverParams)
{
    int vertBlocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int DBCBlocks = (solverData.numDBC + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    float dt = solverParams.dt;
    int len = solverData.numVerts * 3 + 48 * solverData.numTets;
    int ASize = 3 * solverData.numVerts;
    // positional constraints
    len += solverData.numDBC * 3;
    hipMalloc((void**)&sn, sizeof(float) * ASize);
    hipMalloc((void**)&sn_old, sizeof(float) * ASize);
    hipMalloc((void**)&next_x, sizeof(float) * ASize);
    hipMalloc((void**)&prev_x, sizeof(float) * ASize);
    hipMalloc((void**)&b, sizeof(float) * ASize);
    hipMalloc((void**)&massDt_2s, sizeof(float) * solverData.numVerts);
    hipMalloc((void**)&matrix_diag, sizeof(float) * solverData.numVerts);
    hipMemset(matrix_diag, 0, sizeof(float) * solverData.numVerts);

    int* AColIdx, * ARowIdx;
    hipMalloc((void**)&AColIdx, sizeof(int) * len);
    hipMemset(AColIdx, 0, sizeof(int) * len);
    hipMalloc((void**)&ARowIdx, sizeof(int) * len);
    hipMemset(ARowIdx, 0, sizeof(int) * len);

    float* AVal;
    hipMalloc((void**)&AVal, sizeof(int) * len);
    hipMemset(AVal, 0, sizeof(int) * len);

    size_t offset = 0;
    PdUtil::computeSiTSi << < tetBlocks, threadsPerBlock >> > (ARowIdx, AColIdx, AVal, matrix_diag, solverData.V0, solverData.DmInv, solverData.Tet, solverData.mu, solverData.numTets, solverData.numVerts);
    offset += 48 * solverData.numTets;
    PdUtil::setMDt_2 << < vertBlocks, threadsPerBlock >> > (ARowIdx, AColIdx, AVal, offset, solverData.mass, dt * dt, massDt_2s, solverData.numVerts);
    offset += solverData.numVerts * 3;
    if (solverData.numDBC > 0)
        PdUtil::setOne << < DBCBlocks, threadsPerBlock >> > (solverData.numDBC, solverData.DBC, offset, ARowIdx, AColIdx, AVal, positional_weight);

    bHost = (float*)malloc(sizeof(float) * ASize);
    std::vector<int>ARowIdxHost(len);
    std::vector<int>AColIdxHost(len);
    std::vector<float>tmpValHost(len);

    hipMemcpy(ARowIdxHost.data(), ARowIdx, sizeof(int) * len, hipMemcpyDeviceToHost);
    hipMemcpy(AColIdxHost.data(), AColIdx, sizeof(int) * len, hipMemcpyDeviceToHost);
    hipMemcpy(tmpValHost.data(), AVal, sizeof(float) * len, hipMemcpyDeviceToHost);

    try
    {
        std::vector<Eigen::Triplet<float>> A_triplets;
        for (auto i = 0; i < len; ++i)
        {
            A_triplets.push_back({ ARowIdxHost[i], AColIdxHost[i], tmpValHost[i] });
            const auto& triplet = A_triplets.back();
            if (triplet.row() < 0 || triplet.row() >= ASize ||
                triplet.col() < 0 || triplet.col() >= ASize) {
                throw std::invalid_argument("Triplet contains invalid row or column index.");
            }
        }
        Eigen::SparseMatrix<float> A(ASize, ASize);

        A.setFromTriplets(A_triplets.begin(), A_triplets.end());
        cholesky_decomposition_.compute(A);
        A.makeCompressed();
        // transfer A to coo format ARowIdx, AColIdx, tmpVal
        int nnz = A.nonZeros();
        if (nnz != len) {
            ARowIdxHost.resize(nnz);
            AColIdxHost.resize(nnz);
            tmpValHost.resize(nnz);
        }

        int idx = 0;
        for (int k = 0; k < A.outerSize(); ++k)
        {
            for (Eigen::SparseMatrix<float>::InnerIterator it(A, k); it; ++it)
            {
                ARowIdxHost[idx] = it.row();
                AColIdxHost[idx] = it.col();
                tmpValHost[idx] = it.value();
                idx++;
            }
        }
        hipMemcpy(ARowIdx, ARowIdxHost.data(), sizeof(int) * nnz, hipMemcpyHostToDevice);
        hipMemcpy(AColIdx, AColIdxHost.data(), sizeof(int) * nnz, hipMemcpyHostToDevice);
        hipMemcpy(AVal, tmpValHost.data(), sizeof(float) * nnz, hipMemcpyHostToDevice);

        ls = new CholeskySpLinearSolver<float>(threadsPerBlock, ARowIdx, AColIdx, AVal, ASize, nnz);
    }
    catch (const std::exception& e)
    {
        std::cerr << e.what() << ", " << "Cholesky decomposition(Eigen) failed" << std::endl;
    }


    hipFree(ARowIdx);
    hipFree(AColIdx);
    hipFree(AVal);
}

bool PdSolver::SolverStep(SolverData<float>& solverData, const SolverParams<float>& solverParams)
{
    float dt = solverParams.dt;
    float const dtInv = 1.0f / dt;
    float const dt2 = dt * dt;

    int vertBlocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int vert3Blocks = (solverData.numVerts * 3 + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    int DBCBlocks = 0;
    if (solverData.numDBC > 0)
        DBCBlocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;

    thrust::device_ptr<float> x_prime_ptr(prev_x);
    thrust::device_ptr<float> x_ptr(sn);
    thrust::transform(thrust::device_pointer_cast(solverData.mass), thrust::device_pointer_cast(solverData.mass) + solverData.numVerts,
        thrust::device_pointer_cast(solverData.ExtForce), gravity_force(solverParams.gravity));
    PdUtil::computeSn << < vertBlocks, threadsPerBlock >> > (solverData.numVerts, sn, dt, massDt_2s, solverData.X, solverData.V, solverData.ExtForce);
    hipMemcpy(sn_old, sn, sizeof(float) * (solverData.numVerts * 3), hipMemcpyDeviceToDevice);
    if (solverType == PdSolver::SolverType::Jacobi)
        hipMemcpy(prev_x, sn, sizeof(float) * (solverData.numVerts * 3), hipMemcpyDeviceToDevice);
    else
        hipMemset(prev_x, 0, solverData.numVerts * 3);
    float err{ 1 };
    for (int i = 0; i < solverParams.numIterations && sqrt(err) >= solverParams.tol; i++)
    {
        performanceData[0].second +=
            measureExecutionTime([&]() {
            PdUtil::addM_h2Sn << < vertBlocks, threadsPerBlock >> > (b, sn_old, massDt_2s, solverData.numVerts);
            PdUtil::computeLocal << < tetBlocks, threadsPerBlock >> > (solverData.V0, solverData.mu, b, solverData.DmInv, sn, solverData.Tet, solverData.numTets, solverType == PdSolver::SolverType::Jacobi);
            if (solverData.numDBC > 0)
                PdUtil::computeDBCLocal << < DBCBlocks, threadsPerBlock >> > (solverData.numDBC, solverData.DBC, solverData.X0, positional_weight, b);
                }, perf);
        performanceData[1].second +=
            measureExecutionTime([&]()
                {
                    switch (solverType)
                    {
                    case PdSolver::SolverType::EigenCholesky:
                    {
                        hipMemcpy(bHost, b, sizeof(float) * (solverData.numVerts * 3), hipMemcpyDeviceToHost);
                        Eigen::VectorXf bh = Eigen::Map<Eigen::VectorXf, Eigen::Unaligned>(bHost, solverData.numVerts * 3);
                        Eigen::VectorXf res = cholesky_decomposition_.solve(bh);
                        hipMemcpy(sn, res.data(), sizeof(float) * (solverData.numVerts * 3), hipMemcpyHostToDevice);
                        break;
                    }
                    case PdSolver::SolverType::CuSolverCholesky:
                    {
                        ls->Solve(solverData.numVerts * 3, b, sn);
                        err = computeError(x_ptr, x_prime_ptr, solverData.numVerts * 3);
                        hipMemcpy(prev_x, sn, sizeof(float) * (solverData.numVerts * 3), hipMemcpyDeviceToDevice);
                        break;
                    }
                    case PdSolver::SolverType::Jacobi:
                        PdUtil::getErrorKern << < vertBlocks, threadsPerBlock >> > (solverData.numVerts, next_x, b, massDt_2s, sn, matrix_diag);
                        if (i <= 10)		omega = 1;
                        else if (i == 11)	omega = 2 / (2 - solverParams.rho * solverParams.rho);
                        else			omega = 4 / (4 - solverParams.rho * solverParams.rho * omega);
                        PdUtil::chebyshevKern << < vert3Blocks, threadsPerBlock >> > (solverData.numVerts * 3, next_x, prev_x, sn, omega);
                        break;
                    default:
                        break;
                    }
                }, perf);
    }
    PdUtil::updateVelPos << < vertBlocks, threadsPerBlock >> > (sn, dtInv, solverData.XTilde, solverData.V, solverData.numVerts);
    return true;
}

void PdSolver::Update(SolverData<float>& solverData, const SolverParams<float>& solverParams)
{
    if (!solverReady)
    {
        SolverPrepare(solverData, solverParams);
        solverReady = true;
    }
    SolverStep(solverData, solverParams);
    if (solverParams.handleCollision) {
        performanceData[3].second +=
            measureExecutionTime([&]() {
            solverData.pCollisionDetection->DetectCollision(solverData.numVerts, solverData.numTris, solverData.Tri, solverData.X, solverData.XTilde, solverData.dev_TriFathers, solverData.dev_tIs, solverData.dev_Normals, true);
            int blocks = (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock;
            CCDKernel << <blocks, threadsPerBlock >> > (solverData.X, solverData.XTilde, solverData.V, solverData.dev_tIs, solverData.dev_Normals, solverParams.muT, solverParams.muN, solverData.numVerts, solverParams.dt);
                }, perf);
    }
    else
        hipMemcpy(solverData.X, solverData.XTilde, sizeof(glm::vec3) * solverData.numVerts, hipMemcpyDeviceToDevice);
    performanceData[2].second +=
        measureExecutionTime([&]() {
        solverData.pFixedBodies->HandleCollisions(solverData.XTilde, solverData.V, solverData.numVerts, solverParams.muT, solverParams.muN);
            }, perf);
}

void PdSolver::Reset()
{
    Solver::Reset();
    for (auto& pd : performanceData)
    {
        pd.second = 0.0f;
    }
}

float computeError(thrust::device_ptr<float> sn, thrust::device_ptr<float> sn_old, int size)
{
    return thrust::transform_reduce(
        thrust::counting_iterator<indexType>(0),
        thrust::counting_iterator<indexType>(size),
        [=]__host__ __device__(indexType vertIdx) {
        return (sn_old[vertIdx] - sn[vertIdx]) * (sn_old[vertIdx] - sn[vertIdx]);
    },
        0.0,
        thrust::plus<float>()) / size;
}