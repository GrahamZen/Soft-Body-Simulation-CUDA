#include <linear/choleskyImmed.h>

template<typename T>
inline CholeskySpImmedSolver<T>::CholeskySpImmedSolver(int N)
{
    hipsparseCreate(&handle);
    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipMalloc((void**)&d_rowPtrA, sizeof(int) * (N + 1));
}

template<typename T>
CholeskySpImmedSolver<T>::~CholeskySpImmedSolver()
{
    hipsparseDestroyMatDescr(descrA);
    hipsolverSpDestroy(cusolverHandle);
    hipFree(d_rowPtrA);
}

template class CholeskySpImmedSolver<float>;
template class CholeskySpImmedSolver<double>;

template<>
void CholeskySpImmedSolver<double>::Solve(int N, double* d_b, double* d_x, double* A, int nz, int* rowIdx, int* colIdx, double* d_guess)
{
    int singularity;
    sort_coo(N, nz, A, rowIdx, colIdx, d_A, d_rowIdx, d_colIdx);
    CHECK_CUSPARSE(hipsparseXcoo2csr(handle, d_rowIdx, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSOLVER(hipsolverSpDcsrlsvchol(cusolverHandle, N, nz, descrA, d_A, d_rowPtrA, d_colIdx, d_b, 0, 0, d_x, &singularity));
}

template<>
void CholeskySpImmedSolver<float>::Solve(int N, float* d_b, float* d_x, float* A, int nz, int* rowIdx, int* colIdx, float* d_guess)
{
    int singularity;
    sort_coo(N, nz, A, rowIdx, colIdx, d_A, d_rowIdx, d_colIdx);
    CHECK_CUSPARSE(hipsparseXcoo2csr(handle, d_rowIdx, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSOLVER(hipsolverSpScsrlsvchol(cusolverHandle, N, nz, descrA, d_A, d_rowPtrA, d_colIdx, d_b, 0, 0, d_x, &singularity));
}
