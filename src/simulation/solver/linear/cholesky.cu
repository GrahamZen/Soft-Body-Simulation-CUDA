#include "hip/hip_runtime.h"
#include <simulation/solver/linear/cholesky.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

__global__ void FillMatrixA(int* AIdx, float* tmpVal, float* d_A, int n, int ASize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    int row = AIdx[idx] / ASize;
    int col = AIdx[idx] % ASize;
    atomicAdd(&d_A[row * ASize + col], tmpVal[idx]);
}

__global__ void initAMatrix(int* idx, int* row, int* col, int rowLen, int totalNumber)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < totalNumber)
    {
        row[index] = idx[index] / rowLen;
        col[index] = idx[index] % rowLen;
    }
}

CholeskyDnLinearSolver::~CholeskyDnLinearSolver()
{
    hipFree(d_info);
    hipFree(d_predecomposedA);
    hipFree(d_work);
}

CholeskyDnLinearSolver::CholeskyDnLinearSolver(int threadsPerBlock, int* AIdx, float* tmpVal, int ASize, int len) {
    hipMalloc(&d_predecomposedA, sizeof(float) * ASize * ASize);
    FillMatrixA << < (len + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (AIdx, tmpVal, d_predecomposedA, len, ASize);
    hipsolverDnCreate(&cusolverHandle);
    hipsolverDnCreateParams(&params);

    // Matrix dimension and leading dimension
    int n = ASize;
    int lda = n;  // Leading dimension of A
    int info = 0;
    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void* h_work = nullptr;              /* host workspace */
    // Allocate memory for dense matrix A
    hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int));

    // Copy your matrix data from host to device
    // Assuming h_A is the host matrix with size n x n

    hipsolverDnXpotrf_bufferSize(
        cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, hipDataType::HIP_R_32F, d_predecomposedA, lda,
        hipDataType::HIP_R_32F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost);

    hipMalloc(reinterpret_cast<void**>(&d_work), workspaceInBytesOnDevice);
    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void*>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    hipsolverDnXpotrf(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, hipDataType::HIP_R_32F,
        d_predecomposedA, lda, hipDataType::HIP_R_32F, d_work, workspaceInBytesOnDevice,
        h_work, workspaceInBytesOnHost, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    std::printf("after Xpotrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    free(h_work);
}

CholeskySpLinearSolver::~CholeskySpLinearSolver()
{
    cusolverSpDestroyCsrcholInfo(d_info);
    hipsparseDestroyMatDescr(descrA);
    hipsolverSpDestroy(cusolverHandle);
    hipFree(buffer_gpu);
}

CholeskySpLinearSolver::CholeskySpLinearSolver(int threadsPerBlock, int* AIdx, float* tmpVal, int ASize, int len) {
    int* newIdx;
    float* newVal;

    hipMalloc((void**)&newIdx, sizeof(int) * len);
    hipMalloc((void**)&newVal, sizeof(float) * len);

    thrust::sort_by_key(thrust::device, AIdx, AIdx + len, tmpVal);
    thrust::pair<int*, float*> newEnd = thrust::reduce_by_key(thrust::device, AIdx, AIdx + len, tmpVal, newIdx, newVal);

    int* ARow, * ACol;
    float* AVal;

    int nnz = newEnd.first - newIdx;

    hipMalloc((void**)&ARow, sizeof(int) * nnz);
    hipMemset(ARow, 0, sizeof(int) * nnz);

    hipMalloc((void**)&ACol, sizeof(int) * nnz);
    hipMemset(ACol, 0, sizeof(int) * nnz);

    hipMalloc((void**)&AVal, sizeof(float) * nnz);
    hipMemcpy(AVal, newVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    int blocks = (nnz + threadsPerBlock - 1) / threadsPerBlock;

    initAMatrix << < blocks, threadsPerBlock >> > (newIdx, ARow, ACol, ASize, nnz);

    // transform ARow into csr format
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseXcoo2csr(handle, ARow, nnz, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    size_t cholSize = 0;
    size_t internalSize = 0;
    cusolverSpCreateCsrcholInfo(&d_info);
    cusolverSpXcsrcholAnalysis(cusolverHandle, ASize, nnz, descrA, ARow, ACol, d_info);
    cusolverSpScsrcholBufferInfo(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, &internalSize, &cholSize);
    hipMalloc(&buffer_gpu, sizeof(char) * cholSize);
    cusolverSpScsrcholFactor(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, buffer_gpu);

    hipFree(newIdx);
    hipFree(newVal);
    hipFree(ARow);
    hipFree(ACol);
    hipFree(AVal);
}
void CholeskyDnLinearSolver::Solve(int N, float* d_b, float* d_x, float* d_A, int nz, int* d_rowIdx, int* d_colIdx, float* d_guess) {
    hipsolverDnXpotrs(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, N, 1, /* nrhs */
        hipDataType::HIP_R_32F, d_predecomposedA, N,
        hipDataType::HIP_R_32F, d_b, N, d_info);
    hipMemcpy(d_x, d_b, sizeof(float) * (N), hipMemcpyDeviceToDevice);
}

void CholeskySpLinearSolver::Solve(int N, float* d_b, float* d_x, float* d_A, int nz, int* d_rowIdx, int* d_colIdx, float* d_guess)
{
    cusolverSpScsrcholSolve(cusolverHandle, N, d_b, d_x, d_info, buffer_gpu);
}
