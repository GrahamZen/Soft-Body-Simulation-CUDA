#include "hip/hip_runtime.h"
#include <simulation/solver/linear/cholesky.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

__global__ void FillMatrixA(int* AIdx, float* tmpVal, float* d_A, int n, int ASize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    int row = AIdx[idx] / ASize;
    int col = AIdx[idx] % ASize;
    atomicAdd(&d_A[row * ASize + col], tmpVal[idx]);
}

__global__ void initAMatrix(int* idx, int* row, int* col, int rowLen, int totalNumber)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < totalNumber)
    {
        row[index] = idx[index] / rowLen;
        col[index] = idx[index] % rowLen;
    }
}

CholeskyDnlinearSolver::~CholeskyDnlinearSolver()
{
    hipFree(d_info);
    hipFree(d_A);
    hipFree(d_work);
}

CholeskyDnlinearSolver::CholeskyDnlinearSolver(int threadsPerBlock, int* AIdx, float* tmpVal, int ASize, int len) {
    hipMalloc(&d_A, sizeof(float) * ASize * ASize);
    FillMatrixA << < (len + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (AIdx, tmpVal, d_A, len, ASize);
    hipsolverDnCreate(&cusolverHandle);
    hipsolverDnCreateParams(&params);

    // Matrix dimension and leading dimension
    int n = ASize;
    int lda = n;  // Leading dimension of A
    int info = 0;
    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void* h_work = nullptr;              /* host workspace */
    // Allocate memory for dense matrix A
    hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int));

    // Copy your matrix data from host to device
    // Assuming h_A is the host matrix with size n x n

    hipsolverDnXpotrf_bufferSize(
        cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, hipDataType::HIP_R_32F, d_A, lda,
        hipDataType::HIP_R_32F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost);

    hipMalloc(reinterpret_cast<void**>(&d_work), workspaceInBytesOnDevice);
    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void*>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    hipsolverDnXpotrf(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, hipDataType::HIP_R_32F,
        d_A, lda, hipDataType::HIP_R_32F, d_work, workspaceInBytesOnDevice,
        h_work, workspaceInBytesOnHost, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    std::printf("after Xpotrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    free(h_work);
}

CholeskySplinearSolver::~CholeskySplinearSolver()
{
    cusolverSpDestroyCsrcholInfo(d_info);
    hipsparseDestroyMatDescr(descrA);
    hipsolverSpDestroy(cusolverHandle);
    hipFree(buffer_gpu);
}

CholeskySplinearSolver::CholeskySplinearSolver(int threadsPerBlock, int* AIdx, float* tmpVal, int ASize, int len) {
    int* newIdx;
    float* newVal;

    hipMalloc((void**)&newIdx, sizeof(int) * len);
    hipMalloc((void**)&newVal, sizeof(float) * len);

    thrust::sort_by_key(thrust::device, AIdx, AIdx + len, tmpVal);


    thrust::pair<int*, float*> newEnd = thrust::reduce_by_key(thrust::device, AIdx, AIdx + len, tmpVal, newIdx, newVal);

    int* ARow;
    int* ACol;
    float* AVal;

    nnz = newEnd.first - newIdx;
    std::cout << nnz << std::endl;

    hipMalloc((void**)&ARow, sizeof(int) * nnz);
    hipMemset(ARow, 0, sizeof(int) * nnz);

    hipMalloc((void**)&ACol, sizeof(int) * nnz);
    hipMemset(ACol, 0, sizeof(int) * nnz);

    hipMalloc((void**)&AVal, sizeof(float) * nnz);
    hipMemcpy(AVal, newVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    int* ARowTmp;
    hipMalloc((void**)&ARowTmp, sizeof(int) * nnz);
    hipMemset(ARowTmp, 0, sizeof(int) * nnz);

    int blocks = (nnz + threadsPerBlock - 1) / threadsPerBlock;

    initAMatrix << < blocks, threadsPerBlock >> > (newIdx, ARowTmp, ACol, ASize, nnz);

    // transform ARow into csr format
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseXcoo2csr(handle, ARowTmp, nnz, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    size_t cholSize = 0;
    size_t internalSize = 0;
    cusolverSpCreateCsrcholInfo(&d_info);
    ApproximateMinimumDegree(ASize, ARow, ACol, AVal);
    cusolverSpXcsrcholAnalysis(cusolverHandle, ASize, nnz, descrA, ARow, ACol, d_info);
    cusolverSpScsrcholBufferInfo(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, &internalSize, &cholSize);
    hipMalloc(&buffer_gpu, sizeof(char) * cholSize);
    cusolverSpScsrcholFactor(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, buffer_gpu);

    hipFree(newIdx);
    hipFree(newVal);
    hipFree(ARowTmp);
    hipFree(ARow);
    hipFree(ACol);
    hipFree(AVal);
}

void CholeskyDnlinearSolver::Solve(float* d_b, int bSize, float* d_x) {
    hipsolverDnXpotrs(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, bSize, 1, /* nrhs */
        hipDataType::HIP_R_32F, d_A, bSize,
        hipDataType::HIP_R_32F, d_b, bSize, d_info);
    hipMemcpy(d_x, d_b, sizeof(float) * (bSize), hipMemcpyDeviceToDevice);
}

void CholeskySplinearSolver::Solve(float* d_b, int bSize, float* d_x)
{
    cusolverSpScsrcholSolve(cusolverHandle, bSize, d_b, d_x, d_info, buffer_gpu);
}

void CholeskySplinearSolver::ApproximateMinimumDegree(int ASize, int* dev_ARow, int* dev_ACol, float* dev_AVal)
{
    size_t size_perm = 0;
    void* buffer_cpu = nullptr;
    int* h_csrRowPtrA = new int[ASize + 1];
    int* h_csrColIndA = new int[nnz];
    float* h_csrValA = new float[nnz];

    hipMemcpy(h_csrRowPtrA, dev_ARow, sizeof(int) * (ASize + 1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColIndA, dev_ACol, sizeof(int) * nnz, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrValA, dev_AVal, sizeof(float) * nnz, hipMemcpyDeviceToHost);

    int colsA = ASize, rowsA = ASize, nnzA = nnz;

    int*  h_Q = (int*)malloc(sizeof(int) * colsA);
    int*  h_csrRowPtrB = (int*)malloc(sizeof(int) * (rowsA + 1));
    int*  h_csrColIndB = (int*)malloc(sizeof(int) * nnzA);
    double*  h_csrValB = (double*)malloc(sizeof(double) * nnzA);
    int*  h_mapBfromA = (int*)malloc(sizeof(int) * nnzA);

    assert(NULL != h_Q);
    assert(NULL != h_csrRowPtrB);
    assert(NULL != h_csrColIndB);
    assert(NULL != h_csrValB);
    assert(NULL != h_mapBfromA);

    (cusolverSpXcsrsymamdHost(
        cusolverHandle, rowsA, nnzA,
        descrA, h_csrRowPtrA, h_csrColIndA,
        h_Q));

    // B = Q*A*Q^T
    memcpy(h_csrRowPtrB, h_csrRowPtrA, sizeof(int) * (rowsA + 1));
    memcpy(h_csrColIndB, h_csrColIndA, sizeof(int) * nnzA);

    (cusolverSpXcsrperm_bufferSizeHost(
        cusolverHandle, rowsA, colsA, nnzA,
        descrA, h_csrRowPtrB, h_csrColIndB,
        h_Q, h_Q,
        &size_perm));

    if (buffer_cpu)
    {
        free(buffer_cpu);
    }
    buffer_cpu = (void*)malloc(sizeof(char) * size_perm);
    assert(NULL != buffer_cpu);

    // h_mapBfromA = Identity
    for (int j = 0; j < nnzA; j++)
    {
        h_mapBfromA[j] = j;
    }
    (cusolverSpXcsrpermHost(
        cusolverHandle, rowsA, colsA, nnzA,
        descrA, h_csrRowPtrB, h_csrColIndB,
        h_Q, h_Q,
        h_mapBfromA,
        buffer_cpu));

    // B = A( mapBfromA )
    for (int j = 0; j < nnzA; j++)
    {
        h_csrValB[j] = h_csrValA[h_mapBfromA[j]];
    }

    // A := B
    memcpy(h_csrRowPtrA, h_csrRowPtrB, sizeof(int) * (rowsA + 1));
    memcpy(h_csrColIndA, h_csrColIndB, sizeof(int) * nnzA);
    memcpy(h_csrValA, h_csrValB, sizeof(double) * nnzA);
    
    if (buffer_cpu)
    {
        free(buffer_cpu);
    }
}
