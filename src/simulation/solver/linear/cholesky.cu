#include "hip/hip_runtime.h"
#include <simulation/solver/linear/cholesky.h>
#include <thrust/execution_policy.h>
#include <linear/linearUtils.cuh>

template<typename T>
__global__ void FillMatrixA(int* AIdx, T* tmpVal, T* d_A, int n, int ASize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    int row = AIdx[idx] / ASize;
    int col = AIdx[idx] % ASize;
    atomicAdd(&d_A[row * ASize + col], tmpVal[idx]);
}

__global__ void initAMatrix(int* idx, int* row, int* col, int rowLen, int totalNumber)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < totalNumber)
    {
        row[index] = idx[index] / rowLen;
        col[index] = idx[index] % rowLen;
    }
}

template<typename T>
CholeskyDnLinearSolver<T>::~CholeskyDnLinearSolver()
{
    hipFree(d_info);
    hipFree(d_predecomposedA);
    hipFree(d_work);
}

template<typename T>
CholeskyDnLinearSolver<T>::CholeskyDnLinearSolver(int threadsPerBlock, int* AIdx, T* tmpVal, int ASize, int len) {
    hipMalloc(&d_predecomposedA, sizeof(T) * ASize * ASize);
    FillMatrixA << < (len + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (AIdx, tmpVal, d_predecomposedA, len, ASize);
    hipsolverDnCreate(&cusolverHandle);
    hipsolverDnCreateParams(&params);

    // Matrix dimension and leading dimension
    int n = ASize;
    int lda = n;  // Leading dimension of A
    int info = 0;
    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void* h_work = nullptr;              /* host workspace */
    // Allocate memory for dense matrix A
    hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int));

    // Copy your matrix data from host to device
    // Assuming h_A is the host matrix with size n x n

    hipsolverDnXpotrf_bufferSize(
        cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, dataType, d_predecomposedA, lda,
        dataType, &workspaceInBytesOnDevice, &workspaceInBytesOnHost);

    hipMalloc(reinterpret_cast<void**>(&d_work), workspaceInBytesOnDevice);
    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void*>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    hipsolverDnXpotrf(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, dataType,
        d_predecomposedA, lda, dataType, d_work, workspaceInBytesOnDevice,
        h_work, workspaceInBytesOnHost, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    std::printf("after Xpotrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    free(h_work);
}

template<typename T>
CholeskySpLinearSolver<T>::~CholeskySpLinearSolver()
{
    cusolverSpDestroyCsrcholInfo(d_info);
    hipsparseDestroyMatDescr(descrA);
    hipsolverSpDestroy(cusolverHandle);
    hipFree(buffer_gpu);
    hipFree(dev_x_permuted);
    hipFree(dev_b_permuted);
}

template<typename T>
void CholeskySpLinearSolver<T>::ComputeAMD(hipsolverSpHandle_t handle, int rowsA, int nnzA, int* dev_csrRowPtrA, int* dev_csrColIndA, T* dev_csrValA) {
    std::vector<int> h_Q(rowsA);
    std::vector<int> h_csrRowPtrB(rowsA + 1);
    std::vector<int> h_csrColIndB(nnzA);
    std::vector<T> h_csrValB(nnzA);
    std::vector<int> h_mapBfromA(nnzA);

    std::vector<int> h_csrRowPtrA(rowsA + 1);
    std::vector<int> h_csrColIndA(nnzA);
    std::vector<T> h_csrValA(nnzA);

    hipMemcpy(h_csrRowPtrA.data(), dev_csrRowPtrA, sizeof(int) * (rowsA + 1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColIndA.data(), dev_csrColIndA, sizeof(int) * nnzA, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrValA.data(), dev_csrValA, sizeof(T) * nnzA, hipMemcpyDeviceToHost);

    cusolverSpXcsrsymamdHost(
        handle, rowsA, nnzA,
        descrA, h_csrRowPtrA.data(), h_csrColIndA.data(),
        h_Q.data());

    // B = Q*A*Q^T
    memcpy(h_csrRowPtrB.data(), h_csrRowPtrA.data(), sizeof(int) * (rowsA + 1));
    memcpy(h_csrColIndB.data(), h_csrColIndA.data(), sizeof(int) * nnzA);

    size_t size_perm;
    cusolverSpXcsrperm_bufferSizeHost(
        handle, rowsA, rowsA, nnzA,
        descrA, h_csrRowPtrB.data(), h_csrColIndB.data(),
        h_Q.data(), h_Q.data(),
        &size_perm);
    void* buffer_cpu = nullptr;
    buffer_cpu = (void*)malloc(sizeof(char) * size_perm);
    assert(NULL != buffer_cpu);

    // h_mapBfromA.data() = Identity
    for (int j = 0; j < nnzA; j++)
    {
        h_mapBfromA.data()[j] = j;
    }
    cusolverSpXcsrpermHost(
        handle, rowsA, rowsA, nnzA,
        descrA, h_csrRowPtrB.data(), h_csrColIndB.data(),
        h_Q.data(), h_Q.data(),
        h_mapBfromA.data(),
        buffer_cpu);

    // B = A( mapBfromA )
    for (int j = 0; j < nnzA; j++)
    {
        h_csrValB[j] = h_csrValA[h_mapBfromA[j]];
    }

    hipMemcpy(dev_csrRowPtrA, h_csrRowPtrB.data(), sizeof(int) * (rowsA + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_csrColIndA, h_csrColIndB.data(), sizeof(int) * nnzA, hipMemcpyHostToDevice);
    hipMemcpy(dev_csrValA, h_csrValB.data(), sizeof(T) * nnzA, hipMemcpyHostToDevice);
    hipMalloc(&d_p, sizeof(int) * rowsA);
    hipMemcpy(d_p, h_Q.data(), sizeof(int) * rowsA, hipMemcpyHostToDevice);
    free(buffer_cpu);
}

template<>
CholeskySpLinearSolver<double>::CholeskySpLinearSolver(int threadsPerBlock, int* ARow, int* ACol, double* AVal, int ASize, int len) {
    sort_coo(ASize, len, AVal, ARow, ACol);
    int nnz = len;
    // transform ARow into csr format
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseXcoo2csr(handle, ARow, nnz, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    ComputeAMD(cusolverHandle, ASize, nnz, ARow, ACol, AVal);
    size_t cholSize = 0;
    size_t internalSize = 0;
    cusolverSpCreateCsrcholInfo(&d_info);
    cusolverSpXcsrcholAnalysis(cusolverHandle, ASize, nnz, descrA, ARow, ACol, d_info);
    cusolverSpDcsrcholBufferInfo(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, &internalSize, &cholSize);
    hipMalloc((void**)&buffer_gpu, sizeof(char) * cholSize);
    hipMalloc((void**)&dev_b_permuted, sizeof(double) * ASize);
    hipMalloc((void**)&dev_x_permuted, sizeof(double) * ASize);
    cusolverSpDcsrcholFactor(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, buffer_gpu);
}

template<> CholeskySpLinearSolver<float>::CholeskySpLinearSolver(int threadsPerBlock, int* ARow, int* ACol, float* AVal, int ASize, int len) {
    sort_coo(ASize, len, AVal, ARow, ACol);
    int nnz = len;
    // transform ARow into csr format
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseXcoo2csr(handle, ARow, nnz, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    ComputeAMD(cusolverHandle, ASize, nnz, ARow, ACol, AVal);
    size_t cholSize = 0;
    size_t internalSize = 0;
    cusolverSpCreateCsrcholInfo(&d_info);
    cusolverSpXcsrcholAnalysis(cusolverHandle, ASize, nnz, descrA, ARow, ACol, d_info);
    cusolverSpScsrcholBufferInfo(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, &internalSize, &cholSize);
    hipMalloc((void**)&buffer_gpu, sizeof(char) * cholSize);
    hipMalloc((void**)&dev_b_permuted, sizeof(float) * ASize);
    hipMalloc((void**)&dev_x_permuted, sizeof(float) * ASize);
    cusolverSpScsrcholFactor(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, buffer_gpu);
}

template<typename T>
void CholeskyDnLinearSolver<T>::Solve(int N, T* d_b, T* d_x, T* d_A, int nz, int* d_rowIdx, int* d_colIdx, T* d_guess) {
    hipsolverDnXpotrs(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, N, 1, /* nrhs */
        dataType, d_predecomposedA, N,
        dataType, d_b, N, d_info);
    hipMemcpy(d_x, d_b, sizeof(T) * (N), hipMemcpyDeviceToDevice);
}

template<typename T>
__global__ void permuteVector(const T* b, T* b_permuted, const int* p, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        b_permuted[idx] = b[p[idx]];
    }
}

template<typename T>
__global__ void permuteVectorInv(const T* x_permuted, T* x, const int* p, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        x[p[idx]] = x_permuted[idx];
    }
}

template<> void CholeskySpLinearSolver<float>::Solve(int N, float* d_b, float* d_x, float* d_A, int nz, int* d_rowIdx, int* d_colIdx, float* d_guess)
{
    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    permuteVector << <blocks, threadsPerBlock >> > (d_b, dev_b_permuted, d_p, N);
    cusolverSpScsrcholSolve(cusolverHandle, N, dev_b_permuted, dev_x_permuted, d_info, buffer_gpu);
    permuteVectorInv << <blocks, threadsPerBlock >> > (dev_x_permuted, d_x, d_p, N);
}

template<> void CholeskySpLinearSolver<double>::Solve(int N, double* d_b, double* d_x, double* d_A, int nz, int* d_rowIdx, int* d_colIdx, double* d_guess)
{
    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    permuteVector << <blocks, threadsPerBlock >> > (d_b, dev_b_permuted, d_p, N);
    cusolverSpDcsrcholSolve(cusolverHandle, N, dev_b_permuted, dev_x_permuted, d_info, buffer_gpu);
    permuteVectorInv << <blocks, threadsPerBlock >> > (dev_x_permuted, d_x, d_p, N);
}

template CholeskySpLinearSolver<double>;
template CholeskySpLinearSolver<float>;