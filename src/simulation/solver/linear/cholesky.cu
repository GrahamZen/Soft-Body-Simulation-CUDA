#include "hip/hip_runtime.h"
#include <simulation/solver/linear/cholesky.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

__global__ void FillMatrixA(int* AIdx, float* tmpVal, float* d_A, int n, int ASize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    int row = AIdx[idx] / ASize;
    int col = AIdx[idx] % ASize;
    atomicAdd(&d_A[row * ASize + col], tmpVal[idx]);
}

__global__ void initAMatrix(int* idx, int* row, int* col, int rowLen, int totalNumber)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < totalNumber)
    {
        row[index] = idx[index] / rowLen;
        col[index] = idx[index] % rowLen;
    }
}

CholeskyDnlinearSolver::~CholeskyDnlinearSolver()
{
    hipFree(d_info);
    hipFree(d_A);
    hipFree(d_work);
}

CholeskyDnlinearSolver::CholeskyDnlinearSolver(int threadsPerBlock, int* AIdx, float* tmpVal, int ASize, int len) {
    hipMalloc(&d_A, sizeof(float) * ASize * ASize);
    FillMatrixA << < (len + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (AIdx, tmpVal, d_A, len, ASize);
    hipsolverDnCreate(&cusolverHandle);
    hipsolverDnCreateParams(&params);

    // Matrix dimension and leading dimension
    int n = ASize;
    int lda = n;  // Leading dimension of A
    int info = 0;
    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void* h_work = nullptr;              /* host workspace */
    // Allocate memory for dense matrix A
    hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int));

    // Copy your matrix data from host to device
    // Assuming h_A is the host matrix with size n x n

    hipsolverDnXpotrf_bufferSize(
        cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, hipDataType::HIP_R_32F, d_A, lda,
        hipDataType::HIP_R_32F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost);

    hipMalloc(reinterpret_cast<void**>(&d_work), workspaceInBytesOnDevice);
    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void*>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    hipsolverDnXpotrf(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, hipDataType::HIP_R_32F,
        d_A, lda, hipDataType::HIP_R_32F, d_work, workspaceInBytesOnDevice,
        h_work, workspaceInBytesOnHost, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    std::printf("after Xpotrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    free(h_work);
}

CholeskySplinearSolver::~CholeskySplinearSolver()
{
    cusolverSpDestroyCsrcholInfo(d_info);
    hipsparseDestroyMatDescr(descrA);
    hipsolverSpDestroy(cusolverHandle);
    hipFree(buffer_gpu);
}

CholeskySplinearSolver::CholeskySplinearSolver(int threadsPerBlock, int* AIdx, float* tmpVal, int ASize, int len) {
    int* newIdx;
    float* newVal;

    hipMalloc((void**)&newIdx, sizeof(int) * len);
    hipMalloc((void**)&newVal, sizeof(float) * len);

    thrust::sort_by_key(thrust::device, AIdx, AIdx + len, tmpVal);


    thrust::pair<int*, float*> newEnd = thrust::reduce_by_key(thrust::device, AIdx, AIdx + len, tmpVal, newIdx, newVal);

    int* ARow;
    int* ACol;
    float* AVal;

    nnz = newEnd.first - newIdx;
    std::cout << nnz << std::endl;

    hipMalloc((void**)&ARow, sizeof(int) * nnz);
    hipMemset(ARow, 0, sizeof(int) * nnz);

    hipMalloc((void**)&ACol, sizeof(int) * nnz);
    hipMemset(ACol, 0, sizeof(int) * nnz);

    hipMalloc((void**)&AVal, sizeof(float) * nnz);
    hipMemcpy(AVal, newVal, sizeof(float) * nnz, hipMemcpyDeviceToDevice);

    int* ARowTmp;
    hipMalloc((void**)&ARowTmp, sizeof(int) * nnz);
    hipMemset(ARowTmp, 0, sizeof(int) * nnz);

    int blocks = (nnz + threadsPerBlock - 1) / threadsPerBlock;

    initAMatrix << < blocks, threadsPerBlock >> > (newIdx, ARowTmp, ACol, ASize, nnz);

    // transform ARow into csr format
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseXcoo2csr(handle, ARowTmp, nnz, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t descrA;
    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    size_t cholSize = 0;
    size_t internalSize = 0;
    cusolverSpCreateCsrcholInfo(&d_info);
    cusolverSpXcsrcholAnalysis(cusolverHandle, ASize, nnz, descrA, ARow, ACol, d_info);
    cusolverSpScsrcholBufferInfo(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, &internalSize, &cholSize);
    hipMalloc(&buffer_gpu, sizeof(char) * cholSize);
    cusolverSpScsrcholFactor(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, buffer_gpu);

    hipFree(newIdx);
    hipFree(newVal);
    hipFree(ARowTmp);
    hipFree(ARow);
    hipFree(ACol);
    hipFree(AVal);
}

void CholeskyDnlinearSolver::Solve(float* d_b, int bSize, float* d_x) {
    hipsolverDnXpotrs(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, bSize, 1, /* nrhs */
        hipDataType::HIP_R_32F, d_A, bSize,
        hipDataType::HIP_R_32F, d_b, bSize, d_info);
    hipMemcpy(d_x, d_b, sizeof(float) * (bSize), hipMemcpyDeviceToDevice);
}

void CholeskySplinearSolver::Solve(float* d_b, int bSize, float* d_x)
{
    cusolverSpScsrcholSolve(cusolverHandle, bSize, d_b, d_x, d_info, buffer_gpu);
}
