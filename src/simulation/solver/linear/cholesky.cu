#include "hip/hip_runtime.h"
#include <simulation/solver/linear/cholesky.h>
#include <thrust/execution_policy.h>
#include <linear/linearUtils.cuh>

__global__ void FillMatrixA(int* AIdx, float* tmpVal, float* d_A, int n, int ASize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    int row = AIdx[idx] / ASize;
    int col = AIdx[idx] % ASize;
    atomicAdd(&d_A[row * ASize + col], tmpVal[idx]);
}

__global__ void initAMatrix(int* idx, int* row, int* col, int rowLen, int totalNumber)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < totalNumber)
    {
        row[index] = idx[index] / rowLen;
        col[index] = idx[index] % rowLen;
    }
}

CholeskyDnLinearSolver::~CholeskyDnLinearSolver()
{
    hipFree(d_info);
    hipFree(d_predecomposedA);
    hipFree(d_work);
}

CholeskyDnLinearSolver::CholeskyDnLinearSolver(int threadsPerBlock, int* AIdx, float* tmpVal, int ASize, int len) {
    hipMalloc(&d_predecomposedA, sizeof(float) * ASize * ASize);
    FillMatrixA << < (len + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (AIdx, tmpVal, d_predecomposedA, len, ASize);
    hipsolverDnCreate(&cusolverHandle);
    hipsolverDnCreateParams(&params);

    // Matrix dimension and leading dimension
    int n = ASize;
    int lda = n;  // Leading dimension of A
    int info = 0;
    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void* h_work = nullptr;              /* host workspace */
    // Allocate memory for dense matrix A
    hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int));

    // Copy your matrix data from host to device
    // Assuming h_A is the host matrix with size n x n

    hipsolverDnXpotrf_bufferSize(
        cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, hipDataType::HIP_R_32F, d_predecomposedA, lda,
        hipDataType::HIP_R_32F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost);

    hipMalloc(reinterpret_cast<void**>(&d_work), workspaceInBytesOnDevice);
    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void*>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    hipsolverDnXpotrf(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, n, hipDataType::HIP_R_32F,
        d_predecomposedA, lda, hipDataType::HIP_R_32F, d_work, workspaceInBytesOnDevice,
        h_work, workspaceInBytesOnHost, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    std::printf("after Xpotrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    free(h_work);
}

CholeskySpLinearSolver::~CholeskySpLinearSolver()
{
    cusolverSpDestroyCsrcholInfo(d_info);
    hipsparseDestroyMatDescr(descrA);
    hipsolverSpDestroy(cusolverHandle);
    hipFree(buffer_gpu);
    hipFree(dev_x_permuted);
    hipFree(dev_b_permuted);
}

void CholeskySpLinearSolver::ComputeAMD(hipsolverSpHandle_t handle, int rowsA, int nnzA, int* dev_csrRowPtrA, int* dev_csrColIndA, float* dev_csrValA) {
    std::vector<int> h_Q(rowsA);
    std::vector<int> h_csrRowPtrB(rowsA + 1);
    std::vector<int> h_csrColIndB(nnzA);
    std::vector<float> h_csrValB(nnzA);
    std::vector<int> h_mapBfromA(nnzA);

    std::vector<int> h_csrRowPtrA(rowsA + 1);
    std::vector<int> h_csrColIndA(nnzA);
    std::vector<float> h_csrValA(nnzA);

    hipMemcpy(h_csrRowPtrA.data(), dev_csrRowPtrA, sizeof(int) * (rowsA + 1), hipMemcpyDeviceToHost);
    hipMemcpy(h_csrColIndA.data(), dev_csrColIndA, sizeof(int) * nnzA, hipMemcpyDeviceToHost);
    hipMemcpy(h_csrValA.data(), dev_csrValA, sizeof(float) * nnzA, hipMemcpyDeviceToHost);

    cusolverSpXcsrsymamdHost(
        handle, rowsA, nnzA,
        descrA, h_csrRowPtrA.data(), h_csrColIndA.data(),
        h_Q.data());

    // B = Q*A*Q^T
    memcpy(h_csrRowPtrB.data(), h_csrRowPtrA.data(), sizeof(int) * (rowsA + 1));
    memcpy(h_csrColIndB.data(), h_csrColIndA.data(), sizeof(int) * nnzA);

    size_t size_perm;
    cusolverSpXcsrperm_bufferSizeHost(
        handle, rowsA, rowsA, nnzA,
        descrA, h_csrRowPtrB.data(), h_csrColIndB.data(),
        h_Q.data(), h_Q.data(),
        &size_perm);
    void* buffer_cpu = nullptr;
    buffer_cpu = (void*)malloc(sizeof(char) * size_perm);
    assert(NULL != buffer_cpu);

    // h_mapBfromA.data() = Identity
    for (int j = 0; j < nnzA; j++)
    {
        h_mapBfromA.data()[j] = j;
    }
    cusolverSpXcsrpermHost(
        handle, rowsA, rowsA, nnzA,
        descrA, h_csrRowPtrB.data(), h_csrColIndB.data(),
        h_Q.data(), h_Q.data(),
        h_mapBfromA.data(),
        buffer_cpu);

    // B = A( mapBfromA )
    for (int j = 0; j < nnzA; j++)
    {
        h_csrValB[j] = h_csrValA[h_mapBfromA[j]];
    }

    hipMemcpy(dev_csrRowPtrA, h_csrRowPtrB.data(), sizeof(int) * (rowsA + 1), hipMemcpyHostToDevice);
    hipMemcpy(dev_csrColIndA, h_csrColIndB.data(), sizeof(int) * nnzA, hipMemcpyHostToDevice);
    hipMemcpy(dev_csrValA, h_csrValB.data(), sizeof(float) * nnzA, hipMemcpyHostToDevice);
    hipMalloc(&d_p, sizeof(int) * rowsA);
    hipMemcpy(d_p, h_Q.data(), sizeof(int) * rowsA, hipMemcpyHostToDevice);
    free(buffer_cpu);
}

CholeskySpLinearSolver::CholeskySpLinearSolver(int threadsPerBlock,int* ARow, int* ACol, float* AVal, int ASize, int len) {
    sort_coo(ASize, len, AVal, ARow, ACol);
    int nnz = len;
    // transform ARow into csr format
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseXcoo2csr(handle, ARow, nnz, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    ComputeAMD(cusolverHandle, ASize, nnz, ARow, ACol, AVal);
    size_t cholSize = 0;
    size_t internalSize = 0;
    cusolverSpCreateCsrcholInfo(&d_info);
    cusolverSpXcsrcholAnalysis(cusolverHandle, ASize, nnz, descrA, ARow, ACol, d_info);
    cusolverSpScsrcholBufferInfo(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, &internalSize, &cholSize);
    hipMalloc((void**)&buffer_gpu, sizeof(char) * cholSize);
    hipMalloc((void**)&dev_b_permuted, sizeof(float) * ASize);
    hipMalloc((void**)&dev_x_permuted, sizeof(float) * ASize);
    cusolverSpScsrcholFactor(cusolverHandle, ASize, nnz, descrA, AVal, ARow, ACol, d_info, buffer_gpu);
}

void CholeskyDnLinearSolver::Solve(int N, float* d_b, float* d_x, float* d_A, int nz, int* d_rowIdx, int* d_colIdx, float* d_guess) {
    hipsolverDnXpotrs(cusolverHandle, params, HIPBLAS_FILL_MODE_LOWER, N, 1, /* nrhs */
        hipDataType::HIP_R_32F, d_predecomposedA, N,
        hipDataType::HIP_R_32F, d_b, N, d_info);
    hipMemcpy(d_x, d_b, sizeof(float) * (N), hipMemcpyDeviceToDevice);
}

__global__ void permuteVector(const float* b, float* b_permuted, const int* p, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        b_permuted[idx] = b[p[idx]];
    }
}

__global__ void permuteVectorInv(const float* x_permuted, float* x, const int* p, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        x[p[idx]] = x_permuted[idx];
    }
}

void CholeskySpLinearSolver::Solve(int N, float* d_b, float* d_x, float* d_A, int nz, int* d_rowIdx, int* d_colIdx, float* d_guess)
{
    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    permuteVector << <blocks, threadsPerBlock >> > (d_b, dev_b_permuted, d_p, N);
    cusolverSpScsrcholSolve(cusolverHandle, N, dev_b_permuted, dev_x_permuted, d_info, buffer_gpu);
    permuteVectorInv << <blocks, threadsPerBlock >> > (dev_x_permuted, d_x, d_p, N);
}
