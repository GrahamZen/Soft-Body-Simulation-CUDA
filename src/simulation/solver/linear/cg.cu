#include <linear/cg.h>
#include <linear/error_helper.h>
#include <iostream>
#include <vector>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
template <typename T>
void inspectHost(const T* host_ptr, int size) {
    std::cout << "---------------------------inspectHost--------------------------------" << std::endl;

    for (int i = 0; i < size; i++) {
        std::cout << host_ptr[i] << std::endl;
    }
    std::cout << "------------------------inspectHost--END------------------------------" << std::endl;
}


template <typename T>
void inspectGLM(T* dev_ptr, int size) {
    std::vector<T> host_ptr(size);
    hipMemcpy(host_ptr.data(), dev_ptr, sizeof(T) * size, hipMemcpyDeviceToHost);
    inspectHost(host_ptr.data(), size);
}
void printSparseCOOToFull(int N, int nz, float* d_val, int* d_rowIdx, int* d_colIdx) {
    std::vector<int> rowIdx(nz);
    std::vector<int> colIdx(nz);
    std::vector<float> val(nz);

    hipMemcpy(rowIdx.data(), d_rowIdx, sizeof(int) * nz, hipMemcpyDeviceToHost);
    hipMemcpy(colIdx.data(), d_colIdx, sizeof(int) * nz, hipMemcpyDeviceToHost);
    hipMemcpy(val.data(), d_val, sizeof(float) * nz, hipMemcpyDeviceToHost);

    std::vector<std::vector<float>> full(N, std::vector<float>(N, 0));

    for (int i = 0; i < nz; i++) {
        full[rowIdx[i]][colIdx[i]] = val[i];
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << full[i][j] << " ";
        }
        std::cout << std::endl;
    }
}
void printSparseCSRToFull(int N, int nz, float* d_val, int* d_rowIdx, int* d_colIdx) {
    std::vector<int> rowIdx(N + 1);
    std::vector<int> colIdx(nz);
    std::vector<float> val(nz);

    hipMemcpy(rowIdx.data(), d_rowIdx, sizeof(int) * (N + 1), hipMemcpyDeviceToHost);
    hipMemcpy(colIdx.data(), d_colIdx, sizeof(int) * nz, hipMemcpyDeviceToHost);
    hipMemcpy(val.data(), d_val, sizeof(float) * nz, hipMemcpyDeviceToHost);

    std::vector<std::vector<float>> full(N, std::vector<float>(N, 0));

    // csr to full
    for (int i = 0; i < N; i++) {
        for (int j = rowIdx[i]; j < rowIdx[i + 1]; j++) {
            full[i][colIdx[j]] = val[j];
        }
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << full[i][j] << " ";
        }
        std::cout << std::endl;
    }
}

CGSolver::CGSolver(int N) :N(N)
{
    if (cubHandle == NULL)
    {
        error_check(hipblasCreate(&cubHandle));
    }


    // create cuSPARSE cusHandle
    if (cusHandle == NULL)
    {
        error_check(hipsparseCreate(&cusHandle));
    }

    // create descriptor for matrix A
    error_check(hipsparseCreateMatDescr(&descrA));

    // initialize properties of matrix A
    error_check(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    error_check(hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_LOWER));
    error_check(hipsparseSetMatDiagType(descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    error_check(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    // create descriptor for matrix L
    error_check(hipsparseCreateMatDescr(&descrL));

    // initialize properties of matrix L
    error_check(hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL));
    error_check(hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER));
    error_check(hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO));
    error_check(hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    error_check(hipMalloc(&d_x, N * sizeof(float)));
    error_check(hipMalloc(&d_y, N * sizeof(float)));
    error_check(hipMalloc(&d_z, N * sizeof(float)));
    error_check(hipMalloc(&d_r, N * sizeof(float)));
    error_check(hipMalloc(&d_rt, N * sizeof(float)));
    error_check(hipMalloc(&d_xt, N * sizeof(float)));
    error_check(hipMalloc(&d_q, N * sizeof(float)));
    error_check(hipMalloc(&d_p, N * sizeof(float)));
    error_check(hipMalloc(&d_rowPtrA, (N + 1) * sizeof(int)));
    error_check(hipMemset(d_x, 0, N * sizeof(float)));
    error_check(hipMemset(d_y, 0, N * sizeof(float)));
    error_check(hipMemset(d_z, 0, N * sizeof(float)));
    error_check(hipMemset(d_r, 0, N * sizeof(float)));
    error_check(hipMemset(d_rt, 0, N * sizeof(float)));
    error_check(hipMemset(d_xt, 0, N * sizeof(float)));
    error_check(hipMemset(d_q, 0, N * sizeof(float)));
    error_check(hipMemset(d_p, 0, N * sizeof(float)));
}

CGSolver::~CGSolver()
{
    error_check(hipFree(&d_x));
    error_check(hipFree(&d_y));
    error_check(hipFree(&d_z));
    error_check(hipFree(&d_r));
    error_check(hipFree(&d_rt));
    error_check(hipFree(&d_xt));
    error_check(hipFree(&d_q));
    error_check(hipFree(&d_p));
}

void sort_coo(int N, int nz, float* d_A, int* d_rowIdx, int* d_colIdx) {
    // 将裸指针转换为 Thrust 的 device_ptr
    thrust::device_ptr<int> d_rowIdx_ptr(d_rowIdx);
    thrust::device_ptr<int> d_colIdx_ptr(d_colIdx);
    thrust::device_ptr<float> d_A_ptr(d_A);

    // 创建 zip_iterator
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(d_rowIdx_ptr, d_colIdx_ptr, d_A_ptr));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(d_rowIdx_ptr + nz, d_colIdx_ptr + nz, d_A_ptr + nz));

    // 按 rowIdx 排序，按 colIdx 次序排序
    thrust::sort(begin, end, thrust::less<thrust::tuple<int, int, float>>());
}

void CGSolver::Solve(int N, float* d_b, float* d_x, float* d_A, int nz, int* d_rowIdx, int* d_colIdx, float* d_guess)
{
    sort_coo(N, nz, d_A, d_rowIdx, d_colIdx);
    printSparseCOOToFull(N, nz, d_A, d_rowIdx, d_colIdx);
    hipsparseXcoo2csr(cusHandle, d_rowIdx, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);
    inspectGLM(d_rowPtrA, N + 1);
    printSparseCSRToFull(N, nz, d_A, d_rowPtrA, d_colIdx);
    error_check(hipsparseCreateCsr(&spMatDescrA, N, N, nz, d_rowPtrA, d_colIdx, d_A, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    hipsparseDnVecDescr_t dvec_p;
    error_check(hipsparseCreateDnVec(&dvec_p, N, d_p, HIP_R_32F));

    hipsparseDnVecDescr_t dvec_q;
    error_check(hipsparseCreateDnVec(&dvec_q, N, d_q, HIP_R_32F));
    error_check(hipsparseCreateDnVec(&dvec_x, N, d_x, HIP_R_32F));
    error_check(hipsparseCreateDnVec(&dvec_b, N, d_b, HIP_R_32F));

    // Incomplete Cholesky factorization
    error_check(hipMalloc(&d_ic, nz * sizeof(float)));
    error_check(hipMemcpy(d_ic, d_A, nz * sizeof(float), hipMemcpyDeviceToDevice));

    error_check(hipsparseCreateCsric02Info(&ic02info));

    int ic02BufferSizeInBytes = 0;
    error_check(hipsparseScsric02_bufferSize(cusHandle, N, nz, descrA, d_ic, d_rowPtrA, d_colIdx, ic02info, &ic02BufferSizeInBytes));

    void* ic02Buffer = nullptr;
    error_check(hipMalloc(&ic02Buffer, ic02BufferSizeInBytes));
    error_check(hipsparseScsric02_analysis(cusHandle, N, nz, descrA, d_ic, d_rowPtrA, d_colIdx, ic02info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, ic02Buffer));

    error_check(hipsparseScsric02(cusHandle, N, nz, descrA, d_ic, d_rowPtrA, d_colIdx, ic02info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, ic02Buffer));
    error_check(hipsparseCreateCsr(&spMatDescrL, N, N, nz, d_rowPtrA, d_colIdx, d_ic, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    printSparseCSRToFull(N, nz, d_ic, d_rowPtrA, d_colIdx);
    // Prepare 
    error_check(hipsparseSpSV_createDescr(&spsvDescrL));
    error_check(hipsparseSpSV_createDescr(&spsvDescrU));

    size_t tmpBufferSize = 0;
    size_t bufferSize = 0;
    error_check(hipsparseSpSV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrL, dvec_x, dvec_b, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &tmpBufferSize));
    error_check(hipsparseSpSV_bufferSize(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, spMatDescrL, dvec_x, dvec_b, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &bufferSize));

    if (tmpBufferSize > bufferSize)
        bufferSize = tmpBufferSize;

    error_check(hipsparseSpMV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrA, dvec_p, &zero, dvec_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, &tmpBufferSize));
    if (tmpBufferSize > bufferSize)
        bufferSize = tmpBufferSize;

    error_check(hipMalloc(&d_buf, bufferSize));
    error_check(hipMalloc(&d_buf1, bufferSize));

    error_check(hipsparseSpSV_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrL, dvec_x, dvec_b, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, d_buf));
    error_check(hipsparseSpSV_analysis(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, spMatDescrL, dvec_x, dvec_b, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, d_buf1));

    // x = 0
    // r0 = b  (since x == 0, b - A*x = b)
    error_check(hipMemcpy(d_r, d_b, N * sizeof(float), hipMemcpyDeviceToDevice));
    inspectGLM(d_b, N);

    error_check(hipsparseCreateDnVec(&dvec_r, N, d_r, HIP_R_32F));
    error_check(hipsparseCreateDnVec(&dvec_y, N, d_y, HIP_R_32F));
    error_check(hipsparseCreateDnVec(&dvec_z, N, d_z, HIP_R_32F));

    if (d_guess != nullptr)
    {
        // x = guess
        error_check(hipMemcpy(d_x, d_guess, N * sizeof(float), hipMemcpyDeviceToDevice));
        // r0 = b - A*x
        //     q = A*x
        //     r0 = -q + b
        error_check(hipsparseSpMV(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrA,
            dvec_x, &zero, dvec_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, d_buf));
        float n_one = -1;
        error_check(hipblasSaxpy(cubHandle, N, &n_one, d_q, 1, d_r, 1));
    }

    for (k = 0; k < max_iter; ++k)
    {
        // if ||rk|| < tolerance
        error_check(hipblasSnrm2(cubHandle, N, d_r, 1, &rTr));
        //std::cout << "Iteration " << k << ": " << rTr << std::endl;
        if (rTr < tolerance)
        {
            break;
        }
        // Solve L*y = rk
        error_check(hipsparseSpSV_solve(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
            spMatDescrL, dvec_r, dvec_y, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));
        inspectGLM(d_y, N);

        // Solve L^T*zk = y
        error_check(hipsparseSpSV_solve(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one,
            spMatDescrL, dvec_y, dvec_z, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));
        inspectGLM(d_z, N);

        // rho_t = r{k-1} * z{k-1}
        rho_t = rho;
        // rho = rk * zk
        error_check(hipblasSdot(cubHandle, N, d_r, 1, d_z, 1, &rho));

        if (k == 0)
        {
            // pk = zk
            error_check(hipblasScopy(cubHandle, N, d_z, 1, d_p, 1));
        }
        else
        {
            // beta = (rk*zk) / (r{k-1}*z{k-1})
            beta = rho / rho_t;
            // pk = zk + beta*p{k-1}
            error_check(hipblasSscal(cubHandle, N, &beta, d_p, 1));
            error_check(hipblasSaxpy(cubHandle, N, &one, d_z, 1, d_p, 1));
        }

        // q = A*pk
        error_check(hipsparseSpMV(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrA,
            dvec_p, &zero, dvec_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, d_buf));

        // alpha = (rk*zk) / (pk*q)
        error_check(hipblasSdot(cubHandle, N, d_p, 1, d_q, 1, &pTq));
        alpha = rho / pTq;

        // x{k+1} = xk + alpha*pk
        error_check(hipblasSaxpy(cubHandle, N, &alpha, d_p, 1, d_x, 1));

        // r{k+1} = rk - alpha*q 
        float n_alpha = -alpha;
        error_check(hipblasSaxpy(cubHandle, N, &n_alpha, d_q, 1, d_r, 1));
    }

    error_check(hipsparseDestroySpMat(spMatDescrA));
    error_check(hipsparseDestroySpMat(spMatDescrL));
    error_check(hipsparseDestroyDnVec(dvec_p));
    error_check(hipsparseDestroyDnVec(dvec_q));
    error_check(hipsparseDestroyDnVec(dvec_x));
    error_check(hipsparseDestroyCsric02Info(ic02info));
    error_check(hipsparseSpSV_destroyDescr(spsvDescrL));
    error_check(hipsparseSpSV_destroyDescr(spsvDescrU));
}
