#include <linear/cg.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
    }                                                                          \
}

CGSolver::CGSolver(int N) :N(N)
{
    CHECK_CUBLAS(hipblasCreate(&cubHandle));

    CHECK_CUSPARSE(hipsparseCreate(&cusHandle));
    // create descriptor for matrix A
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));

    // initialize properties of matrix A
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_LOWER));
    CHECK_CUSPARSE(hipsparseSetMatDiagType(descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    // create descriptor for matrix L
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrL));

    // initialize properties of matrix L
    CHECK_CUSPARSE(hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_NON_UNIT));

    CHECK_CUSPARSE(hipsparseCreateCsric02Info(&ic02info));
    CHECK_CUSPARSE(hipsparseSpSV_createDescr(&spsvDescrL));
    CHECK_CUSPARSE(hipsparseSpSV_createDescr(&spsvDescrU));

    CHECK_CUDA(hipMalloc((void**)&d_y, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_z, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_r, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_rt, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_xt, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_q, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_p, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_rowPtrA, (N + 1) * sizeof(int)));
    CHECK_CUDA(hipMemset(d_y, 0, N * sizeof(float)));
    CHECK_CUDA(hipMemset(d_z, 0, N * sizeof(float)));
    CHECK_CUDA(hipMemset(d_r, 0, N * sizeof(float)));
    CHECK_CUDA(hipMemset(d_rt, 0, N * sizeof(float)));
    CHECK_CUDA(hipMemset(d_xt, 0, N * sizeof(float)));
    CHECK_CUDA(hipMemset(d_q, 0, N * sizeof(float)));
    CHECK_CUDA(hipMemset(d_p, 0, N * sizeof(float)));
}

CGSolver::~CGSolver()
{
    CHECK_CUDA(hipFree(d_y));
    CHECK_CUDA(hipFree(d_z));
    CHECK_CUDA(hipFree(d_r));
    CHECK_CUDA(hipFree(d_rt));
    CHECK_CUDA(hipFree(d_xt));
    CHECK_CUDA(hipFree(d_q));
    CHECK_CUDA(hipFree(d_p));
    CHECK_CUDA(hipFree(d_rowPtrA));
    CHECK_CUDA(hipFree(d_ic));

    CHECK_CUBLAS(hipblasDestroy(cubHandle));
    CHECK_CUSPARSE(hipsparseDestroy(cusHandle));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrL));
    CHECK_CUSPARSE(hipsparseDestroyCsric02Info(ic02info));
    CHECK_CUSPARSE(hipsparseSpSV_destroyDescr(spsvDescrL));
    CHECK_CUSPARSE(hipsparseSpSV_destroyDescr(spsvDescrU));
}

void sort_coo(int N, int nz, float* d_A, int* d_rowIdx, int* d_colIdx) {
    thrust::device_ptr<int> d_rowIdx_ptr(d_rowIdx);
    thrust::device_ptr<int> d_colIdx_ptr(d_colIdx);
    thrust::device_ptr<float> d_A_ptr(d_A);

    auto begin = thrust::make_zip_iterator(thrust::make_tuple(d_rowIdx_ptr, d_colIdx_ptr, d_A_ptr));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(d_rowIdx_ptr + nz, d_colIdx_ptr + nz, d_A_ptr + nz));

    thrust::sort(begin, end, thrust::less<thrust::tuple<int, int, float>>());
}

void CGSolver::Solve(int N, float* d_b, float* d_x, float* d_A, int nz, int* d_rowIdx, int* d_colIdx, float* d_guess)
{
    assert(d_b != nullptr);
    assert(d_x != nullptr);
    assert(d_A != nullptr);
    assert(d_rowIdx != nullptr);
    assert(d_colIdx != nullptr);
    sort_coo(N, nz, d_A, d_rowIdx, d_colIdx);
    hipsparseXcoo2csr(cusHandle, d_rowIdx, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE(hipsparseCreateCsr(&spMatDescrA, N, N, nz, d_rowPtrA, d_colIdx, d_A, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    hipsparseDnVecDescr_t dvec_p;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_p, N, d_p, HIP_R_32F));

    hipsparseDnVecDescr_t dvec_q;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_q, N, d_q, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_x, N, d_x, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_b, N, d_b, HIP_R_32F));

    // Incomplete Cholesky factorization
    CHECK_CUDA(hipMalloc((void**)&d_ic, nz * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_ic, d_A, nz * sizeof(float), hipMemcpyDeviceToDevice));

    int ic02BufferSizeInBytes = 0;
    CHECK_CUSPARSE(hipsparseScsric02_bufferSize(cusHandle, N, nz, descrA, d_ic, d_rowPtrA, d_colIdx, ic02info, &ic02BufferSizeInBytes));

    void* ic02Buffer = nullptr;
    CHECK_CUDA(hipMalloc((void**)&ic02Buffer, ic02BufferSizeInBytes));
    CHECK_CUSPARSE(hipsparseScsric02_analysis(cusHandle, N, nz, descrA, d_ic, d_rowPtrA, d_colIdx, ic02info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, ic02Buffer));

    CHECK_CUSPARSE(hipsparseScsric02(cusHandle, N, nz, descrA, d_ic, d_rowPtrA, d_colIdx, ic02info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, ic02Buffer));
    CHECK_CUSPARSE(hipsparseCreateCsr(&spMatDescrL, N, N, nz, d_rowPtrA, d_colIdx, d_ic, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    // Prepare 


    size_t tmpBufferSize = 0;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSpSV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrL, dvec_x, dvec_b, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &tmpBufferSize));
    CHECK_CUSPARSE(hipsparseSpSV_bufferSize(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, spMatDescrL, dvec_x, dvec_b, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &bufferSize));

    if (tmpBufferSize > bufferSize)
        bufferSize = tmpBufferSize;

    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrA, dvec_p, &zero, dvec_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, &tmpBufferSize));
    if (tmpBufferSize > bufferSize)
        bufferSize = tmpBufferSize;

    CHECK_CUDA(hipMalloc((void**)&d_bufL, bufferSize));
    CHECK_CUDA(hipMalloc((void**)&d_bufU, bufferSize));

    CHECK_CUSPARSE(hipsparseSpSV_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrL, dvec_x, dvec_b, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, d_bufL));
    CHECK_CUSPARSE(hipsparseSpSV_analysis(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, spMatDescrL, dvec_x, dvec_b, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, d_bufU));

    // x = 0
    // r0 = b  (since x == 0, b - A*x = b)
    CHECK_CUDA(hipMemcpy(d_r, d_b, N * sizeof(float), hipMemcpyDeviceToDevice));

    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_r, N, d_r, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_y, N, d_y, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_z, N, d_z, HIP_R_32F));

    if (d_guess != nullptr)
    {
        // x = guess
        CHECK_CUDA(hipMemcpy(d_x, d_guess, N * sizeof(float), hipMemcpyDeviceToDevice));
        // r0 = b - A*x
        //     q = A*x
        //     r0 = -q + b
        CHECK_CUSPARSE(hipsparseSpMV(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrA,
            dvec_x, &zero, dvec_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, d_bufL));
        float n_one = -1;
        CHECK_CUBLAS(hipblasSaxpy(cubHandle, N, &n_one, d_q, 1, d_r, 1));
    }

    for (k = 0; k < max_iter; ++k)
    {
        // if ||rk|| < tolerance
        CHECK_CUBLAS(hipblasSnrm2(cubHandle, N, d_r, 1, &rTr));
        //std::cout << "Iteration " << k << ": " << rTr << std::endl;
        if (rTr < tolerance)
        {
            break;
        }
        // Solve L*y = rk
        CHECK_CUSPARSE(hipsparseSpSV_solve(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
            spMatDescrL, dvec_r, dvec_y, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

        // Solve L^T*zk = y
        CHECK_CUSPARSE(hipsparseSpSV_solve(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one,
            spMatDescrL, dvec_y, dvec_z, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

        // rho_t = r{k-1} * z{k-1}
        rho_t = rho;
        // rho = rk * zk
        CHECK_CUBLAS(hipblasSdot(cubHandle, N, d_r, 1, d_z, 1, &rho));

        if (k == 0)
        {
            // pk = zk
            CHECK_CUBLAS(hipblasScopy(cubHandle, N, d_z, 1, d_p, 1));
        }
        else
        {
            // beta = (rk*zk) / (r{k-1}*z{k-1})
            beta = rho / rho_t;
            // pk = zk + beta*p{k-1}
            CHECK_CUBLAS(hipblasSscal(cubHandle, N, &beta, d_p, 1));
            CHECK_CUBLAS(hipblasSaxpy(cubHandle, N, &one, d_z, 1, d_p, 1));
        }

        // q = A*pk
        CHECK_CUSPARSE(hipsparseSpMV(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, spMatDescrA,
            dvec_p, &zero, dvec_q, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, d_bufL));

        // alpha = (rk*zk) / (pk*q)
        CHECK_CUBLAS(hipblasSdot(cubHandle, N, d_p, 1, d_q, 1, &pTq));
        alpha = rho / pTq;

        // x{k+1} = xk + alpha*pk
        CHECK_CUBLAS(hipblasSaxpy(cubHandle, N, &alpha, d_p, 1, d_x, 1));

        // r{k+1} = rk - alpha*q 
        float n_alpha = -alpha;
        CHECK_CUBLAS(hipblasSaxpy(cubHandle, N, &n_alpha, d_q, 1, d_r, 1));
    }

    CHECK_CUSPARSE(hipsparseDestroySpMat(spMatDescrA));
    CHECK_CUSPARSE(hipsparseDestroySpMat(spMatDescrL));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_r));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_b));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_p));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_q));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_x));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_y));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_z));
}
