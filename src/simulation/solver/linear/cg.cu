#include <linear/cg.h>
#include <linear/cuUtils.cuh>
#include <iostream>

template<typename T>
CGSolver<T>::CGSolver(int N, int max_iter, T tolerance) : N(N), max_iter(max_iter), tolerance(tolerance)
{
    CHECK_CUBLAS(hipblasCreate(&cubHandle));

    CHECK_CUSPARSE(hipsparseCreate(&cusHandle));
    // create descriptor for matrix A
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));

    // initialize properties of matrix A
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_LOWER));
    CHECK_CUSPARSE(hipsparseSetMatDiagType(descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    // create descriptor for matrix L
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrL));

    // initialize properties of matrix L
    CHECK_CUSPARSE(hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_NON_UNIT));

    CHECK_CUSPARSE(hipsparseCreateCsric02Info(&ic02info));
    CHECK_CUSPARSE(hipsparseSpSV_createDescr(&spsvDescrL));
    CHECK_CUSPARSE(hipsparseSpSV_createDescr(&spsvDescrU));

    CHECK_CUDA(hipMalloc((void**)&d_y, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void**)&d_z, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void**)&d_r, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void**)&d_q, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void**)&d_p, N * sizeof(T)));
    CHECK_CUDA(hipMalloc((void**)&d_rowPtrA, (N + 1) * sizeof(int)));
    CHECK_CUDA(hipMemset(d_y, 0, N * sizeof(T)));
    CHECK_CUDA(hipMemset(d_z, 0, N * sizeof(T)));
    CHECK_CUDA(hipMemset(d_r, 0, N * sizeof(T)));
    CHECK_CUDA(hipMemset(d_q, 0, N * sizeof(T)));
    CHECK_CUDA(hipMemset(d_p, 0, N * sizeof(T)));

    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_p, N, d_p, dType));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_q, N, d_q, dType));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_y, N, d_y, dType));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_z, N, d_z, dType));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_r, N, d_r, dType));
}

template<typename T>
CGSolver<T>::~CGSolver()
{
    CHECK_CUDA(hipFree(d_y));
    CHECK_CUDA(hipFree(d_z));
    CHECK_CUDA(hipFree(d_r));
    CHECK_CUDA(hipFree(d_q));
    CHECK_CUDA(hipFree(d_p));
    CHECK_CUDA(hipFree(d_rowPtrA));
    CHECK_CUDA(hipFree(d_ic));
    CHECK_CUDA(hipFree(d_bufL));
    CHECK_CUDA(hipFree(d_bufU));

    CHECK_CUBLAS(hipblasDestroy(cubHandle));
    CHECK_CUSPARSE(hipsparseDestroy(cusHandle));

    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_r));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_p));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_q));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_y));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_z));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrL));
    CHECK_CUSPARSE(hipsparseDestroyCsric02Info(ic02info));
    CHECK_CUSPARSE(hipsparseSpSV_destroyDescr(spsvDescrL));
    CHECK_CUSPARSE(hipsparseSpSV_destroyDescr(spsvDescrU));
}
template<typename T>
void CGSolver<T>::Solve(int N, T* d_b, T* d_x, T* A, int nz, int* rowIdx, int* colIdx, T* d_guess)
{
    assert(d_b != nullptr);
    assert(d_x != nullptr);
    assert(A != nullptr);
    assert(rowIdx != nullptr);
    assert(colIdx != nullptr);
    CHECK_CUDA(hipMemset(d_x, 0, N * sizeof(T)));

    //==============================================================================
    // Sort the COO matrix by row index and convert it to CSR format
    sort_coo(N, nz, A, rowIdx, colIdx, d_A, d_rowIdx, d_colIdx);
    hipsparseXcoo2csr(cusHandle, d_rowIdx, nz, N, d_rowPtrA, HIPSPARSE_INDEX_BASE_ZERO);
    CHECK_CUSPARSE(hipsparseCreateCsr(&d_matA, N, N, nz, d_rowPtrA, d_colIdx, d_A,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, dType));

    //==============================================================================
    // Create dense vectors for p, q, x, b, y, z, r
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_x, N, d_x, dType));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&dvec_b, N, d_b, dType));
    // x = 0, r0 = b  (since x == 0, b - A*x = b)
    CHECK_CUDA(hipMemcpy(d_r, d_b, N * sizeof(T), hipMemcpyDeviceToDevice));

    //==============================================================================
    // L = ichol(A), L is a lower triangular matrix
    if (nz > old_nnz) {
        if (d_ic != nullptr)
            CHECK_CUDA(hipFree(d_ic));
        CHECK_CUDA(hipMalloc((void**)&d_ic, nz * sizeof(T)));
        std::cout << "d_ic malloc." << std::endl;
        old_nnz = nz;
    }

    CHECK_CUDA(hipMemcpy(d_ic, d_A, nz * sizeof(T), hipMemcpyDeviceToDevice));

    int ic02BufferSizeInBytes = 0;
    CHECK_CUSPARSE(cusparsecsric02_bufferSize(cusHandle, N, nz, descrA, d_ic,
        d_rowPtrA, d_colIdx, ic02info, &ic02BufferSizeInBytes));

    if (ic02BufferSizeInBytes > old_ic02BufferSizeInBytes)
    {
        if (ic02Buffer != nullptr)
            CHECK_CUDA(hipFree(ic02Buffer));
        CHECK_CUDA(hipMalloc((void**)&ic02Buffer, ic02BufferSizeInBytes));
        std::cout << "ic02Buffer malloc." << std::endl;
        old_ic02BufferSizeInBytes = ic02BufferSizeInBytes;
    }
    CHECK_CUSPARSE(cusparsecsric02_analysis(cusHandle, N, nz, descrA, d_ic,
        d_rowPtrA, d_colIdx, ic02info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, ic02Buffer));

    CHECK_CUSPARSE(cusparsecsric02(cusHandle, N, nz, descrA, d_ic,
        d_rowPtrA, d_colIdx, ic02info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, ic02Buffer));
    CHECK_CUSPARSE(hipsparseCreateCsr(&d_matL, N, N, nz, d_rowPtrA, d_colIdx, d_ic,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, dType));

    //============================================================================== 
    // Prepare workspace for solving L*y = b and L^T*z = y
    size_t bufferSizeL = 0;
    size_t bufferSizeU = 0;
    size_t tmpBufferSize = 0;

    CHECK_CUSPARSE(hipsparseSpSV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, d_matL,
        dvec_x, dvec_b, dType, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL));
    CHECK_CUSPARSE(hipsparseSpSV_bufferSize(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, d_matL,
        dvec_x, dvec_b, dType, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &bufferSizeU));

    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, d_matA,
        dvec_p, &zero, dvec_q, dType, HIPSPARSE_SPMV_CSR_ALG1, &tmpBufferSize));
    if (tmpBufferSize > bufferSizeL)
        bufferSizeL = tmpBufferSize;

    if (bufferSizeL > old_bufferSizeL)
    {
        if (d_bufL != nullptr)
            CHECK_CUDA(hipFree(d_bufL));
        CHECK_CUDA(hipMalloc((void**)&d_bufL, bufferSizeL));
        std::cout << "d_bufL malloc." << std::endl;
        old_bufferSizeL = bufferSizeL;
    }

    if (bufferSizeU > old_bufferSizeU)
    {
        if (d_bufU != nullptr)
            CHECK_CUDA(hipFree(d_bufU));
        CHECK_CUDA(hipMalloc((void**)&d_bufU, bufferSizeU));
        std::cout << "d_bufU malloc." << std::endl;
        old_bufferSizeU = bufferSizeU;
    }

    CHECK_CUSPARSE(hipsparseSpSV_analysis(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, d_matL,
        dvec_x, dvec_b, dType, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, d_bufL));
    CHECK_CUSPARSE(hipsparseSpSV_analysis(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one, d_matL,
        dvec_x, dvec_b, dType, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, d_bufU));

    //==============================================================================
    // Set initial guess
    if (d_guess != nullptr)
    {
        // x = guess
        CHECK_CUDA(hipMemcpy(d_x, d_guess, N * sizeof(T), hipMemcpyDeviceToDevice));
        // r0 = b - A*x
        // q = A*x
        // r0 = -q + b
        CHECK_CUSPARSE(hipsparseSpMV(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, d_matA,
            dvec_x, &zero, dvec_q, dType, HIPSPARSE_SPMV_CSR_ALG1, d_bufL));
        CHECK_CUBLAS(cublasAxpy(cubHandle, N, (T)-1, d_q, 1, d_r, 1));
    }

    //==============================================================================
    // PCG solver Begin
    for (k = 0; k < max_iter; ++k)
    {
        // if ||rk|| < tolerance
        CHECK_CUBLAS(cublasnrm2(cubHandle, N, d_r, 1, &rTr));
        if (rTr < tolerance)
        {
            break;
        }
        // Solve L*y = rk
        CHECK_CUSPARSE(hipsparseSpSV_solve(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one,
            d_matL, dvec_r, dvec_y, dType, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

        // Solve L^T*zk = y
        CHECK_CUSPARSE(hipsparseSpSV_solve(cusHandle, HIPSPARSE_OPERATION_TRANSPOSE, &one,
            d_matL, dvec_y, dvec_z, dType, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

        // rho_t = r{k-1} * z{k-1}
        rho_t = rho;
        // rho = rk * zk
        CHECK_CUBLAS(cublasdot(cubHandle, N, d_r, 1, d_z, 1, &rho));

        if (k == 0)
        {
            // pk = zk
            CHECK_CUBLAS(cublascopy(cubHandle, N, d_z, 1, d_p, 1));
        }
        else
        {
            // beta = (rk*zk) / (r{k-1}*z{k-1})
            beta = rho / rho_t;
            // pk = zk + beta*p{k-1}
            CHECK_CUBLAS(cublasscal(cubHandle, N, beta, d_p, 1));
            CHECK_CUBLAS(cublasAxpy(cubHandle, N, (T)1, d_z, 1, d_p, 1));
        }

        // q = A*pk
        CHECK_CUSPARSE(hipsparseSpMV(cusHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, d_matA,
            dvec_p, &zero, dvec_q, dType, HIPSPARSE_SPMV_CSR_ALG1, d_bufL));

        // alpha = (rk*zk) / (pk*q)
        CHECK_CUBLAS(cublasdot(cubHandle, N, d_p, 1, d_q, 1, &pTq));
        alpha = rho / pTq;

        // x{k+1} = xk + alpha*pk
        CHECK_CUBLAS(cublasAxpy(cubHandle, N, alpha, d_p, 1, d_x, 1));

        // r{k+1} = rk - alpha*q 
        CHECK_CUBLAS(cublasAxpy(cubHandle, N, -alpha, d_q, 1, d_r, 1));
    }

    CHECK_CUSPARSE(hipsparseDestroySpMat(d_matA));
    CHECK_CUSPARSE(hipsparseDestroySpMat(d_matL));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_b));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(dvec_x));
}

template class CGSolver<float>;
template class CGSolver<double>;