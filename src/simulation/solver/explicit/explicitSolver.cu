#include <simulation/solver/explicit/explicitSolver.h>
#include <simulation/solver/explicit/explicitUtil.cuh>
#include <simulation/solver/solverUtil.cuh>
#include <simulation/simulationContext.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>

ExplicitSolver::ExplicitSolver(int threadsPerBlock, const SolverData<float>& solverData) : FEMSolver(threadsPerBlock)
{
    if (!solverData.dev_ExtForce)
        hipMalloc((void**)&solverData.dev_ExtForce, sizeof(glm::vec3) * solverData.numVerts);
    hipMemset(solverData.dev_ExtForce, 0, sizeof(glm::vec3) * solverData.numVerts);
    if (!solverData.inv_Dm)
        hipMalloc((void**)&solverData.inv_Dm, sizeof(glm::mat4) * solverData.numTets);
    hipMalloc((void**)&V_sum, sizeof(glm::vec3) * solverData.numVerts);
    hipMalloc((void**)&V_num, sizeof(int) * solverData.numVerts);
    hipMemset(V_sum, 0, sizeof(glm::vec3) * solverData.numVerts);
    hipMemset(V_num, 0, sizeof(int) * solverData.numVerts);
    int blocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    ExplicitUtil::computeInvDm << < blocks, threadsPerBlock >> > (solverData.inv_Dm, solverData.numTets, solverData.X, solverData.Tet);
}

ExplicitSolver::~ExplicitSolver()
{
}


void ExplicitSolver::SolverPrepare(SolverData<float>& solverData, SolverParams& solverParams)
{
}


void ExplicitSolver::SolverStep(SolverData<float>& solverData, SolverParams& solverParams)
{
    glm::vec3 gravity{ 0.0f, -solverParams.gravity * solverParams.solverAttr.mass, 0.0f };
    thrust::device_ptr<glm::vec3> dev_ptr(solverData.Force);
    thrust::fill(thrust::device, dev_ptr, dev_ptr + solverData.numVerts, gravity);
    Laplacian_Smoothing(solverData, 0.5);
    ExplicitUtil::ComputeForcesSVD << <(solverData.numTets + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (solverData.Force, solverData.XTilde, solverData.Tet, solverData.numTets, solverData.inv_Dm, solverParams.solverAttr.stiffness_0, solverParams.solverAttr.stiffness_1);
    ExplicitUtil::EulerMethod << <(solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (solverData.XTilde, solverData.V, solverData.Force, solverData.numVerts, solverParams.solverAttr.mass, solverParams.dt);
}


void ExplicitSolver::Update(SolverData<float>& solverData, SolverParams& solverParams)
{
    AddExternal << <(solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (solverData.V, solverData.numVerts, solverParams.solverAttr.jump, solverParams.solverAttr.mass, solverParams.extForce.jump);
    for (size_t i = 0; i < 10; i++)
    {
        SolverStep(solverData, solverParams);
    }
}


void ExplicitSolver::Laplacian_Smoothing(SolverData<float>& solverData, float blendAlpha)
{
    hipMemset(V_sum, 0, sizeof(glm::vec3) * solverData.numVerts);
    hipMemset(V_num, 0, sizeof(int) * solverData.numVerts);
    int blocks = (solverData.numTets + threadsPerBlock - 1) / threadsPerBlock;
    ExplicitUtil::LaplacianGatherKern << < blocks, threadsPerBlock >> > (solverData.V, V_sum, V_num, solverData.numTets, solverData.Tet);
    ExplicitUtil::LaplacianKern << < (solverData.numVerts + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> > (solverData.V, V_sum, V_num, solverData.numVerts, solverData.Tet, blendAlpha);
}