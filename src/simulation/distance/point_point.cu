#include "hip/hip_runtime.h"
#include "point_point.h"
#include <glm/gtx/norm.hpp> 
#include <hip/hip_runtime.h>

namespace ipc {

    template<typename Scalar>
    __device__ Scalar point_point_distance(
        const glm::tvec3<Scalar>& p0,
        const glm::tvec3<Scalar>& p1)
    {
        return glm::length2(p0 - p1);
    }

    template __device__ float point_point_distance<float>(
        const glm::tvec3<float>& p0,
        const glm::tvec3<float>& p1);

    template __device__ double point_point_distance<double>(
        const glm::tvec3<double>& p0,
        const glm::tvec3<double>& p1);

     template<typename Scalar>
     __device__ Vector<Scalar, 6> point_point_distance_gradient(
         const glm::tvec3<Scalar>& p0,
         const glm::tvec3<Scalar>& p1)
     {
         auto v = (Scalar)2.0 * (p0 - p1);
         Vector<Scalar, 6> grad;
         Vector<Scalar, 3> tmp = v;
         Vector<Scalar, 3> tmpM = -v;

         grad.head(3) = tmp;
         grad.tail(3) = tmpM;

         return grad;
     }

    template __device__ Vector<float, 6> point_point_distance_gradient<float>(
        const glm::tvec3<float>& p0,
        const glm::tvec3<float>& p1);

    template __device__ Vector<double, 6> point_point_distance_gradient<double>(
        const glm::tvec3<double>& p0,
        const glm::tvec3<double>& p1);

    // template<typename Scalar>
    // MatrixMax6d point_point_distance_hessian(
    //     const glm::tvec3<Scalar>& p0,
    //     const glm::tvec3<Scalar>& p1)
    // {
    //     int dim = p0.size();
    //     assert(p1.size() == dim);

    //     MatrixMax6d hess(2 * dim, 2 * dim);

    //     hess.setZero();
    //     hess.diagonal().setConstant(2.0);
    //     for (int i = 0; i < dim; i++) {
    //         hess(i, i + dim) = hess(i + dim, i) = -2;
    //     }

    //     return hess;
    // }

} // namespace ipc
