#include <simulationContext.h>
#include <utilities.cuh>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <hipsolver.h>
#include <hipsparse.h>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)

void SoftBody::solverPrepare()
{
    int threadsPerBlock = 64;
    int vertBlocks = (numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (numTets + threadsPerBlock - 1) / threadsPerBlock;
    float dt = mpSimContext->GetDt();
    float const m_1_dt2 = mass / (dt * dt);
    int len = numVerts * 3 + 48 * numTets;
    int ASize = 3 * numVerts;

    hipMalloc((void**)&sn, sizeof(float) * ASize);
    hipMalloc((void**)&b, sizeof(float) * ASize);
    hipMalloc((void**)&masses, sizeof(float) * ASize);

    int* AIdx;
    hipMalloc((void**)&AIdx, sizeof(int) * len);
    hipMemset(AIdx, 0, sizeof(int) * len);

    float* tmpVal;
    hipMalloc((void**)&tmpVal, sizeof(int) * len);
    hipMemset(tmpVal, 0, sizeof(int) * len);

    hipMalloc((void**)&ExtForce, sizeof(glm::vec3) * numVerts);
    hipMemset(ExtForce, 0, sizeof(float) * numVerts);

    computeSiTSi << < tetBlocks, threadsPerBlock >> > (AIdx, tmpVal, V0, inv_Dm, Tet, wi, numTets, numVerts);
    setMDt_2 << < vertBlocks, threadsPerBlock >> > (AIdx, tmpVal, 48 * numTets, m_1_dt2, numVerts);


    if (useEigen)
    {
        bHost = (float*)malloc(sizeof(float) * ASize);

        int* AIdxHost = (int*)malloc(sizeof(int) * len);
        float* tmpValHost = (float*)malloc(sizeof(float) * len);

        hipMemcpy(AIdxHost, AIdx, sizeof(int) * len, hipMemcpyDeviceToHost);
        hipMemcpy(tmpValHost, tmpVal, sizeof(float) * len, hipMemcpyDeviceToHost);

        std::vector<Eigen::Triplet<float>> A_triplets;

        for (auto i = 0; i < len; ++i)
        {
            A_triplets.push_back({ AIdxHost[i] / ASize, AIdxHost[i] % ASize, tmpValHost[i] });
        }
        Eigen::SparseMatrix<float> A(ASize, ASize);

        A.setFromTriplets(A_triplets.begin(), A_triplets.end());
        cholesky_decomposition_.compute(A);

        free(AIdxHost);
        free(tmpValHost);
    }
    else
    {
        int* newIdx;
        float* newVal;

        hipMalloc((void**)&newIdx, sizeof(int) * len);
        hipMalloc((void**)&newVal, sizeof(float) * len);

        thrust::sort_by_key(thrust::device, AIdx, AIdx + len, tmpVal);


        thrust::pair<int*, float*> newEnd = thrust::reduce_by_key(thrust::device, AIdx, AIdx + len, tmpVal, newIdx, newVal);

        nnzNumber = newEnd.first - newIdx;
        std::cout << nnzNumber << std::endl;

        hipMalloc((void**)&ARow, sizeof(int) * nnzNumber);
        hipMemset(ARow, 0, sizeof(int) * nnzNumber);

        hipMalloc((void**)&ACol, sizeof(int) * nnzNumber);
        hipMemset(ACol, 0, sizeof(int) * nnzNumber);

        hipMalloc((void**)&AVal, sizeof(float) * nnzNumber);
        hipMemcpy(AVal, newVal, sizeof(float) * nnzNumber, hipMemcpyDeviceToDevice);

        int* ARowTmp;
        hipMalloc((void**)&ARowTmp, sizeof(int) * nnzNumber);
        hipMemset(ARowTmp, 0, sizeof(int) * nnzNumber);

        //int threadsPerBlock = 64;
        int blocks = (nnzNumber + threadsPerBlock - 1) / threadsPerBlock;

        initAMatrix << < blocks, threadsPerBlock >> > (newIdx, ARowTmp, ACol, ASize, nnzNumber);

        // transform ARow into csr format
        hipsparseHandle_t handle;
        hipsparseCreate(&handle);
        hipsparseXcoo2csr(handle, ARowTmp, nnzNumber, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

        hipFree(newIdx);
        hipFree(newVal);
        hipFree(ARowTmp);
    }
    hipFree(AIdx);
    hipFree(tmpVal);
}


void SoftBody::PDSolverStep()
{

    float dt = mpSimContext->GetDt();
    float const dtInv = 1.0f / dt;
    float const dt2 = dt * dt;
    float const dt2_m_1 = dt2 / mass;
    float const m_1_dt2 = mass / dt2;


    int threadsPerBlock = 64;
    int vertBlocks = (numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (numTets + threadsPerBlock - 1) / threadsPerBlock;

    glm::vec3 gravity = glm::vec3(0.0f, -mpSimContext->GetGravity(), 0.0f);
    setExtForce << < vertBlocks, threadsPerBlock >> > (ExtForce, gravity, numVerts);
    computeSn << < vertBlocks, threadsPerBlock >> > (sn, dt, dt2_m_1, X, V, ExtForce, numVerts);
    computeM_h2Sn << < vertBlocks, threadsPerBlock >> > (masses, sn, m_1_dt2, numVerts);
    hipsolverSpHandle_t cusolverHandle;
    int singularity = 0;
    hipsparseMatDescr_t descrA;
    if (!useEigen)
    {
        hipsolverSpCreate(&cusolverHandle);
        hipsparseCreateMatDescr(&descrA);
        hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    }

    // 10 is the numVerts of iterations
    for (int i = 0; i < 10; i++)
    {
        hipMemset(b, 0, sizeof(float) * numVerts * 3);
        computeLocal << < tetBlocks, threadsPerBlock >> > (V0, wi, b, inv_Dm, sn, Tet, numTets);
        addM_h2Sn << < vertBlocks, threadsPerBlock >> > (b, masses, numVerts);

        if (useEigen)
        {
            hipMemcpy(bHost, b, sizeof(float) * (numVerts * 3), hipMemcpyDeviceToHost);
            Eigen::VectorXf bh = Eigen::Map<Eigen::VectorXf, Eigen::Unaligned>(bHost, numVerts * 3);
            Eigen::VectorXf res = cholesky_decomposition_.solve(bh);
            hipMemcpy(sn, res.data(), sizeof(float) * (numVerts * 3), hipMemcpyHostToDevice);
        }
        else
        {
            hipsolverSpScsrlsvchol(cusolverHandle, numVerts * 3, nnzNumber, descrA, AVal, ARow, ACol, b, 0.0001f, 0, sn, &singularity);
        }
    }

    updateVelPos << < vertBlocks, threadsPerBlock >> > (sn, dtInv, X, V, numVerts);
}