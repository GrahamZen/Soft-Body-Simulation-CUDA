#include <simulationContext.h>
#include <utilities.cuh>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)

void SoftBody::solverPrepare()
{
    int vertBlocks = (numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (numTets + threadsPerBlock - 1) / threadsPerBlock;
    float dt = mcrpSimContext->GetDt();
    float const m_1_dt2 = attrib.mass / (dt * dt);
    int len = numVerts * 3 + 48 * numTets;
    int ASize = 3 * numVerts;

    hipMalloc((void**)&sn, sizeof(float) * ASize);
    hipMalloc((void**)&b, sizeof(float) * ASize);
    hipMalloc((void**)&masses, sizeof(float) * ASize);

    int* AIdx;
    hipMalloc((void**)&AIdx, sizeof(int) * len);
    hipMemset(AIdx, 0, sizeof(int) * len);

    float* tmpVal;
    hipMalloc((void**)&tmpVal, sizeof(int) * len);
    hipMemset(tmpVal, 0, sizeof(int) * len);

    hipMalloc((void**)&ExtForce, sizeof(glm::vec3) * numVerts);
    hipMemset(ExtForce, 0, sizeof(float) * numVerts);

    computeSiTSi << < tetBlocks, threadsPerBlock >> > (AIdx, tmpVal, V0, inv_Dm, Tet, wi, numTets, numVerts);
    setMDt_2 << < vertBlocks, threadsPerBlock >> > (AIdx, tmpVal, 48 * numTets, m_1_dt2, numVerts);

    bHost = (float*)malloc(sizeof(float) * ASize);

    int* AIdxHost = (int*)malloc(sizeof(int) * len);
    float* tmpValHost = (float*)malloc(sizeof(float) * len);

    hipMemcpy(AIdxHost, AIdx, sizeof(int) * len, hipMemcpyDeviceToHost);
    hipMemcpy(tmpValHost, tmpVal, sizeof(float) * len, hipMemcpyDeviceToHost);

    std::vector<Eigen::Triplet<float>> A_triplets;

    for (auto i = 0; i < len; ++i)
    {
        A_triplets.push_back({ AIdxHost[i] / ASize, AIdxHost[i] % ASize, tmpValHost[i] });
    }
    Eigen::SparseMatrix<float> A(ASize, ASize);

    A.setFromTriplets(A_triplets.begin(), A_triplets.end());
    cholesky_decomposition_.compute(A);

    free(AIdxHost);
    free(tmpValHost);

    int* newIdx;
    float* newVal;

    hipMalloc((void**)&newIdx, sizeof(int) * len);
    hipMalloc((void**)&newVal, sizeof(float) * len);

    thrust::sort_by_key(thrust::device, AIdx, AIdx + len, tmpVal);


    thrust::pair<int*, float*> newEnd = thrust::reduce_by_key(thrust::device, AIdx, AIdx + len, tmpVal, newIdx, newVal);

    int* ARow;
    int* ACol;
    float* AVal;

    nnzNumber = newEnd.first - newIdx;
    std::cout << nnzNumber << std::endl;

    hipMalloc((void**)&ARow, sizeof(int) * nnzNumber);
    hipMemset(ARow, 0, sizeof(int) * nnzNumber);

    hipMalloc((void**)&ACol, sizeof(int) * nnzNumber);
    hipMemset(ACol, 0, sizeof(int) * nnzNumber);

    hipMalloc((void**)&AVal, sizeof(float) * nnzNumber);
    hipMemcpy(AVal, newVal, sizeof(float) * nnzNumber, hipMemcpyDeviceToDevice);

    int* ARowTmp;
    hipMalloc((void**)&ARowTmp, sizeof(int) * nnzNumber);
    hipMemset(ARowTmp, 0, sizeof(int) * nnzNumber);

    int blocks = (nnzNumber + threadsPerBlock - 1) / threadsPerBlock;

    initAMatrix << < blocks, threadsPerBlock >> > (newIdx, ARowTmp, ACol, ASize, nnzNumber);

    // transform ARow into csr format
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseXcoo2csr(handle, ARowTmp, nnzNumber, ASize, ARow, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseMatDescr_t descrA;
    hipsolverSpCreate(&cusolverHandle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    size_t cholSize = 0;
    size_t internalSize = 0;
    cusolverSpCreateCsrcholInfo(&d_info);
    cusolverSpXcsrcholAnalysis(cusolverHandle, ASize, nnzNumber, descrA, ARow, ACol, d_info);
    cusolverSpScsrcholBufferInfo(cusolverHandle, ASize, nnzNumber, descrA, AVal, ARow, ACol, d_info, &internalSize, &cholSize);
    hipMalloc(&buffer_gpu, sizeof(char) * cholSize);
    cusolverSpScsrcholFactor(cusolverHandle, ASize, nnzNumber, descrA, AVal, ARow, ACol, d_info, buffer_gpu);

    hipFree(newIdx);
    hipFree(newVal);
    hipFree(ARowTmp);
    hipFree(ARow);
    hipFree(ACol);
    hipFree(AVal);

    hipFree(AIdx);
    hipFree(tmpVal);
}


void SoftBody::PDSolverStep()
{

    float dt = mcrpSimContext->GetDt();
    float const dtInv = 1.0f / dt;
    float const dt2 = dt * dt;
    float const dt2_m_1 = dt2 / attrib.mass;
    float const m_1_dt2 = attrib.mass / dt2;


    int vertBlocks = (numVerts + threadsPerBlock - 1) / threadsPerBlock;
    int tetBlocks = (numTets + threadsPerBlock - 1) / threadsPerBlock;

    glm::vec3 gravity = glm::vec3(0.0f, -mcrpSimContext->GetGravity(), 0.0f);
    setExtForce << < vertBlocks, threadsPerBlock >> > (ExtForce, gravity, numVerts);
    computeSn << < vertBlocks, threadsPerBlock >> > (sn, dt, dt2_m_1, X, V, ExtForce, numVerts);
    computeM_h2Sn << < vertBlocks, threadsPerBlock >> > (masses, sn, m_1_dt2, numVerts);

    // 10 is the numVerts of iterations
    for (int i = 0; i < 10; i++)
    {
        hipMemset(b, 0, sizeof(float) * numVerts * 3);
        computeLocal << < tetBlocks, threadsPerBlock >> > (V0, wi, b, inv_Dm, sn, Tet, numTets);
        addM_h2Sn << < vertBlocks, threadsPerBlock >> > (b, masses, numVerts);

        if (mcrpSimContext->IsEigenGlobalSolver())
        {
            hipMemcpy(bHost, b, sizeof(float) * (numVerts * 3), hipMemcpyDeviceToHost);
            Eigen::VectorXf bh = Eigen::Map<Eigen::VectorXf, Eigen::Unaligned>(bHost, numVerts * 3);
            Eigen::VectorXf res = cholesky_decomposition_.solve(bh);
            hipMemcpy(sn, res.data(), sizeof(float) * (numVerts * 3), hipMemcpyHostToDevice);
        }
        else
        {
            cusolverSpScsrcholSolve(cusolverHandle, numVerts * 3, b, sn, d_info, buffer_gpu);
        }
    }

    updateVelPos << < vertBlocks, threadsPerBlock >> > (sn, dtInv, XTilt, V, numVerts);
}