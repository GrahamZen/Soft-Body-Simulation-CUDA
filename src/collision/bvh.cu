#include "hip/hip_runtime.h"
#pragma once

#include <collision/bvh.h>
#include <utilities.cuh>
#include <collision/bvh.cuh>
#include <simulation/simulationContext.h>
#include <hip/hip_cooperative_groups.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

template<typename Scalar>
__device__ void buildBBox(BVHNode<Scalar>& curr, const BVHNode<Scalar>& left, const BVHNode<Scalar>& right)
{
    glm::tvec3<Scalar> newMin;
    glm::tvec3<Scalar> newMax;
    newMin.x = glm::min(left.bbox.min.x, right.bbox.min.x);
    newMax.x = glm::max(left.bbox.max.x, right.bbox.max.x);
    newMin.y = glm::min(left.bbox.min.y, right.bbox.min.y);
    newMax.y = glm::max(left.bbox.max.y, right.bbox.max.y);
    newMin.z = glm::min(left.bbox.min.z, right.bbox.min.z);
    newMax.z = glm::max(left.bbox.max.z, right.bbox.max.z);

    curr.bbox = AABB<Scalar>{ newMin, newMax };
    curr.isLeaf = 0;
}

template<typename Scalar>
__global__ void buildBBoxesSerial(int leafCount, BVHNode<Scalar>* nodes, BVH<Scalar>::ReadyFlagType* ready) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    if (ind >= leafCount - 1)return;
    BVHNode<Scalar> node = nodes[ind];
    if (ready[ind] != 0)
        return;
    if (ready[node.leftIndex] != 0 && ready[node.rightIndex] != 0)
    {
        buildBBox(nodes[ind], nodes[node.leftIndex], nodes[node.rightIndex]);
        ready[ind] = 1;
    }
}

namespace cg = cooperative_groups;

template<typename Scalar>
__global__ void buildBBoxesCG(int leafCount, BVHNode<Scalar>* nodes, BVH<Scalar>::ReadyFlagType* ready) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    cg::grid_group grid = cg::this_grid();

    if (ind >= leafCount - 1)return;
    bool done = false;
    while (!done) {
        BVHNode<Scalar> node = nodes[ind];
        if (ready[ind] != 0) {}
        else if (ready[node.leftIndex] != 0 && ready[node.rightIndex] != 0)
        {
            buildBBox(nodes[ind], nodes[node.leftIndex], nodes[node.rightIndex]);
            ready[ind] = 1;
        }
        cg::sync(grid);
        done = ready[0] == 1;
        cg::sync(grid);
    }
}

template<typename Scalar>
__global__ void buildBBoxesAtomic(int leafCount, BVHNode<Scalar>* nodes, BVH<Scalar>::ReadyFlagType* ready) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    if (ind >= leafCount - 1) return;
    BVHNode<Scalar> node = nodes[ind];

    while (true) {
        auto leftReady = atomicCAS(&ready[node.leftIndex], 0, 0);
        auto rightReady = atomicCAS(&ready[node.rightIndex], 0, 0);

        if (leftReady != 0 && rightReady != 0) {
            buildBBox(nodes[ind], nodes[node.leftIndex], nodes[node.rightIndex]);
            ready[ind] = 1;
            break;
        }
        __threadfence();
    }
}

template<typename Scalar>
void BVH<Scalar>::Init(int _numTets, int _numVerts, int maxThreads)
{
    numTets = _numTets;
    int numVerts = _numVerts;
    int numNodes = numTets * 2 - 1;
    hipMalloc(&dev_BVHNodes, numNodes * sizeof(BVHNode<Scalar>));
    hipMalloc((void**)&dev_tI, numVerts * sizeof(Scalar));
    hipMemset(dev_tI, 0, numVerts * sizeof(Scalar));
    hipMalloc((void**)&dev_indicesToReport, numVerts * sizeof(int));
    hipMemset(dev_indicesToReport, -1, numVerts * sizeof(int));
    hipMalloc(&dev_mortonCodes, numTets * sizeof(unsigned int));
    hipMalloc(&dev_ready, numNodes * sizeof(ReadyFlagType));
    createBVH(numNodes);
    hipMemset(dev_mortonCodes, 0, numTets * sizeof(unsigned int));
    hipMemset(dev_ready, 0, (numTets - 1) * sizeof(ReadyFlagType));
    hipMemset(&dev_ready[numTets - 1], 1, numTets * sizeof(ReadyFlagType));
    int minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &suggestedBlocksize, buildBBoxesCG<Scalar>, 0, 0);

    if (numTets < maxThreads) {
        std::cout << "Using cooperative group." << std::endl;
        isBuildBBCG = true;
    }
    else {
        std::cout << "Not using cooperative group." << std::endl;
    }
    numblocksTets = (numTets + threadsPerBlock - 1) / threadsPerBlock;
    numblocksVerts = (numVerts + threadsPerBlock - 1) / threadsPerBlock;
    suggestedCGNumblocks = (numTets + suggestedBlocksize - 1) / suggestedBlocksize;
}

template<typename Scalar>
void BVH<Scalar>::BuildBBoxes(BuildType buildType) {
    if (buildType == BuildType::Cooperative && isBuildBBCG) {
        void* args[] = { &numTets, &dev_BVHNodes, &dev_ready };
        hipError_t error = hipLaunchCooperativeKernel((void*)buildBBoxesCG<Scalar>, suggestedCGNumblocks, suggestedBlocksize, args);
        if (error != hipSuccess) {
            std::cerr << "hipLaunchCooperativeKernel failed: " << hipGetErrorString(error) << std::endl;
        }
    }
    else if (buildType == BuildType::Atomic) {
        buildBBoxesAtomic<Scalar> << < numblocksTets, threadsPerBlock >> > (numTets, dev_BVHNodes, dev_ready);
    }
    else if (buildType == BuildType::Serial) {
        ReadyFlagType treeBuild = 0;
        while (treeBuild == 0) {
            buildBBoxesSerial<Scalar> << < numblocksTets, threadsPerBlock >> > (numTets, dev_BVHNodes, dev_ready);
            hipMemcpy(&treeBuild, dev_ready, sizeof(ReadyFlagType), hipMemcpyDeviceToHost);
        }
    }
}

template<typename Scalar>
BVH<Scalar>::BVH<Scalar>(const int _threadsPerBlock) :
    threadsPerBlock(_threadsPerBlock) {}

template<typename Scalar>
BVH<Scalar>::~BVH<Scalar>()
{
    hipFree(dev_BVHNodes);
    hipFree(dev_tI);
    hipFree(dev_indicesToReport);

    hipFree(dev_ready);
    hipFree(dev_mortonCodes);
}

template<typename Scalar>
void BVH<Scalar>::PrepareRenderData()
{
    glm::vec3* pos;
    Wireframe::MapDevicePosPtr(&pos);
    int numNodes = numTets * 2 - 1;
    dim3 numThreadsPerBlock(numNodes / threadsPerBlock + 1);
    populateBVHNodeAABBPos << <numThreadsPerBlock, threadsPerBlock >> > (dev_BVHNodes, pos, numNodes);
    Wireframe::UnMapDevicePtr();
}

template<typename Scalar>
const BVHNode<Scalar>* BVH<Scalar>::GetBVHNodes() const
{
    return dev_BVHNodes;
}

template<typename Scalar>
void CollisionDetection<Scalar>::DetectCollision(Scalar* tI, glm::vec3* nors)
{
    thrust::device_ptr<Scalar> dev_ptr(tI);
    thrust::fill(dev_ptr, dev_ptr + numVerts, 1.0f);
    if (BroadPhase()) {
        PrepareRenderData();
        NarrowPhase(tI, nors);
    }
}

template<typename Scalar>
void CollisionDetection<Scalar>::SetBuildType(typename BVH<Scalar>::BuildType _buildType)
{
    buildType = _buildType;
}

template<typename Scalar>
typename BVH<Scalar>::BuildType CollisionDetection<Scalar>::GetBuildType()
{
    return buildType;
}

template class BVH<float>;
template class BVH<double>;

template class CollisionDetection<float>;
template class CollisionDetection<double>;