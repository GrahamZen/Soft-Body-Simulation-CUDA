#include "hip/hip_runtime.h"
#pragma once

#include <glm/glm.hpp>
#include <bvh.cuh>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <intersections.h>
#include <hip/hip_runtime.h>
#include <utilities.cuh>

__device__ AABB computeTetTrajBBox(const glm::vec3& v0, const glm::vec3& v1, const glm::vec3& v2, const glm::vec3& v3,
    const glm::vec3& v4, const glm::vec3& v5, const glm::vec3& v6, const glm::vec3& v7)
{
    glm::vec3 min, max;
    min.x = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.x, v1.x), v2.x), v3.x), v4.x), v5.x), v6.x), v7.x);
    min.y = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.y, v1.y), v2.y), v3.y), v4.y), v5.y), v6.y), v7.y);
    min.z = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.z, v1.z), v2.z), v3.z), v4.z), v5.z), v6.z), v7.z);
    max.x = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.x, v1.x), v2.x), v3.x), v4.x), v5.x), v6.x), v7.x);
    max.y = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.y, v1.y), v2.y), v3.y), v4.y), v5.y), v6.y), v7.y);
    max.z = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.z, v1.z), v2.z), v3.z), v4.z), v5.z), v6.z), v7.z);

    return AABB{ min, max };
}

struct MinOp {
    __host__ __device__
        glm::vec3 operator()(const glm::vec3& a, const glm::vec3& b) const {
        return glm::min(a, b);
    }
};

struct MaxOp {
    __host__ __device__
        glm::vec3 operator()(const glm::vec3& a, const glm::vec3& b) const {
        return glm::max(a, b);
    }
};

AABB computeBoundingBox(const thrust::device_ptr<glm::vec3>& begin, const thrust::device_ptr<glm::vec3>& end) {
    glm::vec3 min = thrust::reduce(begin, end, glm::vec3(FLT_MAX), MinOp());
    glm::vec3 max = thrust::reduce(begin, end, glm::vec3(-FLT_MAX), MaxOp());

    return AABB{ min, max };
}

AABB AABB::expand(const AABB& aabb)const {
    return AABB{
        glm::min(min, aabb.min),
        glm::max(max, aabb.max)
    };
}

bool isCollision(const glm::vec3& v, const AABB& box, float threshold = EPSILON) {
    glm::vec3 nearestPoint;
    nearestPoint.x = std::max(box.min.x, std::min(v.x, box.max.x));
    nearestPoint.y = std::max(box.min.y, std::min(v.y, box.max.y));
    nearestPoint.z = std::max(box.min.z, std::min(v.z, box.max.z));
    glm::vec3 diff = v - nearestPoint;
    float distanceSquared = glm::dot(diff, diff);
    return distanceSquared <= threshold;
}

__device__ float traverseTree(const BVHNode* nodes, const glm::vec3* Xs, const glm::vec3* XTilts, glm::vec3 X0, glm::vec3 XTilt, int& hitTetId)
{
    // record the closest intersection
    float closest = FLT_MAX;

    int bvhStart = 0;
    int stack[64];
    int stackPtr = 0;
    int bvhPtr = bvhStart;
    stack[stackPtr++] = bvhStart;

    while (stackPtr)
    {
        bvhPtr = stack[--stackPtr];
        BVHNode currentNode = nodes[bvhPtr];
        // all the left and right indexes are 0
        BVHNode leftChild = nodes[currentNode.leftIndex + bvhStart];
        BVHNode rightChild = nodes[currentNode.rightIndex + bvhStart];

        bool hitLeft = edgeBboxIntersectionTest(X0, XTilt, leftChild.bbox);
        bool hitRight = edgeBboxIntersectionTest(X0, XTilt, rightChild.bbox);
        if (hitLeft)
        {
            // check triangle intersection
            if (leftChild.isLeaf == 1)
            {
                float distance = tetrahedronTrajIntersectionTest(X0, XTilt, Xs, XTilts, leftChild.TetrahedronIndex);
                if (distance < closest)
                {
                    hitTetId = leftChild.TetrahedronIndex;
                    closest = distance;
                }
            }
            else
            {
                stack[stackPtr++] = currentNode.leftIndex + bvhStart;
            }

        }
        if (hitRight)
        {
            // check triangle intersection
            if (rightChild.isLeaf == 1)
            {
                float distance = tetrahedronTrajIntersectionTest(X0, XTilt, Xs, XTilts, rightChild.TetrahedronIndex);
                if (distance < closest)
                {
                    hitTetId = rightChild.TetrahedronIndex;
                    closest = distance;
                }
            }
            else
            {
                stack[stackPtr++] = currentNode.rightIndex + bvhStart;
            }

        }
    }
    return closest;
}


__global__ void detectCollisionCandidatesKern(int numVerts, const BVHNode* nodes, const GLuint* tetIds, const glm::vec3* Xs, const glm::vec3* XTilts, int* indicesToReport, float* tI)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numVerts)
    {
        int hitTetId = -1;
        const glm::vec3& X = Xs[index];
        const glm::vec3& XTilt = XTilts[index];
        int tetId = tetIds[index];
        float distance = traverseTree(nodes, Xs, XTilts, X, XTilt, hitTetId);
        if (distance != -1)
        {
            indicesToReport[index] = hitTetId;
            tI[index] = distance;
        }
        else {
            tI[index] = 1;
            indicesToReport[index] = -1;
        }
    }
}

float* BVH::DetectCollisionCandidates(const GLuint* Tet, const glm::vec3* Xs, const glm::vec3* XTilts, const GLuint* TetId) const
{
    int blockSize1d = 128;
    dim3 numblocks = (numVerts + blockSize1d - 1) / blockSize1d;
    detectCollisionCandidatesKern << <numblocks, blockSize1d >> > (numVerts, dev_BVHNodes, TetId, Xs, XTilts, dev_indicesToReport, dev_tI);
    return dev_tI;
}

void BVH::PrepareRenderData()
{
    glm::vec3* pos;
    Wireframe::mapDevicePosPtr(&pos);
    dim3 numThreadsPerBlock(numNodes / threadsPerBlock + 1);
    populateBVHNodeAABBPos << <numThreadsPerBlock, threadsPerBlock >> > (dev_BVHNodes, pos, numNodes);
    Wireframe::unMapDevicePtr();
}

BVH::BVH(int& _threadsPerBlock) : threadsPerBlock(_threadsPerBlock) {}

BVH::~BVH()
{
    hipFree(dev_BVHNodes);
    hipFree(dev_tI);
    hipFree(dev_indicesToReport);

    hipFree(dev_ready);
    hipFree(dev_mortonCodes);
}

void BVH::Init(int _numTets, int _numVerts)
{
    numTets = _numTets;
    numVerts = _numVerts;
    numNodes = numTets * 2 - 1;
    hipMalloc(&dev_BVHNodes, numNodes * sizeof(BVHNode));
    hipMalloc((void**)&dev_tI, numVerts * sizeof(float));
    hipMemset(dev_tI, 0, numVerts * sizeof(float));
    hipMalloc((void**)&dev_indicesToReport, numVerts * sizeof(int));
    hipMemset(dev_indicesToReport, -1, numVerts * sizeof(int));
    hipMalloc(&dev_mortonCodes, numTets * sizeof(unsigned int));
    hipMalloc(&dev_ready, numNodes * sizeof(unsigned char));
    createBVH(numNodes);
}