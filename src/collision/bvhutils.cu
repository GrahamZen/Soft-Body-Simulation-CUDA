#include "hip/hip_runtime.h"
#pragma once

#include <glm/glm.hpp>
#include <bvh.cuh>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <utilities.cuh>

//input the aabb box of a Tetrahedron
//generate a 30-bit morton code
__device__ unsigned int genMortonCode(AABB bbox, glm::vec3 geoMin, glm::vec3 geoMax)
{
    float x = (bbox.min.x + bbox.max.x) * 0.5f;
    float y = (bbox.min.y + bbox.max.y) * 0.5f;
    float z = (bbox.min.y + bbox.max.y) * 0.5f;
    float normalizedX = (x - geoMin.x) / (geoMax.x - geoMin.x);
    float normalizedY = (y - geoMin.y) / (geoMax.y - geoMin.y);
    float normalizedZ = (z - geoMin.z) / (geoMax.z - geoMin.z);

    normalizedX = glm::min(glm::max(normalizedX * 1024.0f, 0.0f), 1023.0f);
    normalizedY = glm::min(glm::max(normalizedY * 1024.0f, 0.0f), 1023.0f);
    normalizedZ = glm::min(glm::max(normalizedZ * 1024.0f, 0.0f), 1023.0f);

    unsigned int xx = expandBits((unsigned int)normalizedX);
    unsigned int yy = expandBits((unsigned int)normalizedY);
    unsigned int zz = expandBits((unsigned int)normalizedZ);

    return xx * 4 + yy * 2 + zz;
}


__device__ unsigned long long expandMorton(int index, unsigned int mortonCode)
{
    unsigned long long exMortonCode = mortonCode;
    exMortonCode <<= 32;
    exMortonCode += index;
    return exMortonCode;
}

/**
* please sort the morton code first then get split pairs
thrust::stable_sort_by_key(mortonCodes, mortonCodes + TetrahedronCount, TetrahedronIndex);*/

//total input is a 30 x N matrix
//currentIndex is between 0 - N-1
//the input morton codes should be in the reduced form, no same elements are expected to appear twice!
__device__ int getSplit(unsigned int* mortonCodes, unsigned int currIndex, int nextIndex, unsigned int bound)
{
    if (nextIndex < 0 || nextIndex >= bound)
        return -1;
    //NOTE: if use small size model, this step can be skipped
    // just to ensure the morton codes are unique!
    //unsigned int mask = mortonCodes[currIndex] ^ mortonCodes[nextIndex];
    unsigned long long mask = expandMorton(currIndex, mortonCodes[currIndex]) ^ expandMorton(nextIndex, mortonCodes[nextIndex]);
    // __clzll gives the number of consecutive zero bits in that number
    // this gives us the index of the most significant bit between the two numbers
    int commonPrefix = __clzll(mask);
    return commonPrefix;
}

__device__ void buildBBox(BVHNode& curr, const BVHNode& left, const BVHNode& right)
{
    glm::vec3 newMin;
    glm::vec3 newMax;
    newMin.x = glm::min(left.bbox.min.x, right.bbox.min.x);
    newMax.x = glm::max(left.bbox.max.x, right.bbox.max.x);
    newMin.y = glm::min(left.bbox.min.y, right.bbox.min.y);
    newMax.y = glm::max(left.bbox.max.y, right.bbox.max.y);
    newMin.z = glm::min(left.bbox.min.z, right.bbox.min.z);
    newMax.z = glm::max(left.bbox.max.z, right.bbox.max.z);

    curr.bbox = AABB{ newMin, newMax };
    curr.isLeaf = 0;
}

// build the bounding box and morton code for each SoftBody
__global__ void buildLeafMorton(int startIndex, int numTri, float minX, float minY, float minZ,
    float maxX, float maxY, float maxZ, const GLuint* tet, const glm::vec3* X, const glm::vec3* XTilt, BVHNode* leafNodes,
    unsigned int* mortonCodes)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind < numTri)
    {
        int leafPos = ind + numTri - 1;
        leafNodes[leafPos].bbox = computeTetTrajBBox(X[tet[ind * 4]], X[tet[ind * 4 + 1]], X[tet[ind * 4 + 2]], X[tet[ind * 4 + 3]],
            XTilt[tet[ind * 4]], XTilt[tet[ind * 4 + 1]], XTilt[tet[ind * 4 + 2]], XTilt[tet[ind * 4 + 3]]);
        leafNodes[leafPos].isLeaf = 1;
        leafNodes[leafPos].leftIndex = -1;
        leafNodes[leafPos].rightIndex = -1;
        leafNodes[leafPos].TetrahedronIndex = ind;
        mortonCodes[ind + startIndex] = genMortonCode(leafNodes[ind + numTri - 1].bbox, glm::vec3(minX, minY, minZ), glm::vec3(maxX, maxY, maxZ));
    }
}


//input the unique morton code
//codeCount is the size of the unique morton code
//splitList is 30 x N list
// the size of unique morton is less than 2^30 : [1, 2^30]
__global__ void buildSplitList(int codeCount, unsigned int* uniqueMorton, BVHNode* nodes)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind < codeCount - 1)
    {
        int sign = getSign(getSplit(uniqueMorton, ind, ind + 1, codeCount) - getSplit(uniqueMorton, ind, ind - 1, codeCount));
        int dMin = getSplit(uniqueMorton, ind, ind - sign, codeCount);
        int lenMax = 2;
        int k = getSplit(uniqueMorton, ind, ind + lenMax * sign, codeCount);
        while (k > dMin)
        {
            lenMax *= 2;
            k = getSplit(uniqueMorton, ind, ind + lenMax * sign, codeCount);
        }

        int len = 0;
        int last = lenMax >> 1;
        while (last > 0)
        {
            int tmp = ind + (len + last) * sign;
            int diff = getSplit(uniqueMorton, ind, tmp, codeCount);
            if (diff > dMin)
            {
                len = len + last;
            }
            last >>= 1;
        }
        //last in range
        int j = ind + len * sign;

        int currRange = getSplit(uniqueMorton, ind, j, codeCount);
        int split = 0;
        do {
            len = (len + 1) >> 1;
            if (getSplit(uniqueMorton, ind, ind + (split + len) * sign, codeCount) > currRange)
            {
                split += len;
            }
        } while (len > 1);

        int tmp = ind + split * sign + glm::min(sign, 0);

        if (glm::min(ind, j) == tmp)
        {
            //leaf node
            // the number of internal nodes is N - 1
            nodes[ind].leftIndex = tmp + codeCount - 1;
            nodes[tmp + codeCount - 1].parent = ind;
        }
        else
        {
            // internal node
            nodes[ind].leftIndex = tmp;
            nodes[tmp].parent = ind;
        }
        if (glm::max(ind, j) == tmp + 1)
        {
            nodes[ind].rightIndex = tmp + codeCount;
            nodes[tmp + codeCount].parent = ind;
        }
        else
        {
            nodes[ind].rightIndex = tmp + 1;
            nodes[tmp + 1].parent = ind;
        }
    }

}

namespace cg = cooperative_groups;

__global__ void buildBBoxesCooperativeKern(int leafCount, BVHNode* nodes, int* ready) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    cg::grid_group grid = cg::this_grid();

    if (ind >= leafCount - 1)return;
    for (int i = 0; i < leafCount; i++) {
        cg::sync(grid);
        BVHNode node = nodes[ind];
        if (ready[ind] != 0)
            continue;
        if (ready[node.leftIndex] != 0 && ready[node.rightIndex] != 0)
        {
            buildBBox(nodes[ind], nodes[node.leftIndex], nodes[node.rightIndex]);
            ready[ind] = 1;
        }
    }
}

__global__ void buildBBoxesSerial(int leafCount, BVHNode* nodes, int* ready) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    if (ind >= leafCount - 1)return;
    BVHNode node = nodes[ind];
    if (ready[ind] != 0)
        return;
    if (ready[node.leftIndex] != 0 && ready[node.rightIndex] != 0)
    {
        buildBBox(nodes[ind], nodes[node.leftIndex], nodes[node.rightIndex]);
        ready[ind] = 1;
    }
}

__global__ void buildBBoxes(int leafCount, BVHNode* nodes, int* ready) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    if (ind >= leafCount - 1) return;

    BVHNode node = nodes[ind];

    int leftReady = 0;
    int rightReady = 0;
    if (ready[ind] != 0)
        return;
    while (true) {
        leftReady = atomicAdd(&ready[node.leftIndex], 0);
        rightReady = atomicAdd(&ready[node.rightIndex], 0);

        if (leftReady != 0 && rightReady != 0) break;
    }
    node = nodes[ind];
    buildBBox(nodes[ind], nodes[node.leftIndex], nodes[node.rightIndex]);

    atomicExch(&ready[ind], 1);
    __threadfence();
    __threadfence_block();
}


void BVH::BuildBVHTree(const AABB& ctxAABB, int numTets, const glm::vec3* X, const glm::vec3* XTilt, const GLuint* tets)
{
    hipMemset(dev_BVHNodes, 0, (numTets * 2 - 1) * sizeof(BVHNode));
    hipMemset(dev_mortonCodes, 0, numTets * sizeof(unsigned int));
    hipMemset(dev_ready, 0, numTets * sizeof(int));
    hipMemset(&dev_ready[numTets - 1], 1, numTets * sizeof(int));

    dim3 numblocks = (numTets + threadsPerBlock - 1) / threadsPerBlock;
    buildLeafMorton << <numblocks, threadsPerBlock >> > (0, numTets, ctxAABB.min.x, ctxAABB.min.y, ctxAABB.min.z, ctxAABB.max.x, ctxAABB.max.y, ctxAABB.max.z,
        tets, X, XTilt, dev_BVHNodes, dev_mortonCodes);

    thrust::stable_sort_by_key(thrust::device, dev_mortonCodes, dev_mortonCodes + numTets, dev_BVHNodes + numTets - 1);

    buildSplitList << <numblocks, threadsPerBlock >> > (numTets, dev_mortonCodes, dev_BVHNodes);

    //can use atomic operation for further optimization
    void* args[] = { &numTets, &dev_BVHNodes, &dev_ready };
    switch (buildMethod)
    {
    case BVH::BuildMethodType::SERIAL:
        for (int i = 0; i < numTets; i++) {
            buildBBoxesSerial << < numblocks, threadsPerBlock >> > (numTets, dev_BVHNodes, dev_ready);
        }
        break;
    case BVH::BuildMethodType::PARALLEL:
        buildBBoxes << < numblocks, threadsPerBlock >> > (numTets, dev_BVHNodes, dev_ready);
        break;
    case BVH::BuildMethodType::COOPERATIVE_GROUP:
        hipLaunchCooperativeKernel((void*)buildBBoxesCooperativeKern, numblocks, threadsPerBlock, args);
        break;
    default:
        throw std::runtime_error("Invalid build method for BVH.");
        break;
    }
}