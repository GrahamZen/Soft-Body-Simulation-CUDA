#include "hip/hip_runtime.h"
#pragma once

#include <glm/glm.hpp>
#include <glm/gtx/intersect.hpp>
#include <glm/gtc/constants.hpp>

#include <intersections.h>
#include <utilities.h>
#include <glm/gtx/norm.hpp>

/**
 * Handy-dandy hash function that provides seeds for random number generation.
 */
__host__ __device__ inline unsigned int utilhash(unsigned int a) {
    a = (a + 0x7ed55d16) + (a << 12);
    a = (a ^ 0xc761c23c) ^ (a >> 19);
    a = (a + 0x165667b1) + (a << 5);
    a = (a + 0xd3a2646c) ^ (a << 9);
    a = (a + 0xfd7046c5) + (a << 3);
    a = (a ^ 0xb55a4f09) ^ (a >> 16);
    return a;
}

/**
 * Multiplies a mat4 and a vec4 and returns a vec3 clipped from the vec4.
 */
__host__ __device__ glmVec3 multiplyMV(glmMat4 m, glmVec4 v) {
    return glmVec3(m * v);
}

template<typename T>
__host__ __device__ void swap(T& lhs, T& rhs) {
    T tmp = lhs;
    lhs = rhs;
    rhs = tmp;
}

__host__ __device__ dataType signed_vf_distance(const glmVec3& x,
    const glmVec3& y0, const glmVec3& y1, const glmVec3& y2,
    glmVec3* n, glmVec4& w) {
    *n = cross(normalize(y1 - y0), normalize(y2 - y0));
    if (length2(*n) < 1e-6)
        return FLT_MAX;
    *n = normalize(*n);
    dataType h = dot(x - y0, *n);
    dataType b0 = stp(y1 - x, y2 - x, *n),
        b1 = stp(y2 - x, y0 - x, *n),
        b2 = stp(y0 - x, y1 - x, *n);
    w[0] = 1;
    w[1] = -b0 / (b0 + b1 + b2);
    w[2] = -b1 / (b0 + b1 + b2);
    w[3] = -b2 / (b0 + b1 + b2);
    return h;
}

__host__ __device__ dataType signed_ve_distance(const glmVec3& x, const glmVec3& y0, const glmVec3& y1,
    glmVec3* n, dataType* w) {
    glmVec3 e = y1 - y0;
    dataType d = dot(x - y0, e) / length2(e);
    if (d < 0 || d > 1.0)
        return FLT_MAX;
    if (w) {
        w[0] = 1;
        w[1] = -(1.0 - d);
        w[2] = -d;
        w[3] = 0;
    }
    glmVec3 dist = x - (y0 + d * e);
    dataType l = length(dist);
    if (n && fabs(l) > 1e-16) *n = dist / l;
    return l;
}

__host__ __device__ dataType signed_ee_distance(const glmVec3& x0, const glmVec3& x1,
    const glmVec3& y0, const glmVec3& y1,
    glmVec3* n, dataType* w) {
    glmVec3 _n; if (!n) n = &_n;
    dataType _w[4]; if (!w) w = _w;
    *n = cross(normalize(x1 - x0), normalize(y1 - y0));
    if (glm::length2(*n) < 1e-8) {
        // special case: parallel lines
        glmVec3 e0 = normalize(x1 - x0), e1 = normalize(y1 - y0);

        dataType p0min = dot(x0, e0), p0max = dot(x1, e0), p1min = dot(y0, e0), p1max = dot(y1, e0);
        if (p1max < p1min) swap(p1max, p1min);

        dataType a = glm::max(p0min, p1min), b = glm::min(p0max, p1max), c = 0.5 * (a + b);
        if (a > b) return FLT_MAX;

        glmVec3 d = (y0 - x0) - dot(y0 - x0, e0) * e0;

        if (n) *n = normalize(-d);
        if (w) {
            w[1] = (c - dot(x0, e0)) / glm::length(x1 - x0);
            w[0] = 1.0 - w[1];
            w[3] = -(dot(e0, e1) * c - dot(y0, e1)) / glm::length(y1 - y0);
            w[2] = -1.0 - w[3];
        }
        return glm::length(d);
    }
    *n = normalize(*n);
    dataType h = dot(x0 - y0, *n);
    dataType a0 = stp(y1 - x1, y0 - x1, *n), a1 = stp(y0 - x0, y1 - x0, *n),
        b0 = stp(x0 - y1, x1 - y1, *n), b1 = stp(x1 - y0, x0 - y0, *n);
    w[0] = a0 / (a0 + a1);
    w[1] = a1 / (a0 + a1);
    w[2] = -b0 / (b0 + b1);
    w[3] = -b1 / (b0 + b1);
    return h;
}


__host__ __device__ bool edgeBboxIntersectionTest(const glmVec3& X0, const glmVec3& XTilt, const AABB& bbox) {
    const dataType eps = glm::epsilon<dataType>();
    glmVec3 d = XTilt - X0;
    glmVec3 ood = 1.0 / d;
    dataType tmin = 0.0;
    dataType tmax = 1.0;
#pragma unroll
    for (int i = 0; i < 3; i++) {
        if (glm::abs<dataType>(d[i]) < eps) {
            if (X0[i] < bbox.min[i] || X0[i] > bbox.max[i]) return false;
        }
        else {
            dataType t1 = (bbox.min[i] - X0[i]) * ood[i];
            dataType t2 = (bbox.max[i] - X0[i]) * ood[i];
            if (t1 > t2) swap(t1, t2);

            tmin = glm::max(tmin, t1);
            tmax = glm::min(tmax, t2);

            if (tmin > tmax) return false;
        }
    }
    return true;
}

__host__ __device__ bool edgeBboxIntersectionTest(const glmVec3& X0, const glmVec3& XTilt, const AABB& bbox, dataType& tmin, dataType& tmax) {
    const dataType eps = glm::epsilon<dataType>();
    glmVec3 d = XTilt - X0;
    glmVec3 ood = 1.0 / d;
    tmin = 0.0;
    tmax = 1.0;

#pragma unroll
    for (int i = 0; i < 3; i++) {
        if (glm::abs<dataType>(d[i]) < eps) {
            if (X0[i] < bbox.min[i] || X0[i] > bbox.max[i]) return false;
        }
        else {
            dataType t1 = (bbox.min[i] - X0[i]) * ood[i];
            dataType t2 = (bbox.max[i] - X0[i]) * ood[i];

            if (t1 > t2) swap(t1, t2);

            tmin = glm::max(tmin, t1);
            tmax = glm::min(tmax, t2);

            if (tmin > tmax) return false;
        }
    }
    return true;
}

template<typename T>
__host__ __device__ int solveQuadratic(T a, T b, T c, T* x) {
    // http://en.wikipedia.org/wiki/Quadratic_formula#dataTypeing_point_implementation
    T d = b * b - 4 * a * c;
    if (d < 0) {
        x[0] = -b / (2 * a);
        return 0;
    }
    T q = -(b + glm::sign(b) * sqrt(d)) / 2;
    int i = 0;
    if (abs(a) > 1e-12 * abs(q))
        x[i++] = q / a;
    if (abs(q) > 1e-12 * abs(c))
        x[i++] = c / q;
    if (i == 2 && x[0] > x[1]) {
        T tmp = x[0];
        x[0] = x[1];
        x[1] = tmp;
    }
    return i;
}

template<typename T>
__host__ __device__ int solveCubic(T a, T b, T c, T d, T* x) {
    T xc[2];
    int ncrit = solveQuadratic(3 * a, 2 * b, c, xc);
    if (ncrit == 0) {
        x[0] = newtonsMethod(a, b, c, d, xc[0], 0);
        return 1;
    }
    else if (ncrit == 1) {// cubic is actually quadratic
        return solveQuadratic(b, c, d, x);
    }
    else {
        T yc[2] = { d + xc[0] * (c + xc[0] * (b + xc[0] * a)),
                        d + xc[1] * (c + xc[1] * (b + xc[1] * a)) };
        int i = 0;
        if (yc[0] * a >= 0)
            x[i++] = newtonsMethod(a, b, c, d, xc[0], -1);
        if (yc[0] * yc[1] <= 0) {
            int closer = abs(yc[0]) < abs(yc[1]) ? 0 : 1;
            x[i++] = newtonsMethod(a, b, c, d, xc[closer], closer == 0 ? 1 : -1);
        }
        if (yc[1] * a <= 0)
            x[i++] = newtonsMethod(a, b, c, d, xc[1], 1);
        return i;
    }
}

template<typename T>
__host__ __device__ T newtonsMethod(T a, T b, T c, T d, T x0,
    int init_dir) {
    if (init_dir != 0) {
        // quadratic approximation around x0, assuming y' = 0
        T y0 = d + x0 * (c + x0 * (b + x0 * a)),
            ddy0 = 2 * b + x0 * (6 * a);
        x0 += init_dir * sqrt(abs(2 * y0 / ddy0));
    }
    for (int iter = 0; iter < 100; iter++) {
        T y = d + x0 * (c + x0 * (b + x0 * a));
        T dy = c + x0 * (2 * b + x0 * 3 * a);
        if (dy == 0)
            return x0;
        T x1 = x0 - y / dy;
        if (abs(x0 - x1) < 1e-6)
            return x0;
        x0 = x1;
    }
    return x0;
}

template<typename T>
__host__ __device__ T solveCubicRange01(T a, T b, T c, T d, T* x) {
    T roots[3];
    int j = 0;
    int numRoots = solveCubic(a, b, c, d, roots);
    for (int i = 0; i < numRoots; i++) {
        if (roots[i] >= 0 && roots[i] <= 1) {
            x[j++] = roots[i];
        }
    }
    return j;
}
template dataType solveCubicRange01<dataType>(dataType a, dataType b, dataType c, dataType d, dataType* x);

__host__ __device__ dataType stp(const glmVec3& u, const glmVec3& v, const glmVec3& w) { return glm::dot(u, glm::cross(v, w)); }

__host__ __device__ dataType ccdTriangleIntersectionTest(const glmVec3& x0, const glmVec3& v0,
    const glmVec3& x1, const glmVec3& x2, const glmVec3& x3, const glmVec3& v1, const glmVec3& v2, const glmVec3& v3,
    const glmVec3& xTilt0, const glmVec3& xTilt1, const glmVec3& xTilt2, const glmVec3& xTilt3, glmVec3& n) {
    glmVec3 x01 = x1 - x0;
    glmVec3 x02 = x2 - x0;
    glmVec3 x03 = x3 - x0;
    glmVec3 v01 = v1 - v0;
    glmVec3 v02 = v2 - v0;
    glmVec3 v03 = v3 - v0;
    dataType a0 = stp(x01, x02, x03);
    dataType a1 = stp(v01, x02, x03) + stp(x01, v02, x03) + stp(x01, x02, v03);
    dataType a2 = stp(x01, v02, v03) + stp(v01, x02, v03) + stp(v01, v02, x03);
    dataType a3 = stp(v01, v02, v03);
    if (abs(a0) < 1e-6 * length(x01) * length(x02) * length(x03))
        return 1.0; // initially coplanar
    dataType t[3];
    dataType minRoot = FLT_MAX;
    int nsol = solveCubic<dataType>(a3, a2, a1, a0, t);
    for (int i = 0; i < nsol; i++) {
        if (t[i] < -1e-3 || t[i] > 1)
            continue;
        glmVec3 xt0 = x0 + t[i] * v0, xt1 = x1 + t[i] * v1,
            xt2 = x2 + t[i] * v2, xt3 = x3 + t[i] * v3;
        glmVec4 w;
        dataType d;
        bool inside;
        d = signed_vf_distance(xt0, xt1, xt2, xt3, &n, w);
        inside = (glm::min(-w[1], glm::min(-w[2], -w[3])) >= -1e-3);
        if (glm::dot(n, w[1] * v1 + w[2] * v2 + w[3] * v3) > 0)
            n = -n;
        if (abs(d) < 1e-6 && inside)
            return t[i];
    }
    return 1.0;
}

__host__ __device__ dataType ccdTetrahedronIntersectionTest(const glmVec3& X0, const glmVec3& XTilt,
    const glmVec3& x0, const glmVec3& x1, const glmVec3& x2, const glmVec3& x3,
    const glmVec3 xTilt0, const glmVec3 xTilt1, const glmVec3 xTilt2, const glmVec3 xTilt3) {
    const glmVec3& V0 = XTilt - X0;
    const glmVec3 v0 = xTilt0 - x0;
    const glmVec3 v1 = xTilt1 - x1;
    const glmVec3 v2 = xTilt2 - x2;
    const glmVec3 v3 = xTilt3 - x3;
    dataType t = 1.f;
    glmVec3 nor;
    t = glm::min(t, ccdTriangleIntersectionTest(X0, V0, x0, x1, x2, v0, v1, v2, XTilt, xTilt0, xTilt1, xTilt2, nor));
    t = glm::min(t, ccdTriangleIntersectionTest(X0, V0, x0, x1, x3, v0, v1, v3, XTilt, xTilt0, xTilt1, xTilt3, nor));
    t = glm::min(t, ccdTriangleIntersectionTest(X0, V0, x0, x2, x3, v0, v2, v3, XTilt, xTilt0, xTilt2, xTilt3, nor));
    t = glm::min(t, ccdTriangleIntersectionTest(X0, V0, x1, x2, x3, v1, v2, v3, XTilt, xTilt1, xTilt2, xTilt3, nor));
    return t;
}

__host__ __device__ dataType tetrahedronTrajIntersectionTest(const GLuint* tets, const glmVec3& X0, const glmVec3& XTilt, const glm::vec3* Xs, const glm::vec3* XTilts, GLuint tetId) {
    const glmVec3& x0 = Xs[tets[tetId * 4 + 0]];
    const glmVec3& x1 = Xs[tets[tetId * 4 + 1]];
    const glmVec3& x2 = Xs[tets[tetId * 4 + 2]];
    const glmVec3& x3 = Xs[tets[tetId * 4 + 3]];

    const glmVec3& xTilt0 = XTilts[tets[tetId * 4 + 0]];
    const glmVec3& xTilt1 = XTilts[tets[tetId * 4 + 1]];
    const glmVec3& xTilt2 = XTilts[tets[tetId * 4 + 2]];
    const glmVec3& xTilt3 = XTilts[tets[tetId * 4 + 3]];

    return ccdTetrahedronIntersectionTest(X0, XTilt, x0, x1, x2, x3, xTilt0, xTilt1, xTilt2, xTilt3);
}