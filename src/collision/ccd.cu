#include "hip/hip_runtime.h"
#pragma once

#include <collision/bvh.h>
#include <utilities.cuh>
#include <surfaceshader.h>
#include <context.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>

__constant__ double AABBThreshold = 0.01;

template<typename Scalar>
struct HighPtoFloatP {
    __host__ __device__ glm::vec3 operator()(const glm::tvec3<Scalar>& d) {
        return glm::vec3(static_cast<float>(d.x), static_cast<float>(d.y), static_cast<float>(d.z));
    }
};

template<typename Scalar>
__device__ AABB<Scalar> computeTetTrajBBox(const glm::tvec3<Scalar>& v0, const glm::tvec3<Scalar>& v1, const glm::tvec3<Scalar>& v2, const glm::tvec3<Scalar>& v3,
    const glm::tvec3<Scalar>& v4, const glm::tvec3<Scalar>& v5, const glm::tvec3<Scalar>& v6, const glm::tvec3<Scalar>& v7)
{
    glm::tvec3<Scalar> min, max;
    min.x = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.x, v1.x), v2.x), v3.x), v4.x), v5.x), v6.x), v7.x);
    min.y = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.y, v1.y), v2.y), v3.y), v4.y), v5.y), v6.y), v7.y);
    min.z = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.z, v1.z), v2.z), v3.z), v4.z), v5.z), v6.z), v7.z);
    max.x = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.x, v1.x), v2.x), v3.x), v4.x), v5.x), v6.x), v7.x);
    max.y = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.y, v1.y), v2.y), v3.y), v4.y), v5.y), v6.y), v7.y);
    max.z = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.z, v1.z), v2.z), v3.z), v4.z), v5.z), v6.z), v7.z);

    return AABB<Scalar>{ min - (Scalar)AABBThreshold, max + (Scalar)AABBThreshold };
}

template<typename Scalar>
__device__ AABB<Scalar> computeTriTrajBBox(const glm::tvec3<Scalar>& v0, const glm::tvec3<Scalar>& v1, const glm::tvec3<Scalar>& v2, const glm::tvec3<Scalar>& v3,
    const glm::tvec3<Scalar>& v4, const glm::tvec3<Scalar>& v5)
{
    glm::tvec3<Scalar> min, max;
    min.x = fminf(fminf(fminf(fminf(fminf(v0.x, v1.x), v2.x), v3.x), v4.x), v5.x);
    min.y = fminf(fminf(fminf(fminf(fminf(v0.y, v1.y), v2.y), v3.y), v4.y), v5.y);
    min.z = fminf(fminf(fminf(fminf(fminf(v0.z, v1.z), v2.z), v3.z), v4.z), v5.z);
    max.x = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.x, v1.x), v2.x), v3.x), v4.x), v5.x);
    max.y = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.y, v1.y), v2.y), v3.y), v4.y), v5.y);
    max.z = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.z, v1.z), v2.z), v3.z), v4.z), v5.z);

    return AABB<Scalar>{ min - (Scalar)AABBThreshold, max + (Scalar)AABBThreshold };
}

template __device__ AABB<float> computeTriTrajBBox(const glm::tvec3<float>& v0, const glm::tvec3<float>& v1, const glm::tvec3<float>& v2, const glm::tvec3<float>& v3,
    const glm::tvec3<float>& v4, const glm::tvec3<float>& v5);

template __device__ AABB<double> computeTriTrajBBox(const glm::tvec3<double>& v0, const glm::tvec3<double>& v1, const glm::tvec3<double>& v2, const glm::tvec3<double>& v3,
    const glm::tvec3<double>& v4, const glm::tvec3<double>& v5);

template<typename Scalar>
AABB<Scalar> AABB<Scalar>::expand(const AABB<Scalar>& aabb)const {
    return AABB<Scalar>{
        glm::min(min, aabb.min),
            glm::max(max, aabb.max)
    };
}

template AABB<float> AABB<float>::expand(const AABB<float>& aabb)const;
template AABB<double> AABB<double>::expand(const AABB<double>& aabb)const;

template<typename Scalar>
CollisionDetection<Scalar>::CollisionDetection(const SolverData<Scalar>* solverData, const Context* context, const int _threadsPerBlock, size_t _maxNumQueries) :
    mpSolverData(solverData), mpContext(context), threadsPerBlock(_threadsPerBlock), maxNumQueries(_maxNumQueries), m_bvh(_threadsPerBlock)
{
    hipMalloc(&dev_queries, maxNumQueries * sizeof(Query));

    hipMalloc(&dev_numQueries, sizeof(size_t));
    hipMemset(dev_numQueries, 0, sizeof(size_t));

    hipMalloc(&dev_overflowFlag, sizeof(bool));
    mSqDisplay.create();
}

template<typename Scalar>
CollisionDetection<Scalar>::~CollisionDetection<Scalar>()
{
    hipFree(dev_queries);
    hipFree(dev_numQueries);
    hipFree(dev_overflowFlag);
}

__global__ void processQueries(const Query* queries, int numQueries, glm::vec4* color) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numQueries) {
        Query q = queries[idx];
        atomicAdd(&color[q.v0].x, 0.05);
        atomicAdd(&color[q.v0].y, 0.05);
        atomicExch(&color[q.v0].w, 1);
    }
}

template<typename Scalar>
void CollisionDetection<Scalar>::PrepareRenderData()
{
    if (mpContext->guiData->QueryVis && numQueries > 0) {
        glm::vec3* pos;
        glm::vec4* col;
        MapDevicePosPtr(&pos, &col);
        thrust::device_ptr<const glm::tvec3<Scalar>> dvec3_ptr(mpSolverData->X);
        thrust::device_ptr<glm::vec3> vec3_ptr(pos);

        thrust::transform(dvec3_ptr, dvec3_ptr + numVerts, vec3_ptr,
            [] __host__ __device__(const glm::tvec3<Scalar> &d) {
            return glm::vec3(static_cast<float>(d.x), static_cast<float>(d.y), static_cast<float>(d.z));
        });
        hipMemset(col, 0, numVerts * sizeof(glm::vec4));
        dim3 numBlocks((numQueries + threadsPerBlock - 1) / threadsPerBlock);
        processQueries << <numBlocks, threadsPerBlock >> > (dev_queries, numQueries, col);
        UnMapDevicePtr();
    }
    if (mpContext->guiData->BVHVis) {
        m_bvh.PrepareRenderData();
    }
}

template<typename Scalar>
void CollisionDetection<Scalar>::Draw(SurfaceShader* flatShaderProgram)
{
    if (mpContext->guiData->BVHVis)
        flatShaderProgram->draw(m_bvh, 0);
    if (mpContext->guiData->QueryVis)
        flatShaderProgram->drawPoints(*this);
    if (mpContext->guiData->QueryDebugMode) {
        glLineWidth(mpContext->guiData->LineWidth);
        flatShaderProgram->drawSingleQuery(GetSQDisplay(mpContext->guiData->CurrQueryId, mpSolverData->X,
            mpContext->guiData->QueryDirty ? mpContext->guiData->mPQuery : nullptr));
        mpContext->guiData->QueryDirty = false;
    }
}

template<typename Scalar>
SingleQueryDisplay& CollisionDetection<Scalar>::GetSQDisplay(int i, const glm::tvec3<Scalar>* X, Query* guiQuery)
{
    if (numQueries == 0) {
        mSqDisplay.SetCount(0);
        return mSqDisplay;
    }
    mSqDisplay.SetCount(6);
    Query q;
    hipMemcpy(&q, &dev_queries[i], sizeof(Query), hipMemcpyDeviceToHost);
    if (guiQuery)
        *guiQuery = q;
    if (q.type == QueryType::EE) mSqDisplay.SetIsLine(true);
    else mSqDisplay.SetIsLine(false);
    if (mSqDisplay.IsLine()) {
        glm::vec3* pos;
        mSqDisplay.MapDevicePtr(&pos, nullptr, nullptr);
        thrust::device_ptr<glm::vec3> dev_ptr(pos);
        thrust::device_ptr<const glm::tvec3<Scalar>> dev_ptr_X(X);
        thrust::transform(dev_ptr_X + q.v0, dev_ptr_X + q.v0 + 1, dev_ptr, HighPtoFloatP<Scalar>());
        thrust::transform(dev_ptr_X + q.v1, dev_ptr_X + q.v1 + 1, dev_ptr + 1, HighPtoFloatP<Scalar>());
        thrust::transform(dev_ptr_X + q.v2, dev_ptr_X + q.v2 + 1, dev_ptr + 2, HighPtoFloatP<Scalar>());
        thrust::transform(dev_ptr_X + q.v3, dev_ptr_X + q.v3 + 1, dev_ptr + 3, HighPtoFloatP<Scalar>());

        glm::vec3 v0Pos, v1Pos;
        hipMemcpy(&v0Pos, pos + 1, sizeof(glm::vec3), hipMemcpyDeviceToHost);
        hipMemcpy(&v1Pos, pos + 2, sizeof(glm::vec3), hipMemcpyDeviceToHost);

        hipMemcpy(&pos[4], &((v0Pos + v1Pos) / 2.f), sizeof(glm::vec3), hipMemcpyHostToDevice);
        // the third line point from the middle of v0 and v1 towards the normal direction
        glm::vec3 normalPoint = (v0Pos + v1Pos) / 2.f + q.normal * 10.f;
        hipMemcpy(&pos[5], &normalPoint, sizeof(glm::vec3), hipMemcpyHostToDevice);
        mSqDisplay.UnMapDevicePtr(&pos, nullptr, nullptr);
    }
    else {
        glm::vec3* pos, * vertPos, * triPos;
        mSqDisplay.MapDevicePtr(&pos, &vertPos, &triPos);
        thrust::device_ptr<glm::vec3> dev_vertPos(vertPos);
        thrust::device_ptr<glm::vec3> dev_triPos(triPos);
        thrust::device_ptr<const glm::tvec3<Scalar>> dev_ptr_X(X);
        thrust::transform(dev_ptr_X + q.v0, dev_ptr_X + q.v0 + 1, dev_vertPos, HighPtoFloatP<Scalar>());
        thrust::transform(dev_ptr_X + q.v1, dev_ptr_X + q.v1 + 1, dev_triPos, HighPtoFloatP<Scalar>());
        thrust::transform(dev_ptr_X + q.v2, dev_ptr_X + q.v2 + 1, dev_triPos + 1, HighPtoFloatP<Scalar>());
        thrust::transform(dev_ptr_X + q.v3, dev_ptr_X + q.v3 + 1, dev_triPos + 2, HighPtoFloatP<Scalar>());
        glm::vec3 v0Pos;
        hipMemcpy(&v0Pos, vertPos, sizeof(glm::vec3), hipMemcpyDeviceToHost);
        glm::vec3 normalPoint = v0Pos + q.normal * 10.f;
        hipMemcpy(&pos[0], &v0Pos, sizeof(glm::vec3), hipMemcpyHostToDevice);
        hipMemcpy(&pos[1], &normalPoint, sizeof(glm::vec3), hipMemcpyHostToDevice);
        mSqDisplay.UnMapDevicePtr(&pos, &vertPos, &triPos);
    }
    return mSqDisplay;
}

template class CollisionDetection<float>;
template class CollisionDetection<double>;