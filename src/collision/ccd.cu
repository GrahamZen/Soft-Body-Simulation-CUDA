#include "hip/hip_runtime.h"
#pragma once

#include <utilities.cuh>
#include <surfaceshader.h>
#include <collision/bvh.h>
#include <simulation/simulationContext.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>

__constant__ dataType AABBThreshold = 0.01;

__device__ AABB computeTetTrajBBox(const glmVec3& v0, const glmVec3& v1, const glmVec3& v2, const glmVec3& v3,
    const glmVec3& v4, const glmVec3& v5, const glmVec3& v6, const glmVec3& v7)
{
    glmVec3 min, max;
    min.x = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.x, v1.x), v2.x), v3.x), v4.x), v5.x), v6.x), v7.x);
    min.y = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.y, v1.y), v2.y), v3.y), v4.y), v5.y), v6.y), v7.y);
    min.z = fminf(fminf(fminf(fminf(fminf(fminf(fminf(v0.z, v1.z), v2.z), v3.z), v4.z), v5.z), v6.z), v7.z);
    max.x = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.x, v1.x), v2.x), v3.x), v4.x), v5.x), v6.x), v7.x);
    max.y = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.y, v1.y), v2.y), v3.y), v4.y), v5.y), v6.y), v7.y);
    max.z = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(v0.z, v1.z), v2.z), v3.z), v4.z), v5.z), v6.z), v7.z);

    return AABB{ min - AABBThreshold, max + AABBThreshold };
}

AABB AABB::expand(const AABB& aabb)const {
    return AABB{
        glm::min(min, aabb.min),
        glm::max(max, aabb.max)
    };
}

CollisionDetection::CollisionDetection(const SimulationCUDAContext* simContext, const int _threadsPerBlock, size_t _maxNumQueries) : mPSimContext(simContext), threadsPerBlock(_threadsPerBlock), maxNumQueries(_maxNumQueries), m_bvh(_threadsPerBlock)
{
    hipMalloc(&dev_queries, maxNumQueries * sizeof(Query));

    hipMalloc(&dev_numQueries, sizeof(size_t));
    hipMemset(dev_numQueries, 0, sizeof(size_t));

    hipMalloc(&dev_overflowFlag, sizeof(bool));
    mSqDisplay.create();
}

CollisionDetection::~CollisionDetection()
{
    hipFree(dev_queries);
    hipFree(dev_numQueries);
    hipFree(dev_overflowFlag);
}

__global__ void processQueries(const Query* queries, int numQueries, glm::vec4* color) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numQueries) {
        Query q = queries[idx];
        atomicAdd(&color[q.v0].x, 0.05);
        atomicAdd(&color[q.v0].y, 0.05);
        atomicExch(&color[q.v0].w, 1);
    }
}

void CollisionDetection::PrepareRenderData()
{
    if (mPSimContext->context->guiData->QueryVis) {
        glm::vec3* pos;
        glm::vec4* col;
        MapDevicePosPtr(&pos, &col);
        hipMemcpy(pos, mPSimContext->mSolverData.X, numVerts * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        hipMemset(col, 0, numVerts * sizeof(glm::vec4));
        dim3 numBlocks((numQueries + threadsPerBlock - 1) / threadsPerBlock);
        processQueries << <numBlocks, threadsPerBlock >> > (dev_queries, numQueries, col);
        UnMapDevicePtr();
    }
    if (mPSimContext->context->guiData->BVHVis) {
        m_bvh.PrepareRenderData();
    }
}

void CollisionDetection::Draw(SurfaceShader* flatShaderProgram)
{
    if (mPSimContext->context->guiData->BVHVis)
        flatShaderProgram->draw(m_bvh, 0);
    if (mPSimContext->context->guiData->QueryVis)
        flatShaderProgram->drawPoints(*this);
    if (mPSimContext->context->guiData->QueryDebugMode) {
        glLineWidth(mPSimContext->context->guiData->LineWidth);
        flatShaderProgram->drawSingleQuery(GetSQDisplay(mPSimContext->context->guiData->CurrQueryId, mPSimContext->mSolverData.X,
            mPSimContext->context->guiData->QueryDirty ? mPSimContext->context->guiData->mPQuery : nullptr));
        mPSimContext->context->guiData->QueryDirty = false;
    }
}

template<typename HighP>
SingleQueryDisplay& CollisionDetection::GetSQDisplay(int i, const glm::tvec3<HighP>* X, Query* guiQuery)
{
    if (numQueries == 0) {
        mSqDisplay.SetCount(0);
        return mSqDisplay;
    }
    mSqDisplay.SetCount(6);
    Query q;
    hipMemcpy(&q, &dev_queries[i], sizeof(Query), hipMemcpyDeviceToHost);
    if (guiQuery)
        *guiQuery = q;
    if (q.type == QueryType::EE) mSqDisplay.SetIsLine(true);
    else mSqDisplay.SetIsLine(false);
    if (mSqDisplay.IsLine()) {
        glm::vec3* pos;
        mSqDisplay.MapDevicePtr(&pos, nullptr, nullptr);
        hipMemcpy(pos, &X[q.v0], sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        hipMemcpy(pos + 1, &X[q.v1], sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        hipMemcpy(pos + 2, &X[q.v2], sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        hipMemcpy(pos + 3, &X[q.v3], sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        glm::vec3 v0Pos, v1Pos;
        hipMemcpy(&v0Pos, &X[q.v0], sizeof(glm::vec3), hipMemcpyDeviceToHost);
        hipMemcpy(&v1Pos, &X[q.v1], sizeof(glm::vec3), hipMemcpyDeviceToHost);
        hipMemcpy(&pos[4], &((v0Pos + v1Pos) / 2.f), sizeof(glm::vec3), hipMemcpyHostToDevice);
        // the third line point from the middle of v0 and v1 towards the normal direction
        glm::vec3 normalPoint = (v0Pos + v1Pos) / 2.f + q.normal * 10.f;
        hipMemcpy(&pos[5], &normalPoint, sizeof(glm::vec3), hipMemcpyHostToDevice);
        mSqDisplay.UnMapDevicePtr();
    }
    else {
        glm::vec3* pos, * vertPos, * triPos;
        mSqDisplay.MapDevicePtr(&pos, &vertPos, &triPos);
        hipMemcpy(vertPos, &X[q.v0], sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        hipMemcpy(triPos, &X[q.v1], sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        hipMemcpy(triPos + 1, &X[q.v2], sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        hipMemcpy(triPos + 2, &X[q.v3], sizeof(glm::vec3), hipMemcpyDeviceToDevice);
        glm::vec3 v0Pos;
        hipMemcpy(&v0Pos, &X[q.v0], sizeof(glm::vec3), hipMemcpyDeviceToHost);
        glm::vec3 normalPoint = v0Pos + q.normal * 10.f;
        hipMemcpy(&pos[0], &v0Pos, sizeof(glm::vec3), hipMemcpyHostToDevice);
        hipMemcpy(&pos[1], &normalPoint, sizeof(glm::vec3), hipMemcpyHostToDevice);
        mSqDisplay.UnMapDevicePtr();
    }
    return mSqDisplay;
}
